#include "hip/hip_runtime.h"
#include "attention.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>

#include <thrust/extrema.h>

#define FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])


// lock-based
__device__ volatile int g_mutex;
// GPU lock-based synchronization function
__device__ void __gpu_sync(int goalVal )
{
    // thread ID in a block
    int tid_in_block = threadIdx.x * blockDim.y + threadIdx.y;
    // only thread 0 is used for synchronization
    if (tid_in_block == 0)
    {
    	atomicAdd((int*) &g_mutex, 1);
    	// only when all blocks add 1 go g_mutex
    	// will g_mutex equal to goalVal
    	while (g_mutex != goalVal)
    	{
    		// Do nothing here
    	}
    }
    __syncthreads();
}
  

template <class DataType>
__global__ void sparse_attention(DataType *a,  DataType *b,  DataType *c, DataType *out,const int *select_index,const int block_size,const int head_size){


    const int tidx = threadIdx.x;
    const int tidy = threadIdx.y;
    const int bidx = blockIdx.x;
    const int bidy = blockIdx.y;
    const int b_dimx = blockDim.x;
    const int g_dimy = gridDim.y;


    const int data_offset_a = (bidx*g_dimy + bidy) * block_size * head_size;
    // const int data_offset_b = (tidx*select_index[]) * block_size * K;
    const int data_offset_b = (select_index[(bidx*g_dimy+bidy)*11+tidx]*g_dimy +bidy) * block_size * head_size;
    const int data_offset_out = tidx*block_size;

    const int A_BM = 32;
    const int A_BK = 64;
    const int B_BK = 32;
    const int B_BN = 4;
    const int C_BK = 32;
    const int C_BN = 4;


    __shared__ float smem_a[A_BM*A_BK],smem_b[11][B_BK*B_BN],temp_score[11][A_BM*B_BN],smem_c[11][C_BK*C_BN];

    __shared__ float out_temp[A_BM*64],sum_scores[11][32],global_sum_scores[32],max_values[11][32],pre_max_score[32],max_score[32];

    float zero4[4] = {0.0f,0.0f,0.0f,0.0f};



    const int block_dim_x = blockDim.x;

    for(int a_bm = 0; a_bm< block_size/A_BM; a_bm++){
        if(tidx < 8){
            const int smem_index =  32*8*tidx + tidy*8; // warp_size * 8
            const int global_a_index_i = (smem_index / 32);
            const int global_a_index_j = (smem_index % 32 + a_bm*A_BM);

            #pragma unroll
            for(int i=0;i<8;i+=4){
                FLOAT4(smem_a[smem_index+i]) = FLOAT4(a[data_offset_a + global_a_index_i*block_size+global_a_index_j+i]); 
                FLOAT4(out_temp[smem_index+i]) = FLOAT4(zero4[0]);
            }
        }

        if(tidx == 8){
            max_score[tidy] = 0.0f;
            // sum_score_max[tidy] = 0.0f;
            pre_max_score[tidy] = 0.0f;
            global_sum_scores[tidy] = 0.0f;
        }

        for(int b_bn=0;b_bn<block_size/B_BN;b_bn++){
            #pragma unroll
            for(int b_bk=0;b_bk<head_size/B_BK;b_bk++){

                const int smem_index = tidy*B_BN; // warp_size * 8
                const int global_b_index_i = (smem_index / 32 + b_bn*B_BN);
                const int global_b_index_j = (smem_index % 32 + b_bk*B_BK);
                #pragma unroll
                for(int i=0;i<B_BN;i+=4){
                    FLOAT4(smem_b[tidx][smem_index+i]) = FLOAT4(b[data_offset_b+global_b_index_i*head_size+global_b_index_j+i]);
                }

                // const int smem_index = tidy*B_BN;
                // const int global_b_index_i = tidy + b_bk*B_BK;
                // const int global_b_index_j = b_bn*B_BN;
                // for(int i=0;i<B_BN;i+=4){
                //     FLOAT4(smem_b[tidx][smem_index+i]) = FLOAT4(b[data_offset_b+global_b_index_i*block_size+global_b_index_j+i]);
                // }

                if(b_bk == 0){
                    for(int i=0;i<B_BN;i+=4){
                        FLOAT4(temp_score[tidx][smem_index+i]) = FLOAT4(zero4[0]);
                    }
                }
                __syncthreads();

                for(int i=0;i<B_BK;i++){
                    for(int j=0;j<B_BN;j++){
                        temp_score[tidx][j*32+tidy] += smem_a[(i+b_bk*B_BK)*32+tidy]*smem_b[tidx][j*B_BK+i];
                    }
                }
                __syncthreads();

            }
            //计算最大值 rowmax
            {
                float max_value = 0.0;
                #pragma unroll
                for(int i=0;i<B_BN;i++){
                    if(max_value<temp_score[tidx][i*32+tidy]){
                        max_value = temp_score[tidx][i*32+tidy];
                    }
                }
                max_values[tidx][tidy] = max_value;
                sum_scores[tidx][tidy] = 0;
                if(tidx == 0){
                    pre_max_score[tidy] = max_score[tidy];
                    float sum = 0.0;
                    #pragma unroll
                    for(int i=0;i<11;i++){
                        if(max_score[tidy] < max_values[i][tidy]){
                            max_score[tidy] = max_values[i][tidy];
                        }
                    }
                }
            }

            //计算差值
            {
                __syncthreads();
                #pragma unroll
                for(int i=0;i<B_BN;i++){
                    float temp =  exp(temp_score[tidx][i*32+tidy] - max_score[tidy]);
                    temp_score[tidx][i*32+tidy] = temp;
                    sum_scores[tidx][tidy] += temp;
                }
                float diff = pre_max_score[tidy] - max_score[tidy];
                if(tidx < 8){
                    const int smem_index =  tidy*64 + tidx*8;
                    if(diff != 0)
                    {
                        diff = exp(diff);
                        #pragma unroll
                        for(int i=0;i<8;i++){
                            out_temp[smem_index+i] *= diff;
                        }
                    }
                }

                __syncthreads();

                if(tidx == 9){
                    global_sum_scores[tidy] *= exp(diff);
                    for(int i=0;i<11;i++)
                        global_sum_scores[tidy] += sum_scores[i][tidy];
                }
            }

            // v0
            // for(int c_bk=0;c_bk<K/C_BK;c_bk++){
            //     const int smem_index = tidy*4; // warp_size * 8
            //     const int global_c_index_i = (smem_index / 32 + b_bn*B_BN + tidx*64);
            //     const int global_c_index_j = (smem_index % 32 + c_bk*C_BK);
            //     for(int i=0;i<C_BN;i+=4){
            //         FLOAT4(smem_c[tidx][smem_index+i]) = FLOAT4(b[global_c_index_i*K+global_c_index_j+i]);
            //     }

            //     __syncthreads();
            //     for(int i=0;i<C_BK;i++){
            //         int temp = i + tidx*2;
            //         temp = temp < 32 ? temp:temp-32;
            //         for(int j=0;j<B_BN;j++){
            //             const int out_global_index_i = tidy;
            //             const int out_global_index_j = temp + c_bk*C_BK;
            //             const int index = out_global_index_i*64+out_global_index_j;
            //             const float score = temp_score[tidx][j*32+tidy];

            //             out_temp[index]  += score*smem_c[tidx][j*32+temp];

            //         }
            //         if(i&1){
            //             __syncthreads();
            //         }
            //     }

            // }

            // v1
            #pragma unroll
            for(int c_bk=0;c_bk<head_size/C_BK;c_bk++){
                const int smem_index = tidy*4; // warp_size * 8
                const int global_c_index_i = (smem_index / 32 + b_bn*B_BN);
                const int global_c_index_j = (smem_index % 32 + c_bk*C_BK);
                #pragma unroll
                for(int i=0;i<C_BN;i+=4){
                    FLOAT4(smem_c[tidx][smem_index+i]) = FLOAT4(c[data_offset_b+global_c_index_i*head_size+global_c_index_j+i]);
                }

                __syncthreads();
                #pragma unroll
                for(int i = 0;i<32;i += b_dimx){
                    int threadx = i+b_dimx < 32 ? b_dimx : 32 - i;
                    if(tidx < threadx){
                        for(int j=0;j<44;j++){
                            const int out_global_index_i = tidx + i;
                            const int out_global_index_j = tidy + c_bk*C_BK;
                            const int index = out_global_index_i*64+out_global_index_j;
                            const float score = temp_score[j/4][(j%4)*32+out_global_index_i];
                            out_temp[index]  += score*smem_c[j/4][(j%4)*32+tidy];
                        }
                    }
                }
                __syncthreads();
            }
        }

        if(tidx < 8){

            const int index_x = (tidx%4)*8;
            const int index_y = tidy + (tidx/4)*32;

            #pragma unroll
            for(int i=0;i<8;i+=1){
                out[data_offset_a+(a_bm*A_BM+index_x+i)*head_size+index_y] = out_temp[(index_x+i)*64+index_y] / global_sum_scores[index_x+i];
                // out[(a_bm*A_BM)*64+smem_index+i] = out_temp[smem_index+i] / global_sum_scores[tidy];

                // out_temp[smem_index+i] = out_temp[smem_index+i] / global_sum_scores[tidy];
            }

        }
        __syncthreads();
        // printf("111\n");
    }
}


void test_cpu(){
    float a = 1008521344.0;
    float b = 3995.0;
    float c = 19228.0;
    float d = 0;
    d = a + b * c;
    printf("%.6f %.6f %.6f %.6f\n",a,b,c,d);
}

void test_gemm_(float *a, float *b,float *c, float *out,int *select_index, int block_num, int head_num,int block_size,int head_size)
{
    // std::cout<<*a<<std::endl;
    // sparse_attention<float><<<dim3(block_num,head_num),dim3(11,32)>>>(a,b,c,out,select_index,64,64);

    hipEvent_t start,stop;
    hipEventCreate( &start );
    hipEventCreate( &stop ) ;
    // test_gemm<float><<<1,dim3(11,32)>>>(a,b,c,m,n,k,64);

    hipEventRecord( start, 0 ) ;
    sparse_attention<float><<<dim3(block_num,head_num),dim3(11,32)>>>(a,b,c,out,select_index,64,64);

    // test_gpu<<<1,1>>>();
    // test_cpu();
    hipEventRecord(stop,0);
    float elapsedTime;
    hipEventSynchronize(stop);
    hipDeviceSynchronize();
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf( "Time to generate:  %f ms\n", elapsedTime );
    // printf("%f\n",*a);

}
