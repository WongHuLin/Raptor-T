#include "hip/hip_runtime.h"
#include "attention.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <cooperative_groups/memcpy_async.h>
#include <thrust/extrema.h>

#define FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])


// lock-based
__device__ volatile int g_mutex;
// GPU lock-based synchronization function
__device__ void __gpu_sync(int goalVal )
{
    // thread ID in a block
    int tid_in_block = threadIdx.x * blockDim.y + threadIdx.y;
    // only thread 0 is used for synchronization
    if (tid_in_block == 0)
    {
    	atomicAdd((int*) &g_mutex, 1);
    	// only when all blocks add 1 go g_mutex
    	// will g_mutex equal to goalVal
    	while (g_mutex != goalVal)
    	{
    		// Do nothing here
    	}
    }
    __syncthreads();
}

//(12,64)(32*8)
//input_data: seq_len * all_head_size * 3   bias: all_head_size  q,k,v: seq * all_head_size
template <class DataType>
__global__ void add_bias_and_transpose(DataType *input_data, DataType *bias, DataType *q, DataType *k, DataType *v, int q_offset, int k_offset, int v_offset,int batch_size, int seq_len, int head_num, int block_size, int head_size, int block_num){
    // For K and V: batch_size * seq_len * all_head_size -> batch_size * head_num * block_num * blcok_size * head_size
    // For Q: batch_size * seq_len * all_head_size -> batch_size * head_num * block_num * head_size* blcok_size
    const int tidy = threadIdx.y;
    const int tidx = threadIdx.x;
    const int bidx = blockIdx.x;
    const int bidy = blockIdx.y;
    const int all_head_size = head_num * head_size;
    const int start_read_data_index = bidy * block_size * head_size * 3 * head_num + bidx * head_size;
    const int start_write_data_index = bidx * block_num * block_size * head_size + bidy * block_size * head_size;
    __shared__ float q_bias[64],k_bias[64],v_bias[64];

    
    

    // load bias
    auto block = cooperative_groups::this_thread_block();
    cooperative_groups::memcpy_async(block, q_bias, bias+bidx*head_size+q_offset, sizeof(float)*64);
    cooperative_groups::memcpy_async(block, k_bias, bias+bidx*head_size+all_head_size, sizeof(float)*64);
    cooperative_groups::memcpy_async(block, v_bias, bias+bidx*head_size+all_head_size*2, sizeof(float)*64);
    
    //smem_q 为32*33 是为了转置时，避免bank conflict
    __shared__ float smem_k[16*64],smem_v[16*64],smem_q[32][33];
    int smem_index = (tidy*32+tidx)*4;
    for(int block_row=0;block_row<block_size;block_row+=32){
        for(int block_col=0;block_col<head_size;block_col+=32){

            int smem_row_index = smem_index / 32;
            int smem_col_index = smem_index % 32;
            int read_index = start_read_data_index + head_num*head_size*3*(block_row+smem_row_index) + block_col + smem_col_index;
            // load q k v

            FLOAT4(smem_k[smem_index]) = FLOAT4(input_data[read_index+k_offset]);
            FLOAT4(smem_v[smem_index]) = FLOAT4(input_data[read_index+v_offset]);
            smem_q[smem_row_index][smem_col_index] = input_data[read_index+q_offset];
            smem_q[smem_row_index][smem_col_index+1] = input_data[read_index+q_offset+1];
            smem_q[smem_row_index][smem_col_index+2] = input_data[read_index+q_offset+2];
            smem_q[smem_row_index][smem_col_index+3] = input_data[read_index+q_offset+3];

            cooperative_groups::wait(block);
            for(int i=0;i<4;i++){
                smem_k[(tidy*4+i)*32+tidx] += k_bias[tidx+block_col];
                smem_v[(tidy*4+i)*32+tidx] += v_bias[tidx+block_col];
                smem_q[(tidy*4+i)][tidx] += q_bias[tidx+block_col];
            }

            int write_index = start_write_data_index + (block_row+smem_row_index)*head_size + block_col + smem_col_index;
            FLOAT4(k[write_index]) = FLOAT4(smem_k[smem_index]);
            FLOAT4(v[write_index]) = FLOAT4(smem_v[smem_index]);

            __syncthreads();
            for(int i=0;i<4;i++){
                q[start_write_data_index + (tidy*4+i+block_col)*64 + tidx + block_row] = smem_q[tidx][(tidy*4+i)];
            }
            __syncthreads();
        }
    }
}
  

template <class DataType>
__global__ void sparse_attention(DataType *a,  DataType *b,  DataType *c, DataType *out,const int *select_index,const int block_size,const int head_size,const int select_block_num){


    const int tidy = threadIdx.y;
    const int tidx = threadIdx.x;
    const int bidx = blockIdx.x;
    const int bidy = blockIdx.y;
    const int b_dimx = 8;
    const int g_dimy = gridDim.y;

    // 计算Q的起始位置
    const int data_offset_a = (bidx*g_dimy + bidy) * block_size * head_size;

    const int A_BM = 32;
    const int A_BK = 64;
    const int B_BK = 32;
    const int B_BN = 4;
    const int C_BK = 32;
    const int C_BN = 4;


    __shared__ float smem_q[64][32],smem_k[32][64],temp_score[32][32],smem_v[32][64];

    __shared__ float out_temp[32][64],global_sum_scores[32],temp_smem[16][32],pre_max_score[32],max_score[32];

    float zero4[4] = {0.0f,0.0f,0.0f,0.0f};

    auto block_k = cooperative_groups::this_thread_block();
    auto block_v = cooperative_groups::this_thread_block();


    const int block_dim_x = blockDim.x;

    for(int a_bm = 0; a_bm< block_size/A_BM; a_bm++){
        int data_b_start = (select_index[(bidx*g_dimy+bidy)*11+0]*g_dimy +bidy) * block_size * head_size;
        
        cooperative_groups::memcpy_async(block_k, smem_k[0], b+data_b_start, sizeof(float)*32*64);
        cooperative_groups::memcpy_async(block_v, smem_v[0], c+data_b_start, sizeof(float)*32*64);


        const int smem_index =  32*8*tidy + tidx*8; // warp_size * 8
        const int global_a_index_i = (smem_index / 32 );
        const int global_a_index_j = (smem_index % 32 + a_bm*A_BM);

        // 加载Q的部分数据,32*64
        #pragma unroll
        for(int i=0;i<8;i+=4){
            FLOAT4(smem_q[smem_index/32][smem_index % 32+i]) = FLOAT4(a[data_offset_a + global_a_index_i*head_size+global_a_index_j+i]); 
            FLOAT4(out_temp[smem_index/64][smem_index % 64+i]) = FLOAT4(zero4[0]);
        }

        // 初始化sharedmem
        if(tidy == 0){
            max_score[tidx] = 0.0f;
            // sum_score_max[tidx] = 0.0f;
            pre_max_score[tidx] = 0.0f;
            global_sum_scores[tidx] = 0.0f;
        }

        __syncthreads();

        // 遍历K、V的每一个Block进行计算
        for(int block_id=0;block_id<select_block_num;block_id++)
        {
            // 计算KV块的起始位置
            const int data_offset_b = (select_index[(bidx*g_dimy+bidy)*11+block_id]*g_dimy +bidy) * block_size * head_size;
            
            // KV按照 32*64 的大小进行加载计算
            for(int b_bn=0;b_bn<block_size/32;b_bn++){
                
                // 计算Q*K
                cooperative_groups::wait(block_k);
                for(int i=0;i<4;i++){
                    float temp = 0.0f;
                    for(int j=0;j<64;j++){
                        temp += smem_q[j][tidx] * smem_k[tidy*4+i][j];
                    }
                    temp_score[tidy*4+i][tidx] = temp;
                }

                //加载下一次使用的数据
                const int next_block_id = b_bn == 1 ? block_id+1:block_id;
                const int next_bn = (b_bn + 1) & 1;
                const int data_b_start = (select_index[(bidx*g_dimy+bidy)*11+next_block_id]*g_dimy +bidy) * block_size * head_size;
                __syncthreads();
                if(block_id != select_block_num - 1 || b_bn != 1)
                {
                    cooperative_groups::memcpy_async(block_k, smem_k[0], b+data_b_start+next_bn*32*head_size, sizeof(float)*32*64);
                }
            
                //计算最大值 rowmax
                {
                    int num = 16;
                    while(num >= 1)
                    {
                        if(num == 16)
                        {
                            float value1 = temp_score[tidy][tidx];
                            float value2 = temp_score[tidy+16][tidx];
                            float value3 = temp_score[tidy+8][tidx];
                            float value4 = temp_score[tidy+24][tidx];

                            temp_smem[tidy][tidx] = value1>value2?value1:value2;
                            temp_smem[tidy+8][tidx] = value3>value4?value3:value4;
                        }
                        else if(tidy < num){
                            float value1 = temp_smem[tidy][tidx];
                            float value2 = temp_smem[tidy+num][tidx];
                            temp_smem[tidy][tidx] = value1>value2?value1:value2;
                        }
                        num = num >> 1;
                        __syncthreads();
                    }
                    if(tidy == 0)
                    {
                        pre_max_score[tidx] = max_score[tidx];
                        max_score[tidx] = max_score[tidx]>temp_smem[0][tidx]?max_score[tidx]:temp_smem[0][tidx];
                    }
                }

                //计算差值

                {
                    __syncthreads();
                    for(int i=0;i<4;i++)
                    {
                        float temp =  exp(temp_score[(i+tidy*4)][tidx] - max_score[tidx]);
                        temp_score[(i+tidy*4)][tidx] = temp;
                    }

                    #pragma unroll
                    for(int i=0;i<4;i++){
                        float diff = (pre_max_score[tidy*4+i] - max_score[tidy*4+i]);
                        if(diff != 0){
                            diff = exp(diff);
                            out_temp[tidy*4+i][tidx] *= diff;
                            out_temp[tidy*4+i][tidx+32] *= diff;
                        }
                    }
                    __syncthreads();
                    if(tidy == 0){
                        float diff = exp(pre_max_score[tidx] - max_score[tidx]);
                        global_sum_scores[tidx] *= diff;
                    }
                    

                }
                {
                    int num = 16;
                    while(num >= 1)
                    {
                        if(num == 16)
                        {
                            int value1 = temp_score[tidy][tidx];
                            int value2 = temp_score[tidy+16][tidx];
                            int value3 = temp_score[tidy+8][tidx];
                            int value4 = temp_score[tidy+24][tidx];
                            temp_smem[tidy][tidx] = value1 + value2;
                            temp_smem[tidy+8][tidx] = value3 + value4;
                        }
                        else if(tidy < num){
                            int value1 = temp_smem[tidy][tidx];
                            int value2 = temp_smem[tidy+num][tidx];
                            temp_smem[tidy][tidx] = value1 + value2;
                        }
                        num = num >> 1;
                        __syncthreads();
                    }
                    if(tidy == 0)
                        global_sum_scores[tidx] += temp_smem[0][tidx];
                }

                //计算S*V
                cooperative_groups::wait(block_v);

                #pragma unroll
                for(int i = 0;i<4; i++){
                    for(int j=0;j<32;j++){
                        out_temp[tidy*4+i][tidx] += temp_score[j][tidy*4+i]*smem_v[j][tidx];
                        out_temp[tidy*4+i][tidx+32] += temp_score[j][tidy*4+i]*smem_v[j][tidx+32];
                    }
                }
                __syncthreads();

                //加载下一次使用的数据
                if(block_id != select_block_num - 1 || b_bn != 1)
                {
                    cooperative_groups::memcpy_async(block_v, smem_v[0], c+data_b_start+next_bn*32*head_size, sizeof(float)*32*64);
                }
            }
        }

        const int index_x = (tidy%4)*8;
        const int index_y = tidx + (tidy/4)*32;
        // 结果写入global mem
        #pragma unroll
        for(int i=0;i<8;i+=1){
            out[data_offset_a+(a_bm*A_BM+index_x+i)*head_size+index_y] = out_temp[(index_x+i)][index_y] / global_sum_scores[index_x+i];
        }

        __syncthreads();
        // printf("111\n");
    }
}

void test_add_bias_and_transpose(float *bias,float *input_data,float *q, float *k,float *v, int q_offset, int k_offset, int v_offset,int batch_size, int seq_len, int head_num, int block_size,int block_num, int head_size){
    hipEvent_t start,stop;
    hipEventCreate( &start );
    hipEventCreate( &stop ) ;
    hipEventRecord( start, 0 ) ;
    add_bias_and_transpose<float><<<dim3(head_num,block_num),dim3(32,8)>>>(input_data,bias,q,k,v,q_offset,k_offset,v_offset,batch_size,seq_len,head_num,block_size,head_size,block_num);
    hipEventRecord(stop,0);
    float elapsedTime;
    hipEventSynchronize(stop);
    hipDeviceSynchronize();
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf( "Time to generate:  %f ms\n", elapsedTime );

}

void test_cpu(){
    float a = 1008521344.0;
    float b = 3995.0;
    float c = 19228.0;
    float d = 0;
    d = a + b * c;
    printf("%.6f %.6f %.6f %.6f\n",a,b,c,d);
}

void test_gemm_(float *a, float *b,float *c, float *out,int *select_index, int block_num, int head_num,int block_size,int head_size)
{
    // std::cout<<*a<<std::endl;
    // sparse_attention<float><<<dim3(block_num,head_num),dim3(11,32)>>>(a,b,c,out,select_index,64,64);

    hipEvent_t start,stop;
    hipEventCreate( &start );
    hipEventCreate( &stop ) ;
    // test_gemm<float><<<1,dim3(11,32)>>>(a,b,c,m,n,k,64);

    hipEventRecord( start, 0 ) ;
    sparse_attention<float><<<dim3(block_num,head_num),dim3(32,8)>>>(a,b,c,out,select_index,64,64,11);

    // test_gpu<<<1,1>>>();
    // test_cpu();
    hipEventRecord(stop,0);
    float elapsedTime;
    hipEventSynchronize(stop);
    hipDeviceSynchronize();
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf( "Time to generate:  %f ms\n", elapsedTime );
    // printf("%f\n",*a);

}
