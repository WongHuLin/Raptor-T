#include "hip/hip_runtime.h"
#include "kernel.h"
#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include ""
#include <stdio.h>
#include <iostream>
#include <numeric>
#include <cooperative_groups/memcpy_async.h>
#include <thrust/extrema.h>
#include <mma.h>
#include <hip/hip_cooperative_groups.h>
// Optionally include for memcpy_async() collective
// Optionally include for reduce() collective
#include <cooperative_groups/reduce.h>
// Optionally include for inclusive_scan() and exclusive_scan() collectives
#include <cuda/barrier>

using namespace cooperative_groups;
// Alternatively use an alias to avoid polluting the namespace with collective algorithms
namespace cg = cooperative_groups;

// #include <hipcub/hipcub.hpp>
#define FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])
#define FLOAT(pointer) (reinterpret_cast<float*>(&(pointer)))
#define FLOAT2(pointer) (reinterpret_cast<float2*>(&(pointer)))
#define HALF(pointer) (reinterpret_cast<half*>(&(pointer)))
#define HALF2(pointer) (reinterpret_cast<half2*>(&(pointer)))
using namespace nvcuda;
namespace sparse_transformers {
namespace layers {
namespace kernels {

// a is not transpose
template <class DataType>
__global__ void sparse_attention_with_tensor_core(DataType *a,  DataType *b,  DataType *c, DataType *out, const int *to_select_index,const int *to_select_index_position,  const int block_size,const int head_size,const int select_block_num){

    const int tidy = threadIdx.y;
    const int tidx = threadIdx.x;
    const int bidx = blockIdx.x;
    const int bidy = blockIdx.y;
    const int b_dimx = 8;
    const int g_dimy = gridDim.y;

    // 计算Q的起始位置
    const int from_block_id = bidy;
    const int data_offset_q = (bidx*g_dimy + from_block_id) * block_size * head_size;

    const int A_BM = 32;
    const int A_BK = 64;
    const int B_BK = 32;
    const int B_BN = 4;
    const int C_BK = 32;
    const int C_BN = 4;


    __shared__ half smem_q[32 * 64],smem_k[32 * 64],smem_v[32 * 64],smem_temp_half[64][32];
    __shared__ float temp_score[32][32],out_temp[32][64],max_score_diff[32];
    __shared__ half temp_score_half[32][32];

    __shared__ float global_sum_scores[32],pre_max_score[32],max_score[32],temp_smem[32];

    typedef hipcub::WarpReduce<float> WarpReduce;
    __shared__ typename WarpReduce::TempStorage temp_storage[8];

    float zero4[4] = {0.0f,0.0f,0.0f,0.0f};
    wmma::fragment<wmma::matrix_a, 8, 32, 16, half, wmma::row_major> frag_q[2];
    wmma::fragment<wmma::matrix_b, 8, 32, 16, half, wmma::col_major> frag_k[2];
    wmma::fragment<wmma::accumulator, 8, 32, 16, half> frag_s_out;

    wmma::fragment<wmma::matrix_a, 8, 32, 16, half, wmma::row_major> frag_s[2];
    wmma::fragment<wmma::matrix_b, 8, 32, 16, half, wmma::row_major> frag_v[2];
    wmma::fragment<wmma::accumulator, 8, 32, 16, float> frag_out;

    auto block_k = cooperative_groups::this_thread_block();
    auto block_v = cooperative_groups::this_thread_block();

    for(int a_bm = 0; a_bm< block_size/A_BM; a_bm++){

        const int to_block_start = to_select_index_position[from_block_id];
        const int to_block_end = to_select_index_position[from_block_id+1];

        

        int to_block_id = to_select_index[to_block_start];

        int data_b_start = (bidx*g_dimy+to_block_id) * block_size * head_size;
        const int smem_index =  32*8*tidy + tidx*8; // warp_size * 8

        // 加载Q的部分数据,32*64
        // Q  tidy
        //      --------------------
        //          0     |     1
        //      --------------------
        //          2     |     3
        //      --------------------
        //          4     |     5
        //      --------------------
        //          6     |     7
        //      --------------------
        const int smem_index_q = 32*8*tidy + (tidx/16)*16*8 + (tidx%16)*8;
        const int global_q_index_i = (a_bm*A_BM + (tidx%16)/2 + (tidy/2)*8);
        const int global_q_index_j = ((tidx%2)*8 + (tidx/16)*16 + (tidy%2)*32);
        FLOAT4(smem_q[smem_index_q]) = FLOAT4(a[data_offset_q+global_q_index_i*head_size+global_q_index_j]); 

        FLOAT4(out_temp[global_q_index_i - a_bm*A_BM][global_q_index_j]) = FLOAT4(zero4[0]);
        // wmma::fill_fragment(frag_s_out, half(0.0));
        wmma::fill_fragment(frag_s_out, __float2half(0.0));
        wmma::fill_fragment(frag_out, 0.0);


        // KT 32*16  tidy  Q*K 8*16*32
        //      -----------------------------------------
        //          0     |     2   |   4     |     6   
        //      -----------------------------------------
        //          1     |     3   |   5     |     7   
        //      -----------------------------------------

        const int global_k_index_i = tidx/2 + (tidy%2)*16;
        const int global_k_index_j = (tidx%2)*8 + (tidy/2)*16;
        FLOAT4(smem_k[smem_index]) = FLOAT4(b[data_b_start + global_k_index_i*head_size+global_k_index_j]); 


        // V 16*32  tidy  S*K 8*16*32
        //      --------------------
        //          0     |     4
        //      --------------------
        //          1     |     5
        //      --------------------
        //          2     |     6
        //      --------------------
        //          3     |     7
        //      --------------------
        const int global_v_index_i = (tidx/4 + ((tidy/2)%2)*16 + (tidy%2)*8);
        const int global_v_index_j = ((tidx*8)%32 + ((tidy/2)/2)*32);
        FLOAT4(smem_v[smem_index]) = FLOAT4(c[data_b_start + global_v_index_i*head_size+global_v_index_j]); 

        wmma::load_matrix_sync(frag_q[0], &smem_q[tidy*2*8*16], 16);
        wmma::load_matrix_sync(frag_q[1], &smem_q[(tidy*2+1)*8*16], 16);

        // 初始化sharedmem
        if(tidy == 0){
            max_score[tidx] = 0.0f;
            // sum_score_max[tidx] = 0.0f;
            pre_max_score[tidx] = 0.0f;
            global_sum_scores[tidx] = 0.0f;
        }


        // 遍历K、V的每一个Block进行计算
        for(int block_id=to_block_start;block_id<to_block_end;block_id++)
        {
            // KV按照 32*64 的大小进行加载计算
            for(int b_bn=0;b_bn<block_size/32;b_bn++){
                
                // 计算Q*K
                // cooperative_groups::wait(block_k);
                wmma::fill_fragment(frag_s_out, __float2half(0.0));

                if(tidy % 2 == 0){
                    wmma::load_matrix_sync(frag_k[0], &smem_k[0], 16);
                    wmma::load_matrix_sync(frag_k[1], &smem_k[16*32], 16);
                }
                else{
                    wmma::load_matrix_sync(frag_k[0], &smem_k[16*32*2], 16);
                    wmma::load_matrix_sync(frag_k[1], &smem_k[16*32*3], 16);
                }
                wmma::mma_sync(frag_s_out, frag_q[0], frag_k[0], frag_s_out);
                wmma::mma_sync(frag_s_out, frag_q[1], frag_k[1], frag_s_out);
                wmma::store_matrix_sync(&smem_temp_half[(tidy/2)*8 + (tidy % 2)*32][0], frag_s_out, 32, wmma::mem_row_major);
                
                __syncthreads();

                for(int i=0;i<4;i++){
                    // temp_score[i*8+tidy][tidx] = __half2float(smem_temp_half[i*8+tidy][tidx] + smem_temp_half[i*8+tidy+32][tidx]);
                    temp_score[i*8+tidy][tidx] = __half2float(smem_temp_half[i*8+tidy][tidx]) + __half2float(smem_temp_half[i*8+tidy+32][tidx]);
                }

                //加载下一次使用的数据
                const int next_block_id = b_bn == 1 ? block_id+1:block_id;
                const int next_bn = (b_bn + 1) & 1;
                to_block_id = to_select_index[next_block_id];
                data_b_start = (bidx*g_dimy+to_block_id) * block_size * head_size;
                if(block_id != to_block_end - 1 || b_bn != 1)
                {
                    FLOAT4(smem_k[smem_index]) = FLOAT4(b[data_b_start + next_bn*32*head_size + global_k_index_i*head_size+global_k_index_j]);
                }
            
                //计算最大值 rowmax
                {
                    float value1 = temp_score[tidy][tidx];
                    float value2 = temp_score[tidy+16][tidx];
                    float value3 = temp_score[tidy+8][tidx];
                    float value4 = temp_score[tidy+24][tidx];
                    
                    temp_smem[tidy] = WarpReduce(temp_storage[tidy]).Reduce(value1, hipcub::Max());
                    temp_smem[tidy+16] = WarpReduce(temp_storage[tidy]).Reduce(value2, hipcub::Max());
                    temp_smem[tidy+8] = WarpReduce(temp_storage[tidy]).Reduce(value3, hipcub::Max());
                    temp_smem[tidy+24] = WarpReduce(temp_storage[tidy]).Reduce(value4, hipcub::Max());

                    __syncthreads();
                    if(tidy == 0)
                    {
                        pre_max_score[tidx] = max_score[tidx];
                        max_score[tidx] = max_score[tidx]>temp_smem[tidx]?max_score[tidx]:temp_smem[tidx];
                        max_score_diff[tidx] = exp(pre_max_score[tidx] - max_score[tidx]);
                    }
                }

                //计算差值
                {
                    __syncthreads();
                    for(int i=0;i<4;i++)
                    {
                        float temp =  exp(temp_score[(i+tidy*4)][tidx] - max_score[tidy*4+i]);
                        temp_score[(i+tidy*4)][tidx] = temp;
                        temp_score_half[(i+tidy*4)][tidx] = __float2half(temp);
                    }

                    const int t = (tidy/2)*8+(tidx%4)*2;
                    for(int i=0;i<4;i++){
                        frag_out.x[i*2] *= max_score_diff[t];
                        frag_out.x[i*2+1] *= max_score_diff[t+1];
                    }
                    __syncthreads();  
                }

                {
                    float value1 = temp_score[tidy][tidx];
                    float value2 = temp_score[tidy+16][tidx];
                    float value3 = temp_score[tidy+8][tidx];
                    float value4 = temp_score[tidy+24][tidx];
                    
                    temp_smem[tidy] = WarpReduce(temp_storage[tidy]).Sum(value1);
                    temp_smem[tidy+16] = WarpReduce(temp_storage[tidy]).Sum(value2);
                    temp_smem[tidy+8] = WarpReduce(temp_storage[tidy]).Sum(value3);
                    temp_smem[tidy+24] = WarpReduce(temp_storage[tidy]).Sum(value4);
                    
                    __syncthreads();

                    if(tidy == 0)
                    {
                        global_sum_scores[tidx] *= max_score_diff[tidx];
                        global_sum_scores[tidx] += temp_smem[tidx];
                    }
                }

                // //计算S*V
                wmma::load_matrix_sync(frag_s[0], &temp_score_half[(tidy/2)*8][0], 32);
                wmma::load_matrix_sync(frag_s[1], &temp_score_half[(tidy/2)*8][16], 32);
                if(tidy % 2 == 0){
                    wmma::load_matrix_sync(frag_v[0], &smem_v[0], 32);
                    wmma::load_matrix_sync(frag_v[1], &smem_v[32*16], 32);
                }
                else{
                    wmma::load_matrix_sync(frag_v[0], &smem_v[32*16*2], 32);
                    wmma::load_matrix_sync(frag_v[1], &smem_v[32*16*3], 32);
                }

                wmma::mma_sync(frag_out, frag_s[0], frag_v[0], frag_out);
                wmma::mma_sync(frag_out, frag_s[1], frag_v[1], frag_out);
                
                //加载下一次使用的数据
                if(block_id != to_block_end - 1 || b_bn != 1)
                {
                    FLOAT4(smem_v[smem_index]) = FLOAT4(c[data_b_start + next_bn*32*head_size + global_v_index_i*head_size+global_v_index_j]); 
                }

            }
        }
        // __syncthreads();

        wmma::store_matrix_sync(&out_temp[(tidy/2)*8][(tidy % 2)*32], frag_out, 64, wmma::mem_row_major);

        __syncthreads();

        const int index_x = (tidy%4)*8;
        const int index_y = tidx + (tidy/4)*32;
        // 结果写入global mem
        #pragma unroll
        for(int i=0;i<8;i+=1){
            out[data_offset_q+(a_bm*A_BM+index_x+i)*head_size+index_y] = __float2half(out_temp[(index_x+i)][index_y] / global_sum_scores[index_x+i]);
        }
        __syncthreads();
    }
}

template <class DataType>
__global__ void sparse_attention(DataType *a,  DataType *b,  DataType *c, 
    DataType *out, const int *to_select_index,const int *to_select_index_position, 
    const int block_size,const int head_size,const int select_block_num){


    const int tidy = threadIdx.y;
    const int tidx = threadIdx.x;
    const int bidx = blockIdx.x;
    const int bidy = blockIdx.y;
    const int b_dimx = 8;
    const int g_dimy = gridDim.y;

    // 计算Q的起始位置
    const int from_block_id = bidy;
    const int data_offset_a = (bidx*g_dimy + from_block_id) * block_size * head_size;

    const int A_BM = 32;
    const int A_BK = 64;
    const int B_BK = 32;
    const int B_BN = 4;
    const int C_BK = 32;
    const int C_BN = 4;


    __shared__ float smem_q[64][32],smem_k[32][64],temp_score[32][32],smem_v[32][64];

    __shared__ float out_temp[32][64],global_sum_scores[32],temp_smem[16][32],pre_max_score[32],max_score[32];

    float zero4[4] = {0.0f,0.0f,0.0f,0.0f};

    auto block_k = cooperative_groups::this_thread_block();
    auto block_v = cooperative_groups::this_thread_block();

    for(int a_bm = 0; a_bm< block_size/A_BM; a_bm++){

        const int to_block_start = to_select_index_position[from_block_id];
        const int to_block_end = to_select_index_position[from_block_id+1];

        int to_block_id = to_select_index[to_block_start];

        int data_b_start = (bidx*g_dimy+to_block_id) * block_size * head_size;
        
        cooperative_groups::memcpy_async(block_k, smem_k[0], b+data_b_start, sizeof(float)*32*64);
        cooperative_groups::memcpy_async(block_v, smem_v[0], c+data_b_start, sizeof(float)*32*64);


        const int smem_index =  32*8*tidy + tidx*8; // warp_size * 8
        const int global_a_index_i = (smem_index / 32 );
        const int global_a_index_j = (smem_index % 32 + a_bm*A_BM);

        // 加载Q的部分数据,32*64
        #pragma unroll
        for(int i=0;i<8;i+=4){
            FLOAT4(smem_q[smem_index/32][smem_index % 32+i]) = FLOAT4(a[data_offset_a + global_a_index_i*head_size+global_a_index_j+i]); 
            FLOAT4(out_temp[smem_index/64][smem_index % 64+i]) = FLOAT4(zero4[0]);
        }

        // 初始化sharedmem
        if(tidy == 0){
            max_score[tidx] = 0.0f;
            // sum_score_max[tidx] = 0.0f;
            pre_max_score[tidx] = 0.0f;
            global_sum_scores[tidx] = 0.0f;
        }

        __syncthreads();

        // 遍历K、V的每一个Block进行计算
        for(int block_id=to_block_start;block_id<to_block_end;block_id++)
        {

            // if(bidx == 0 && bidy == 4 && tidx == 0 && tidy == 0)
            //     printf("%d %d %d %d\n",to_block_start,to_block_end,block_id,to_select_index[block_id]);
            // 计算KV块的起始位置
            
            // KV按照 32*64 的大小进行加载计算
            for(int b_bn=0;b_bn<block_size/32;b_bn++){
                
                // 计算Q*K
                cooperative_groups::wait(block_k);

                //32*64 64*32

                for(int i=0;i<4;i++){
                    float temp = 0.0f;
                    for(int j=0;j<64;j++){
                        temp += smem_q[j][tidx] * smem_k[tidy*4+i][j];
                    }
                    temp_score[tidy*4+i][tidx] = temp;
                }

                //加载下一次使用的数据
                const int next_block_id = b_bn == 1 ? block_id+1:block_id;
                const int next_bn = (b_bn + 1) & 1;
                to_block_id = to_select_index[next_block_id];
                const int data_b_start = (bidx*g_dimy+to_block_id) * block_size * head_size;
                __syncthreads();
                if(block_id != to_block_end - 1 || b_bn != 1)
                {
                    cooperative_groups::memcpy_async(block_k, smem_k[0], b+data_b_start+next_bn*32*head_size, sizeof(float)*32*64);
                }
            
                //计算最大值 rowmax
                {
                    int num = 16;
                    while(num >= 1)
                    {
                        if(num == 16)
                        {
                            float value1 = temp_score[tidy][tidx];
                            float value2 = temp_score[tidy+16][tidx];
                            float value3 = temp_score[tidy+8][tidx];
                            float value4 = temp_score[tidy+24][tidx];

                            temp_smem[tidy][tidx] = value1>value2?value1:value2;
                            temp_smem[tidy+8][tidx] = value3>value4?value3:value4;
                        }
                        else if(tidy < num){
                            float value1 = temp_smem[tidy][tidx];
                            float value2 = temp_smem[tidy+num][tidx];
                            temp_smem[tidy][tidx] = value1>value2?value1:value2;
                        }
                        num = num >> 1;
                        __syncthreads();
                    }
                    if(tidy == 0)
                    {
                        pre_max_score[tidx] = max_score[tidx];
                        max_score[tidx] = max_score[tidx]>temp_smem[0][tidx]?max_score[tidx]:temp_smem[0][tidx];
                    }
                }

                //计算差值
                {
                    __syncthreads();
                    for(int i=0;i<4;i++)
                    {
                        float temp =  exp(temp_score[(i+tidy*4)][tidx] - max_score[tidx]);
                        temp_score[(i+tidy*4)][tidx] = temp;
                    }

                    #pragma unroll
                    for(int i=0;i<4;i++){
                        float diff = (pre_max_score[tidy*4+i] - max_score[tidy*4+i]);
                        if(diff != 0){
                            diff = exp(diff);
                            out_temp[tidy*4+i][tidx] *= diff;
                            out_temp[tidy*4+i][tidx+32] *= diff;
                        }
                    }
                    __syncthreads();
                    if(tidy == 0){
                        float diff = exp(pre_max_score[tidx] - max_score[tidx]);
                        global_sum_scores[tidx] *= diff;
                    }
                    

                }
                
                {
                    int num = 16;
                    while(num >= 1)
                    {
                        if(num == 16)
                        {
                            float value1 = temp_score[tidy][tidx];
                            float value2 = temp_score[tidy+16][tidx];
                            float value3 = temp_score[tidy+8][tidx];
                            float value4 = temp_score[tidy+24][tidx];
                            temp_smem[tidy][tidx] = value1 + value2;
                            temp_smem[tidy+8][tidx] = value3 + value4;
                        }
                        else if(tidy < num){
                            float value1 = temp_smem[tidy][tidx];
                            float value2 = temp_smem[tidy+num][tidx];
                            temp_smem[tidy][tidx] = value1 + value2;
                        }
                        num = num >> 1;
                        __syncthreads();
                    }
                    if(tidy == 0)
                        global_sum_scores[tidx] += temp_smem[0][tidx];
                }

                //计算S*V
                cooperative_groups::wait(block_v);

                #pragma unroll
                for(int i = 0;i<4; i++){
                    for(int j=0;j<32;j++){
                        out_temp[tidy*4+i][tidx] += temp_score[j][tidy*4+i]*smem_v[j][tidx];
                        out_temp[tidy*4+i][tidx+32] += temp_score[j][tidy*4+i]*smem_v[j][tidx+32];
                    }
                }
                __syncthreads();

                //加载下一次使用的数据
                if(block_id != to_block_end - 1 || b_bn != 1)
                {
                    cooperative_groups::memcpy_async(block_v, smem_v[0], c+data_b_start+next_bn*32*head_size, sizeof(float)*32*64);
                }
            }
        }

        const int index_x = (tidy%4)*8;
        const int index_y = tidx + (tidy/4)*32;
        // 结果写入global mem
        #pragma unroll
        for(int i=0;i<8;i+=1){
            out[data_offset_a+(a_bm*A_BM+index_x+i)*head_size+index_y] = out_temp[(index_x+i)][index_y] / global_sum_scores[index_x+i];
        }

        __syncthreads();
        // printf("111\n");
    }
}
    //(12,64)(32*8)  a is transpose 

//32*8
template <class DataType>
__global__ void sparse_attention_with_var(half *a,  half *b,  half *c, 
    half *out,const int *seq_len_info,const int *from_block_index, const int *from_block_index_position, const int *to_select_index,const int *to_select_index_position, const int batch_size,
    const int block_size,const int head_size,const int select_block_num){


    const int tidy = threadIdx.y;
    const int tidx = threadIdx.x;
    const int bidx = blockIdx.x;

    const int A_BM = 32;
    const int A_BK = 64;
    const int B_BK = 32;
    const int B_BN = 4;
    const int C_BK = 32;
    const int C_BN = 4;


    __shared__ half smem_q[32 * 64],smem_k[32 * 64],smem_v[32 * 64],smem_temp_half[64][32];
    __shared__ float temp_score[32][32],out_temp[32][64],max_score_diff[32];
    __shared__ half temp_score_half[32][32];

    __shared__ float global_sum_scores[32],pre_max_score[32],max_score[32],temp_smem[32];

    typedef hipcub::WarpReduce<float> WarpReduce;
    __shared__ typename WarpReduce::TempStorage temp_storage[8];
    wmma::fragment<wmma::matrix_a, 8, 32, 16, half, wmma::row_major> frag_q[2];
    wmma::fragment<wmma::matrix_b, 8, 32, 16, half, wmma::col_major> frag_k[2];
    wmma::fragment<wmma::accumulator, 8, 32, 16, half> frag_s_out;

    wmma::fragment<wmma::matrix_a, 8, 32, 16, half, wmma::row_major> frag_s[2];
    wmma::fragment<wmma::matrix_b, 8, 32, 16, half, wmma::row_major> frag_v[2];
    wmma::fragment<wmma::accumulator, 8, 32, 16, float> frag_out;


    float zero4[4] = {0.0f,0.0f,0.0f,0.0f};

    auto block_k = cooperative_groups::this_thread_block();
    auto block_v = cooperative_groups::this_thread_block();


    // 计算Q的起始位置
    const int compute_block_start = from_block_index_position[bidx];
    const int compute_block_end = from_block_index_position[bidx + 1];
    const int compute_block_num = compute_block_end - compute_block_start;
    for(int from_block_id_index = compute_block_start;from_block_id_index<compute_block_end;from_block_id_index++){
        int from_block_id = from_block_index[from_block_id_index];
        int seq_start_block_index = 0;
        int seq_block_len = 0;
        for(int i = 1;i<batch_size+1;i++){
            if(from_block_id >= seq_len_info[i]*12)
                continue;
            else{
                seq_start_block_index = seq_len_info[i-1];
                seq_block_len = seq_len_info[i] - seq_len_info[i-1];
                break;
            }
        }
        const int head_num = (from_block_id - 12 * seq_start_block_index)/seq_block_len;
        from_block_id = (from_block_id - 12 * seq_start_block_index)%seq_block_len;

        const int seq_start_index = 12 * seq_start_block_index * block_size * head_size + head_num * seq_block_len * block_size * head_size;
        const int data_offset_q = seq_start_index + from_block_id*block_size * head_size;


        // if(tidx == 0 && tidy == 0 ){
        //     printf("%d  %d  %d  %d  %d  %d\n",bidx,head_num,from_block_id,seq_start_block_index,seq_start_index,data_offset_a);
        // }
        
        for(int a_bm = 0; a_bm< block_size/A_BM; a_bm++){
            const int to_block_start = to_select_index_position[from_block_id + seq_start_block_index];
            const int to_block_end = to_select_index_position[from_block_id + seq_start_block_index + 1];

            int to_block_id = to_select_index[to_block_start];
            int data_b_start = seq_start_index + to_block_id * block_size * head_size; 

            const int smem_index =  32*8*tidy + tidx*8; // warp_size * 8

            // 加载Q的部分数据,32*64
            // Q  tidy
            //      --------------------
            //          0     |     1
            //      --------------------
            //          2     |     3
            //      --------------------
            //          4     |     5
            //      --------------------
            //          6     |     7
            //      --------------------
            const int smem_index_q = 32*8*tidy + (tidx/16)*16*8 + (tidx%16)*8;
            const int global_q_index_i = (a_bm*A_BM + (tidx%16)/2 + (tidy/2)*8);
            const int global_q_index_j = ((tidx%2)*8 + (tidx/16)*16 + (tidy%2)*32);
            FLOAT4(smem_q[smem_index_q]) = FLOAT4(a[data_offset_q+global_q_index_i*head_size+global_q_index_j]); 
    
            FLOAT4(out_temp[global_q_index_i - a_bm*A_BM][global_q_index_j]) = FLOAT4(zero4[0]);
            // wmma::fill_fragment(frag_s_out, 0.0);
            wmma::fill_fragment(frag_s_out, __float2half(0.0));

            wmma::fill_fragment(frag_out, 0.0);
    
    
            // KT 32*16  tidy  Q*K 8*16*32
            //      -----------------------------------------
            //          0     |     2   |   4     |     6   
            //      -----------------------------------------
            //          1     |     3   |   5     |     7   
            //      -----------------------------------------
    
            const int global_k_index_i = tidx/2 + (tidy%2)*16;
            const int global_k_index_j = (tidx%2)*8 + (tidy/2)*16;
            FLOAT4(smem_k[smem_index]) = FLOAT4(b[data_b_start + global_k_index_i*head_size+global_k_index_j]); 
    
    
            // V 16*32  tidy  S*K 8*16*32
            //      --------------------
            //          0     |     4
            //      --------------------
            //          1     |     5
            //      --------------------
            //          2     |     6
            //      --------------------
            //          3     |     7
            //      --------------------
            const int global_v_index_i = (tidx/4 + ((tidy/2)%2)*16 + (tidy%2)*8);
            const int global_v_index_j = ((tidx*8)%32 + ((tidy/2)/2)*32);
            FLOAT4(smem_v[smem_index]) = FLOAT4(c[data_b_start + global_v_index_i*head_size+global_v_index_j]); 
    
            wmma::load_matrix_sync(frag_q[0], &smem_q[tidy*2*8*16], 16);
            wmma::load_matrix_sync(frag_q[1], &smem_q[(tidy*2+1)*8*16], 16);
    
            // 初始化sharedmem
            if(tidy == 0){
                max_score[tidx] = 0.0f;
                // sum_score_max[tidx] = 0.0f;
                pre_max_score[tidx] = 0.0f;
                global_sum_scores[tidx] = 0.0f;
            }
            __syncthreads();

            // 遍历K、V的每一个Block进行计算
            for(int block_id=to_block_start;block_id<to_block_end;block_id++)
            {
                // KV按照 32*64 的大小进行加载计算
                for(int b_bn=0;b_bn<block_size/32;b_bn++){
                    // 计算Q*K
                    // cooperative_groups::wait(block_k);
                    wmma::fill_fragment(frag_s_out, __float2half(0.0));
                    if(tidy % 2 == 0){
                        wmma::load_matrix_sync(frag_k[0], &smem_k[0], 16);
                        wmma::load_matrix_sync(frag_k[1], &smem_k[16*32], 16);
                    }
                    else{
                        wmma::load_matrix_sync(frag_k[0], &smem_k[16*32*2], 16);
                        wmma::load_matrix_sync(frag_k[1], &smem_k[16*32*3], 16);
                    }
                    wmma::mma_sync(frag_s_out, frag_q[0], frag_k[0], frag_s_out);
                    wmma::mma_sync(frag_s_out, frag_q[1], frag_k[1], frag_s_out);
                    wmma::store_matrix_sync(&smem_temp_half[(tidy/2)*8 + (tidy % 2)*32][0], frag_s_out, 32, wmma::mem_row_major);

                    __syncthreads();

                    for(int i=0;i<4;i++){
                        temp_score[i*8+tidy][tidx] = __half2float(smem_temp_half[i*8+tidy][tidx]) + __half2float(smem_temp_half[i*8+tidy+32][tidx]);
                    }

                    //加载下一次使用的数据
                    const int next_block_id = b_bn == 1 ? block_id+1:block_id;
                    const int next_bn = (b_bn + 1) & 1;
                    to_block_id = to_select_index[next_block_id];
                    int data_b_start = seq_start_index + to_block_id * block_size * head_size;

                    if(block_id != to_block_end - 1 || b_bn != 1)
                    {
                        FLOAT4(smem_k[smem_index]) = FLOAT4(b[data_b_start + next_bn*32*head_size + global_k_index_i*head_size+global_k_index_j]);

                    }

                    //计算最大值 rowmax
                    {
                        float value1 = temp_score[tidy][tidx];
                        float value2 = temp_score[tidy+16][tidx];
                        float value3 = temp_score[tidy+8][tidx];
                        float value4 = temp_score[tidy+24][tidx];

                        temp_smem[tidy] = WarpReduce(temp_storage[tidy]).Reduce(value1, hipcub::Max());
                        temp_smem[tidy+16] = WarpReduce(temp_storage[tidy]).Reduce(value2, hipcub::Max());
                        temp_smem[tidy+8] = WarpReduce(temp_storage[tidy]).Reduce(value3, hipcub::Max());
                        temp_smem[tidy+24] = WarpReduce(temp_storage[tidy]).Reduce(value4, hipcub::Max());

                        __syncthreads();
                        if(tidy == 0)
                        {
                            pre_max_score[tidx] = max_score[tidx];
                            max_score[tidx] = max_score[tidx]>temp_smem[tidx]?max_score[tidx]:temp_smem[tidx];
                            max_score_diff[tidx] = exp(pre_max_score[tidx] - max_score[tidx]);
                        }
                    }
                    
                    //计算差值
                    {
                        __syncthreads();
                        for(int i=0;i<4;i++)
                        {
                            float temp =  exp(temp_score[(i+tidy*4)][tidx] - max_score[tidy*4+i]);
                            temp_score[(i+tidy*4)][tidx] = temp;
                            temp_score_half[(i+tidy*4)][tidx] = __float2half(temp);
                        }

                        const int t = (tidy/2)*8+(tidx%4)*2;
                        for(int i=0;i<4;i++){
                            frag_out.x[i*2] *= max_score_diff[t];
                            frag_out.x[i*2+1] *= max_score_diff[t+1];
                        }
                        __syncthreads();  
                    }
                    {
                        float value1 = temp_score[tidy][tidx];
                        float value2 = temp_score[tidy+16][tidx];
                        float value3 = temp_score[tidy+8][tidx];
                        float value4 = temp_score[tidy+24][tidx];
                        
                        temp_smem[tidy] = WarpReduce(temp_storage[tidy]).Sum(value1);
                        temp_smem[tidy+16] = WarpReduce(temp_storage[tidy]).Sum(value2);
                        temp_smem[tidy+8] = WarpReduce(temp_storage[tidy]).Sum(value3);
                        temp_smem[tidy+24] = WarpReduce(temp_storage[tidy]).Sum(value4);
                        
                        __syncthreads();

                        if(tidy == 0)
                        {
                            global_sum_scores[tidx] *= max_score_diff[tidx];
                            global_sum_scores[tidx] += temp_smem[tidx];
                        }
                    }


                    // //计算S*V
                    wmma::load_matrix_sync(frag_s[0], &temp_score_half[(tidy/2)*8][0], 32);
                    wmma::load_matrix_sync(frag_s[1], &temp_score_half[(tidy/2)*8][16], 32);
                    if(tidy % 2 == 0){
                        wmma::load_matrix_sync(frag_v[0], &smem_v[0], 32);
                        wmma::load_matrix_sync(frag_v[1], &smem_v[32*16], 32);
                    }
                    else{
                        wmma::load_matrix_sync(frag_v[0], &smem_v[32*16*2], 32);
                        wmma::load_matrix_sync(frag_v[1], &smem_v[32*16*3], 32);
                    }

                    wmma::mma_sync(frag_out, frag_s[0], frag_v[0], frag_out);
                    wmma::mma_sync(frag_out, frag_s[1], frag_v[1], frag_out);
                    
                    __syncthreads();

                    //加载下一次使用的数据
                    if(block_id != to_block_end - 1 || b_bn != 1)
                    {
                        FLOAT4(smem_v[smem_index]) = FLOAT4(c[data_b_start + next_bn*32*head_size + global_v_index_i*head_size+global_v_index_j]); 
                    }

                }
            }

            wmma::store_matrix_sync(&out_temp[(tidy/2)*8][(tidy % 2)*32], frag_out, 64, wmma::mem_row_major);

            __syncthreads();
    
            const int index_x = (tidy%4)*8;
            const int index_y = tidx + (tidy/4)*32;
            // 结果写入global mem
            #pragma unroll
            for(int i=0;i<8;i+=1){
                out[data_offset_q+(a_bm*A_BM+index_x+i)*head_size+index_y] = __float2half(out_temp[(index_x+i)][index_y] / global_sum_scores[index_x+i]);
            }
            __syncthreads();
        }
    }
}

template <class DataType>
__global__ void sparse_attention_without_bank(half *a,  half *b,  half *c, 
    DataType *out,const int *seq_len_info,const int *from_block_index, const int *from_block_index_position, const int *to_select_index,const int *to_select_index_position, const int batch_size,
    const int block_size,const int head_size,const int select_block_num){


    const int tidy = threadIdx.y;
    const int tidx = threadIdx.x;
    const int bidx = blockIdx.x;
    const int bidy = blockIdx.y;


    const int A_BM = 32;
    const int A_BK = 64;
    const int B_BK = 32;
    const int B_BN = 4;
    const int C_BK = 32;
    const int C_BN = 4;
    const int pad = 8;

    __shared__ half smem_q[32][64+pad],smem_k[32][64+pad],smem_v[32][64+pad],smem_temp_half[64][32+pad];
    // __shared__ half smem_q[32][64+pad],smem_k[32 * 64],smem_v[32 * 64],smem_temp_half[64][32+pad];

    __shared__ float temp_score[32][32],out_temp[32][64+4],max_score_diff[32];
    __shared__ half temp_score_half[32][32+8];
    

    __shared__ float global_sum_scores[32],pre_max_score[32],max_score[32],temp_smem[32];

    typedef hipcub::WarpReduce<float> WarpReduce;
    __shared__ typename WarpReduce::TempStorage temp_storage[8];
    wmma::fragment<wmma::matrix_a, 8, 32, 16, half, wmma::row_major> frag_q[2];
    wmma::fragment<wmma::matrix_b, 8, 32, 16, half, wmma::col_major> frag_k[2];
    wmma::fragment<wmma::accumulator, 8, 32, 16, half> frag_s_out;

    wmma::fragment<wmma::matrix_a, 8, 32, 16, half, wmma::row_major> frag_s[2];
    wmma::fragment<wmma::matrix_b, 8, 32, 16, half, wmma::row_major> frag_v[2];
    wmma::fragment<wmma::accumulator, 8, 32, 16, float> frag_out;


    float4 zero4 = {0.0f,0.0f,0.0f,0.0f};

    auto block_k = cooperative_groups::this_thread_block();
    auto block_v = cooperative_groups::this_thread_block();


    // 计算Q的起始位置
    const int compute_block_start = from_block_index_position[bidx];
    const int compute_block_end = from_block_index_position[bidx + 1];
    const int compute_block_num = compute_block_end - compute_block_start;
    for(int from_block_id_index = compute_block_start;from_block_id_index<compute_block_end;from_block_id_index++){
        int from_block_id = from_block_index[from_block_id_index];
        int seq_start_block_index = 0;
        int seq_block_len = 0;
        for(int i = 1;i<batch_size+1;i++){
            if(from_block_id >= seq_len_info[i]*12)
                continue;
            else{
                seq_start_block_index = seq_len_info[i-1];
                seq_block_len = seq_len_info[i] - seq_len_info[i-1];
                break;
            }
        }
        const int head_num = (from_block_id - 12 * seq_start_block_index)/seq_block_len;
        from_block_id = (from_block_id - 12 * seq_start_block_index)%seq_block_len;

        const int seq_start_index = 12 * seq_start_block_index * block_size * head_size + head_num * seq_block_len * block_size * head_size;
        const int data_offset_q = seq_start_index + from_block_id*block_size * head_size;


        // if(tidx == 0 && tidy == 0 ){
        //     printf("%d  %d  %d  %d  %d  %d\n",bidx,head_num,from_block_id,seq_start_block_index,seq_start_index,data_offset_a);
        // }
        
        for(int a_bm = 0; a_bm< block_size/A_BM; a_bm++){
            const int to_block_start = to_select_index_position[from_block_id + seq_start_block_index];
            const int to_block_end = to_select_index_position[from_block_id + seq_start_block_index + 1];

            int to_block_id = to_select_index[to_block_start];
            int data_b_start = seq_start_index + to_block_id * block_size * head_size; 

            const int smem_index =  32*8*tidy + tidx*8; // warp_size * 8

            // 加载Q的部分数据,32*64
            // Q  tidy
            //      --------------------
            //          0     |     1
            //      --------------------
            //          2     |     3
            //      --------------------
            //          4     |     5
            //      --------------------
            //          6     |     7
            //      --------------------
            const int global_q_index_i =  tidy*4+tidx/8;
            const int global_q_index_j = (tidx%8)*8;
            FLOAT4(smem_q[global_q_index_i][global_q_index_j]) = FLOAT4(a[data_offset_q+(global_q_index_i+a_bm*A_BM)*head_size+global_q_index_j]);

            // const int smem_index_q = 32*8*tidy + (tidx/16)*16*8 + (tidx%16)*8;
            // const int global_q_index_i = (a_bm*A_BM + (tidx%16)/2 + (tidy/2)*8);
            // const int global_q_index_j = ((tidx%2)*8 + (tidx/16)*16 + (tidy%2)*32);
            // FLOAT4(smem_q[smem_index_q]) = FLOAT4(a[data_offset_q+global_q_index_i*head_size+global_q_index_j]); 
    
            // FLOAT4(out_temp[global_q_index_i][global_q_index_j]) = zero4;
            // FLOAT4(out_temp[global_q_index_i][global_q_index_j+4]) = zero4;

            // wmma::fill_fragment(frag_s_out, 0.0);
            wmma::fill_fragment(frag_s_out, __float2half(0.0));

            wmma::fill_fragment(frag_out, 0.0);
    
    
            // KT 32*16  tidy  Q*K 8*16*32
            //      -----------------------------------------
            //          0     |     2   |   4     |     6   
            //      -----------------------------------------
            //          1     |     3   |   5     |     7   
            //      -----------------------------------------

            // const int global_q_index_i = a_bm*A_BM + tidy*4+tidx/8;
            // const int global_q_index_j = (tidx%8)*8;
            FLOAT4(smem_k[global_q_index_i][global_q_index_j]) = FLOAT4(b[data_b_start+global_q_index_i*head_size+global_q_index_j]);
            // const int global_k_index_i = tidx/2 + (tidy%2)*16;
            // const int global_k_index_j = (tidx%2)*8 + (tidy/2)*16;
            // FLOAT4(smem_k[smem_index]) = FLOAT4(b[data_b_start + global_k_index_i*head_size+global_k_index_j]); 
    
    
            // V 16*32  tidy  S*K 8*16*32
            //      --------------------
            //          0     |     4
            //      --------------------
            //          1     |     5
            //      --------------------
            //          2     |     6
            //      --------------------
            //          3     |     7
            //      --------------------
            FLOAT4(smem_v[global_q_index_i][global_q_index_j]) = FLOAT4(c[data_b_start+global_q_index_i*head_size+global_q_index_j]);
            // const int global_v_index_i = (tidx/4 + ((tidy/2)%2)*16 + (tidy%2)*8);
            // const int global_v_index_j = ((tidx*8)%32 + ((tidy/2)/2)*32);
            // FLOAT4(smem_v[smem_index]) = FLOAT4(c[data_b_start + global_v_index_i*head_size+global_v_index_j]); 

            __syncthreads();
    
            wmma::load_matrix_sync(frag_q[0], &smem_q[(tidy/2)*8][(tidy%2)*32], 64+pad);
            wmma::load_matrix_sync(frag_q[1], &smem_q[(tidy/2)*8][(tidy%2)*32+16], 64+pad);
            // wmma::load_matrix_sync(frag_q[0], &smem_q[tidy*2*8*16], 16);
            // wmma::load_matrix_sync(frag_q[1], &smem_q[(tidy*2+1)*8*16], 16);
    
            // 初始化sharedmem
            if(tidy == 0){
                max_score[tidx] = 0.0f;
                // sum_score_max[tidx] = 0.0f;
                pre_max_score[tidx] = 0.0f;
                global_sum_scores[tidx] = 0.0f;
            }
            __syncthreads();

            // 遍历K、V的每一个Block进行计算
            for(int block_id=to_block_start;block_id<to_block_end;block_id++)
            {
                // KV按照 32*64 的大小进行加载计算
                for(int b_bn=0;b_bn<block_size/32;b_bn++){
                    // 计算Q*K
                    // cooperative_groups::wait(block_k);
                    wmma::fill_fragment(frag_s_out, __float2half(0.0));
                    if(tidy % 2 == 0){
                        wmma::load_matrix_sync(frag_k[0], &smem_k[0][0], 64+pad);
                        wmma::load_matrix_sync(frag_k[1], &smem_k[0][16], 64+pad);
                    }
                    else{
                        wmma::load_matrix_sync(frag_k[0], &smem_k[0][32], 64+pad);
                        wmma::load_matrix_sync(frag_k[1], &smem_k[0][48], 64+pad);
                    }
                    // if(tidy % 2 == 0){
                    //     wmma::load_matrix_sync(frag_k[0], &smem_k[0], 16);
                    //     wmma::load_matrix_sync(frag_k[1], &smem_k[16*32], 16);
                    // }
                    // else{
                    //     wmma::load_matrix_sync(frag_k[0], &smem_k[16*32*2], 16);
                    //     wmma::load_matrix_sync(frag_k[1], &smem_k[16*32*3], 16);
                    // }
                    wmma::mma_sync(frag_s_out, frag_q[0], frag_k[0], frag_s_out);
                    wmma::mma_sync(frag_s_out, frag_q[1], frag_k[1], frag_s_out);
                    wmma::store_matrix_sync(&smem_temp_half[(tidy/2)*8 + (tidy % 2)*32][0], frag_s_out, 32+pad, wmma::mem_row_major);

                    __syncthreads();

                    for(int i=0;i<4;i++){
                        temp_score[i*8+tidy][tidx] = __half2float(smem_temp_half[i*8+tidy][tidx]) + __half2float(smem_temp_half[i*8+tidy+32][tidx]);
                    }

                    //加载下一次使用的数据
                    const int next_block_id = b_bn == 1 ? block_id+1:block_id;
                    const int next_bn = (b_bn + 1) & 1;
                    to_block_id = to_select_index[next_block_id];
                    int data_b_start = seq_start_index + to_block_id * block_size * head_size;

                    if(block_id != to_block_end - 1 || b_bn != 1)
                    {
                        // FLOAT4(smem_k[smem_index]) = FLOAT4(b[data_b_start + next_bn*32*head_size + global_k_index_i*head_size+global_k_index_j]);

                        FLOAT4(smem_k[global_q_index_i][global_q_index_j]) = FLOAT4(b[data_b_start+next_bn*32*head_size+global_q_index_i*head_size+global_q_index_j]);
                    }

                    //计算最大值 rowmax
                    {
                        float value1 = temp_score[tidy][tidx];
                        float value2 = temp_score[tidy+16][tidx];
                        float value3 = temp_score[tidy+8][tidx];
                        float value4 = temp_score[tidy+24][tidx];

                        temp_smem[tidy] = WarpReduce(temp_storage[tidy]).Reduce(value1, hipcub::Max());
                        temp_smem[tidy+16] = WarpReduce(temp_storage[tidy]).Reduce(value2, hipcub::Max());
                        temp_smem[tidy+8] = WarpReduce(temp_storage[tidy]).Reduce(value3, hipcub::Max());
                        temp_smem[tidy+24] = WarpReduce(temp_storage[tidy]).Reduce(value4, hipcub::Max());

                        __syncthreads();
                        if(tidy == 0)
                        {
                            pre_max_score[tidx] = max_score[tidx];
                            max_score[tidx] = max_score[tidx]>temp_smem[tidx]?max_score[tidx]:temp_smem[tidx];
                            max_score_diff[tidx] = exp(pre_max_score[tidx] - max_score[tidx]);
                        }
                    }
                    
                    //计算差值
                    {
                        __syncthreads();
                        for(int i=0;i<4;i++)
                        {
                            float temp =  exp(temp_score[(i+tidy*4)][tidx] - max_score[tidy*4+i]);
                            temp_score[(i+tidy*4)][tidx] = temp;
                            temp_score_half[(i+tidy*4)][tidx] = __float2half(temp);
                        }

                        const int t = (tidy/2)*8+(tidx%4)*2;
                        for(int i=0;i<4;i++){
                            frag_out.x[i*2] *= max_score_diff[t];
                            frag_out.x[i*2+1] *= max_score_diff[t+1];
                        }
                        // if(tidy == 0 && bidx == 0 && bidy == 0 && a_bm == 0 && block_id==to_block_start && b_bn == 1)
                        // {
                        //     for(int i=0;i<8;i++){
                        //         printf("%d %d %f \n",tidy,tidx,frag_out.x[i]);
                        //     }
                        // }
                       
                        // if(tidy == 0)
                        // wmma::store_matrix_sync(&out_temp[(tidy/2)*8][(tidy % 2)*32], frag_out, 64, wmma::mem_row_major);

                        // __syncthreads();

                        // if(tidx == 0 && tidy == 0 && bidx == 0 && bidy == 0 && a_bm == 0 && block_id==to_block_start && b_bn == 1)
                        // {
                        //     for(int i=0;i<32;i++)
                        //     {
                        //         for(int j=0;j<64;j++)
                        //             printf("%f ",out_temp[i][j]);
                        //         printf("\n");
                        //     }
                        // }
                        __syncthreads();  
                    }
                    {
                        float value1 = temp_score[tidy][tidx];
                        float value2 = temp_score[tidy+16][tidx];
                        float value3 = temp_score[tidy+8][tidx];
                        float value4 = temp_score[tidy+24][tidx];
                        
                        temp_smem[tidy] = WarpReduce(temp_storage[tidy]).Sum(value1);
                        temp_smem[tidy+16] = WarpReduce(temp_storage[tidy]).Sum(value2);
                        temp_smem[tidy+8] = WarpReduce(temp_storage[tidy]).Sum(value3);
                        temp_smem[tidy+24] = WarpReduce(temp_storage[tidy]).Sum(value4);
                        
                        __syncthreads();

                        if(tidy == 0)
                        {
                            global_sum_scores[tidx] *= max_score_diff[tidx];
                            global_sum_scores[tidx] += temp_smem[tidx];
                        }
                    }


                    // //计算S*V
                    wmma::load_matrix_sync(frag_s[0], &temp_score_half[(tidy/2)*8][0], 32+8);
                    wmma::load_matrix_sync(frag_s[1], &temp_score_half[(tidy/2)*8][16], 32+8);
                    if(tidy % 2 == 0){
                        wmma::load_matrix_sync(frag_v[0], &smem_v[0][0], 64+pad);
                        wmma::load_matrix_sync(frag_v[1], &smem_v[16][0], 64+pad);
                    }
                    else{
                        wmma::load_matrix_sync(frag_v[0], &smem_v[0][32], 64+pad);
                        wmma::load_matrix_sync(frag_v[1], &smem_v[16][32], 64+pad);
                    }
                    // if(tidy % 2 == 0){
                    //     wmma::load_matrix_sync(frag_v[0], &smem_v[0], 32);
                    //     wmma::load_matrix_sync(frag_v[1], &smem_v[32*16], 32);
                    // }
                    // else{
                    //     wmma::load_matrix_sync(frag_v[0], &smem_v[32*16*2], 32);
                    //     wmma::load_matrix_sync(frag_v[1], &smem_v[32*16*3], 32);
                    // }

                    wmma::mma_sync(frag_out, frag_s[0], frag_v[0], frag_out);
                    wmma::mma_sync(frag_out, frag_s[1], frag_v[1], frag_out);
                    
                    __syncthreads();

                    //加载下一次使用的数据
                    if(block_id != to_block_end - 1 || b_bn != 1)
                    {
                        // FLOAT4(smem_v[smem_index]) = FLOAT4(c[data_b_start + next_bn*32*head_size + global_v_index_i*head_size+global_v_index_j]); 
                        FLOAT4(smem_v[global_q_index_i][global_q_index_j]) = FLOAT4(c[data_b_start+next_bn*32*head_size+global_q_index_i*head_size+global_q_index_j]);
                    }

                }
            }

            wmma::store_matrix_sync(&out_temp[(tidy/2)*8][(tidy % 2)*32], frag_out, 64+4, wmma::mem_row_major);

            __syncthreads();
    
            const int index_x = (tidy%4)*8;
            const int index_y = tidx + (tidy/4)*32;
            // 结果写入global mem
            #pragma unroll
            for(int i=0;i<8;i+=1){
                out[data_offset_q+(a_bm*A_BM+index_x+i)*head_size+index_y] = out_temp[(index_x+i)][index_y] / global_sum_scores[index_x+i];
            }
            __syncthreads();
        }
    }
}

// template <class DataType>
// __global__ void sparse_attention_(half *a,  half *b,  half *c, 
//     half *out,const int *seq_len_info,const int *from_block_index, const int *from_block_index_position, const int *to_select_index,const int *to_select_index_position, const int batch_size,
//     const int block_size,const int head_size){

//     const int tidy = threadIdx.y;
//     const int tidx = threadIdx.x;
//     const int bidx = blockIdx.x;
//     const int bidy = blockIdx.y;

//     const int A_BM = 32;
//     const int A_BK = 64;
//     const int B_BK = 32;
//     const int B_BN = 4;
//     const int C_BK = 32;
//     const int C_BN = 4;
//     const int pad = 8;

//     typedef hipcub::WarpReduce<float> WarpReduce;
//     __shared__  typename WarpReduce::TempStorage temp_storage[4];
//     __shared__  float global_sum_scores[64],pre_max_score[64],max_score[64];  
//     __shared__  float temp_smem[16],max_score_diff[16];
//     __shared__  half smem_q[2][16][64+pad],smem_k[64][64+pad],smem_v[64][64+pad],smem_temp_score[16][64+pad],out_temp[64][64+pad];


//     wmma::fragment<wmma::matrix_a, 8, 32, 16, half, wmma::row_major> frag_q[4];
//     wmma::fragment<wmma::matrix_b, 8, 32, 16, half, wmma::col_major> frag_k[4];
//     wmma::fragment<wmma::accumulator, 8, 32, 16, half> frag_s_out;

//     wmma::fragment<wmma::matrix_a, 8, 32, 16, half, wmma::row_major> frag_s[4];
//     wmma::fragment<wmma::matrix_b, 8, 32, 16, half, wmma::row_major> frag_v[4];
//     wmma::fragment<wmma::accumulator, 8, 32, 16, half> frag_out;


//     float4 zero4 = {0.0f,0.0f,0.0f,0.0f};

//     // 计算Q的起始位置
//     const int compute_block_start = from_block_index_position[bidx];
//     const int compute_block_end = from_block_index_position[bidx + 1];
//     const int compute_block_num = compute_block_end - compute_block_start;
//     for(int from_block_id_index = compute_block_start;from_block_id_index<compute_block_end;from_block_id_index++){
//         int from_block_id = from_block_index[from_block_id_index];
//         int seq_start_block_index = 0;
//         int seq_block_len = 0;
//         for(int i = 1;i<batch_size+1;i++){
//             if(from_block_id >= seq_len_info[i]*12)
//                 continue;
//             else{
//                 seq_start_block_index = seq_len_info[i-1];
//                 seq_block_len = seq_len_info[i] - seq_len_info[i-1];
//                 break;
//             }
//         }
//         const int head_num = (from_block_id - 12 * seq_start_block_index)/seq_block_len;
//         from_block_id = (from_block_id - 12 * seq_start_block_index)%seq_block_len;

//         const int seq_start_index = 12 * seq_start_block_index * block_size * head_size + head_num * seq_block_len * block_size * head_size;
//         const int data_offset_q = seq_start_index + from_block_id*block_size * head_size;

//         const int to_block_start = to_select_index_position[from_block_id + seq_start_block_index];
//         const int to_block_end = to_select_index_position[from_block_id + seq_start_block_index + 1];
        
//         const int smem_index_i =  tidy*4+tidx/8;
//         const int smem_index_j = (tidx%8)*8;
        

//         FLOAT4(smem_q[0][smem_index_i][smem_index_j]) = FLOAT4(a[data_offset_q+ smem_index_i*head_size+smem_index_j]);

//         int to_block_id = to_select_index[0];
//         int data_k_start = seq_start_index + to_block_id * block_size * head_size;
//         for(int i=0;i<4;i++){
//             FLOAT4(smem_k[i*16 + smem_index_i][smem_index_j]) = FLOAT4(b[data_k_start+(i*16 + smem_index_i)*head_size+smem_index_j]);
//             FLOAT4(smem_v[i*16 + smem_index_i][smem_index_j]) = FLOAT4(c[data_k_start+(i*16 + smem_index_i)*head_size+smem_index_j]);
//         }

//         for(int i=0;i<4;i++){
//             FLOAT4(out_temp[i*16 + smem_index_i][smem_index_j]) = FLOAT4(zero4);
//         }

//         wmma::fill_fragment(frag_out, __float2half(0.0));
//         if(tidy < 2){
//             global_sum_scores[tidy*32+tidx] = 0.0;
//             pre_max_score[tidy*32+tidx] = -1000.0f;
//             max_score[tidy*32+tidx] = -1000.0f;
//         }

//         if(tidy == 0 && tidx < 16)
//         {
//             temp_smem[tidx] = 0.0;
//             max_score_diff[tidx] = 1;
//         }

//         // 遍历K、V的每一个Block进行计算
//         for(int block_id_index=to_block_start;block_id_index<to_block_end;block_id_index++){

//             __syncthreads();

//             for(int i=0;i<4;i++){
//                 wmma::load_matrix_sync(frag_k[i], &smem_k[(tidy%2)*32][i*16], 64+pad);
//                 wmma::load_matrix_sync(frag_v[i], &smem_v[i*16][(tidy%2)*32], 64+pad);
//             }

//             if(block_id_index != to_block_end - 1){
//                 to_block_id = to_select_index[block_id_index+1];
//                 data_k_start = seq_start_index + to_block_id * block_size * head_size;

//                 for(int i=0;i<4;i++){
//                     FLOAT4(smem_k[i*16 + smem_index_i][smem_index_j]) = FLOAT4(b[data_k_start+(i*16 + smem_index_i)*head_size+smem_index_j]);
//                     FLOAT4(smem_v[i*16 + smem_index_i][smem_index_j]) = FLOAT4(c[data_k_start+(i*16 + smem_index_i)*head_size+smem_index_j]);
//                 }
//             }

//             for(int from_block_part_index = 0;from_block_part_index<block_size;from_block_part_index+=16){
//                 wmma::fill_fragment(frag_s_out, __float2half(0.0));

//                 for(int i=0;i<4;i++)
//                 {
//                     wmma::load_matrix_sync(frag_q[i], &smem_q[(from_block_part_index/16)&1][(tidy/2)*8][i*16], 64+pad);
//                 }
//                 for(int i=0;i<4;i++){
//                     wmma::mma_sync(frag_s_out, frag_q[i], frag_k[i], frag_s_out);
//                 }

//                 if(block_id_index != to_block_end - 1 || from_block_part_index != 48)
//                 FLOAT4(smem_q[(from_block_part_index/16 + 1)&1][smem_index_i][smem_index_j]) = FLOAT4(a[data_offset_q+((from_block_part_index+16)%64+ smem_index_i)*head_size+smem_index_j]);

//                 //load next data

//                 wmma::store_matrix_sync(&smem_temp_score[(tidy/2)*8][(tidy%2)*32], frag_s_out, 64+pad, wmma::mem_row_major);
                
//                 __syncthreads();

//                 // 计算最大值 rowmax
//                 {
//                     float value_h2[8];
//                     float score_value[4];
//                     for(int i=0;i<4;i++)
//                     {
//                         value_h2[i*2] = __half2float(smem_temp_score[tidy+i*4][tidx*2]);
//                         value_h2[i*2+1] = __half2float(smem_temp_score[tidy+i*4][tidx*2+1]);
//                     }

//                     for(int i=0;i<4;i++)
//                     {
//                         score_value[i] = value_h2[i*2] > value_h2[i*2+1] ? value_h2[i*2] : value_h2[i*2+1];
//                     }

//                     for(int i=0;i<4;i++)
//                     {
//                         float t = float(score_value[i]);
//                         temp_smem[tidy + i*4] = WarpReduce(temp_storage[tidy]).Reduce(t, hipcub::Max());
//                     }

//                     __syncthreads();
//                     if(tidy == 0 && tidx < 16)
//                     {
//                         int idx = tidx+from_block_part_index;
//                         pre_max_score[idx] = max_score[idx];
//                         max_score[idx] = max_score[idx] > temp_smem[tidx]?max_score[idx]:temp_smem[tidx];
//                         max_score_diff[tidx] = exp(pre_max_score[idx]-max_score[idx]);
//                     }
                    
//                     float out_temp_value[8];
//                     for(int i=0;i<4;i++)
//                     {
//                         out_temp_value[i*2] =  __half2float(out_temp[from_block_part_index+i*4+tidy][tidx*2]);
//                         out_temp_value[i*2+1] =  __half2float(out_temp[from_block_part_index+i*4+tidy][tidx*2 + 1]);
//                     }
//                     __syncthreads();

//                     float value_after_exp[8];
//                     for(int i=0;i<4;i++){
//                         float max_value_h = max_score[tidy+i*4 + from_block_part_index];
//                         float max_score_diff_h2 = max_score_diff[tidy+i*4];
//                         value_after_exp[i*2] = exp(value_h2[i*2]-max_value_h);
//                         value_after_exp[i*2 + 1] = exp(value_h2[i*2 + 1]-max_value_h);
//                         smem_temp_score[tidy+i*4][tidx*2] = __float2half( value_after_exp[i*2]);
//                         smem_temp_score[tidy+i*4][tidx*2 + 1] = __float2half( value_after_exp[i*2 + 1]);

//                         out_temp[from_block_part_index+i*4+tidy][tidx*2] = __float2half(out_temp_value[i*2] * max_score_diff_h2);
//                         out_temp[from_block_part_index+i*4+tidy][tidx*2 + 1] = __float2half(out_temp_value[i*2 + 1] * max_score_diff_h2);
//                     }

//                     for(int i=0;i<4;i++){
//                         float sum_temp = value_after_exp[i*2] + value_after_exp[i*2+1];
//                         temp_smem[tidy + i*4] = WarpReduce(temp_storage[tidy]).Sum(sum_temp);
//                     }

//                     __syncthreads();

//                     if(tidy == 0 && tidx < 16)
//                     {
//                         int idx = tidx+from_block_part_index;
//                         global_sum_scores[idx] *= max_score_diff[tidx];
//                         global_sum_scores[idx] += temp_smem[tidx];
//                     }

//                 }

//                 wmma::load_matrix_sync(frag_out,&out_temp[(tidy/2)*8 + from_block_part_index][(tidy%2)*32],64+pad,wmma::mem_row_major);
//                 for(int i=0;i<4;i++)
//                 {
//                     wmma::load_matrix_sync(frag_s[i], &smem_temp_score[(tidy/2)*8][i*16], 64+pad);
//                 }

//                 for(int i=0;i<4;i++){
//                     wmma::mma_sync(frag_out, frag_s[i], frag_v[i], frag_out);
//                 }

//                 wmma::store_matrix_sync(&out_temp[(tidy/2)*8 + from_block_part_index][(tidy%2)*32],frag_out,64+pad,wmma::mem_row_major);

                
//             }
//         }
//         __syncthreads();
//         for(int i=0;i<16;i++){
//             float sum_score_value = global_sum_scores[tidy*16+i];
//             float2 out_temp_value = __half22float2(HALF2(out_temp[tidy*16+i][tidx*2]));

//             out_temp[tidy*16+i][tidx*2] = __float2half(out_temp_value.x/sum_score_value);
//             out_temp[tidy*16+i][tidx*2 + 1] = __float2half(out_temp_value.y/sum_score_value);
//         }
//         __syncthreads();
//         for(int i=0;i<4;i++)
//             FLOAT4(out[data_offset_q+(i*16 + smem_index_i)*head_size+smem_index_j]) = FLOAT4(out_temp[i*16 + smem_index_i][smem_index_j]);

//     }
// }

//32*4
template <class DataType>
__global__ void sparse_attention_with_q_double_buffer(half *a,  half *b,  half *c, 
    half *out,const int *seq_len_info,const int *from_block_index, const int *from_block_index_position, const int *to_select_index,const int *to_select_index_position, const int batch_size,
    const int block_size,const int head_size,const int select_block_num){

    const int tidy = threadIdx.y;
    const int tidx = threadIdx.x;
    const int bidx = blockIdx.x;
    const int bidy = blockIdx.y;

    const int A_BM = 32;
    const int A_BK = 64;
    const int B_BK = 32;
    const int B_BN = 4;
    const int C_BK = 32;
    const int C_BN = 4;
    const int pad = 8;

    typedef hipcub::WarpReduce<float> WarpReduce;
    __shared__  typename WarpReduce::TempStorage temp_storage[4];
    __shared__  float global_sum_scores[64],pre_max_score[64],max_score[64];  
    __shared__  float temp_smem[16],max_score_diff[16];
    __shared__  half smem_q[2][16][64+pad],smem_k[64][64+pad],smem_v[64][64+pad],smem_temp_score[16][64+pad],out_temp[64][64+pad];


    wmma::fragment<wmma::matrix_a, 8, 32, 16, half, wmma::row_major> frag_q[4];
    wmma::fragment<wmma::matrix_b, 8, 32, 16, half, wmma::col_major> frag_k[4];
    wmma::fragment<wmma::accumulator, 8, 32, 16, half> frag_s_out;

    wmma::fragment<wmma::matrix_a, 8, 32, 16, half, wmma::row_major> frag_s[4];
    wmma::fragment<wmma::matrix_b, 8, 32, 16, half, wmma::row_major> frag_v[4];
    wmma::fragment<wmma::accumulator, 8, 32, 16, half> frag_out;


    float4 zero4 = {0.0f,0.0f,0.0f,0.0f};

    // 计算Q的起始位置
    const int compute_block_start = from_block_index_position[bidx];
    const int compute_block_end = from_block_index_position[bidx + 1];
    const int compute_block_num = compute_block_end - compute_block_start;
    for(int from_block_id_index = compute_block_start;from_block_id_index<compute_block_end;from_block_id_index++){
        int from_block_id = from_block_index[from_block_id_index];
        int seq_start_block_index = 0;
        int seq_block_len = 0;
        for(int i = 1;i<batch_size+1;i++){
            if(from_block_id >= seq_len_info[i]*12)
                continue;
            else{
                seq_start_block_index = seq_len_info[i-1];
                seq_block_len = seq_len_info[i] - seq_len_info[i-1];
                break;
            }
        }
        const int head_num = (from_block_id - 12 * seq_start_block_index)/seq_block_len;
        from_block_id = (from_block_id - 12 * seq_start_block_index)%seq_block_len;

        const int seq_start_index = 12 * seq_start_block_index * block_size * head_size + head_num * seq_block_len * block_size * head_size;
        const int data_offset_q = seq_start_index + from_block_id*block_size * head_size;

        const int to_block_start = to_select_index_position[from_block_id + seq_start_block_index];
        const int to_block_end = to_select_index_position[from_block_id + seq_start_block_index + 1];
        
        const int smem_index_i =  tidy*4+tidx/8;
        const int smem_index_j = (tidx%8)*8;
        

        FLOAT4(smem_q[0][smem_index_i][smem_index_j]) = FLOAT4(a[data_offset_q+ smem_index_i*head_size+smem_index_j]);

        int to_block_id = to_select_index[0];
        int data_k_start = seq_start_index + to_block_id * block_size * head_size;
        for(int i=0;i<4;i++){
            FLOAT4(smem_k[i*16 + smem_index_i][smem_index_j]) = FLOAT4(b[data_k_start+(i*16 + smem_index_i)*head_size+smem_index_j]);
            FLOAT4(smem_v[i*16 + smem_index_i][smem_index_j]) = FLOAT4(c[data_k_start+(i*16 + smem_index_i)*head_size+smem_index_j]);
        }

        for(int i=0;i<4;i++){
            FLOAT4(out_temp[i*16 + smem_index_i][smem_index_j]) = FLOAT4(zero4);
        }

        wmma::fill_fragment(frag_out, __float2half(0.0));
        if(tidy < 2){
            global_sum_scores[tidy*32+tidx] = 0.0;
            pre_max_score[tidy*32+tidx] = -1000.0f;
            max_score[tidy*32+tidx] = -1000.0f;
        }

        if(tidy == 0 && tidx < 16)
        {
            temp_smem[tidx] = 0.0;
            max_score_diff[tidx] = 1;
        }

        // 遍历K、V的每一个Block进行计算
        for(int block_id_index=to_block_start;block_id_index<to_block_end;block_id_index++){

            __syncthreads();

            for(int i=0;i<4;i++){
                wmma::load_matrix_sync(frag_k[i], &smem_k[(tidy%2)*32][i*16], 64+pad);
                wmma::load_matrix_sync(frag_v[i], &smem_v[i*16][(tidy%2)*32], 64+pad);
            }

            if(block_id_index != to_block_end - 1){
                to_block_id = to_select_index[block_id_index+1];
                data_k_start = seq_start_index + to_block_id * block_size * head_size;

                for(int i=0;i<4;i++){
                    FLOAT4(smem_k[i*16 + smem_index_i][smem_index_j]) = FLOAT4(b[data_k_start+(i*16 + smem_index_i)*head_size+smem_index_j]);
                    FLOAT4(smem_v[i*16 + smem_index_i][smem_index_j]) = FLOAT4(c[data_k_start+(i*16 + smem_index_i)*head_size+smem_index_j]);
                }
            }

            for(int from_block_part_index = 0;from_block_part_index<block_size;from_block_part_index+=16){
                wmma::fill_fragment(frag_s_out, __float2half(0.0));

                for(int i=0;i<4;i++)
                {
                    wmma::load_matrix_sync(frag_q[i], &smem_q[(from_block_part_index/16)&1][(tidy/2)*8][i*16], 64+pad);
                }
                
                for(int i=0;i<4;i++){
                    wmma::mma_sync(frag_s_out, frag_q[i], frag_k[i], frag_s_out);
                }

                if(block_id_index != to_block_end - 1 || from_block_part_index != 48)
                FLOAT4(smem_q[(from_block_part_index/16 + 1)&1][smem_index_i][smem_index_j]) = FLOAT4(a[data_offset_q+((from_block_part_index+16)%64+ smem_index_i)*head_size+smem_index_j]);

                //load next data

                wmma::store_matrix_sync(&smem_temp_score[(tidy/2)*8][(tidy%2)*32], frag_s_out, 64+pad, wmma::mem_row_major);
                
                __syncthreads();

                // 计算最大值 rowmax
                {
                    float value_h2[8];
                    float score_value[4];
                    for(int i=0;i<4;i++)
                    {
                        value_h2[i*2] = __half2float(smem_temp_score[tidy+i*4][tidx*2]);
                        value_h2[i*2+1] = __half2float(smem_temp_score[tidy+i*4][tidx*2+1]);
                    }

                    for(int i=0;i<4;i++)
                    {
                        score_value[i] = value_h2[i*2] > value_h2[i*2+1] ? value_h2[i*2] : value_h2[i*2+1];
                    }

                    for(int i=0;i<4;i++)
                    {
                        float t = float(score_value[i]);
                        temp_smem[tidy + i*4] = WarpReduce(temp_storage[tidy]).Reduce(t, hipcub::Max());
                    }

                    __syncthreads();
                    if(tidy == 0 && tidx < 16)
                    {
                        int idx = tidx+from_block_part_index;
                        pre_max_score[idx] = max_score[idx];
                        max_score[idx] = max_score[idx] > temp_smem[tidx]?max_score[idx]:temp_smem[tidx];
                        max_score_diff[tidx] = exp(pre_max_score[idx]-max_score[idx]);
                    }
                    
                    float out_temp_value[8];
                    for(int i=0;i<4;i++)
                    {
                        out_temp_value[i*2] =  __half2float(out_temp[from_block_part_index+i*4+tidy][tidx*2]);
                        out_temp_value[i*2+1] =  __half2float(out_temp[from_block_part_index+i*4+tidy][tidx*2 + 1]);
                    }
                    __syncthreads();

                    float value_after_exp[8];
                    for(int i=0;i<4;i++){
                        float max_value_h = max_score[tidy+i*4 + from_block_part_index];
                        float max_score_diff_h2 = max_score_diff[tidy+i*4];
                        value_after_exp[i*2] = exp(value_h2[i*2]-max_value_h);
                        value_after_exp[i*2 + 1] = exp(value_h2[i*2 + 1]-max_value_h);
                        smem_temp_score[tidy+i*4][tidx*2] = __float2half( value_after_exp[i*2]);
                        smem_temp_score[tidy+i*4][tidx*2 + 1] = __float2half( value_after_exp[i*2 + 1]);

                        
                        out_temp[from_block_part_index+i*4+tidy][tidx*2] = __float2half(out_temp_value[i*2] * max_score_diff_h2);
                        out_temp[from_block_part_index+i*4+tidy][tidx*2 + 1] = __float2half(out_temp_value[i*2 + 1] * max_score_diff_h2);
                    }

                    for(int i=0;i<4;i++){
                        float sum_temp = value_after_exp[i*2] + value_after_exp[i*2+1];
                        temp_smem[tidy + i*4] = WarpReduce(temp_storage[tidy]).Sum(sum_temp);
                    }

                    __syncthreads();

                    if(tidy == 0 && tidx < 16)
                    {
                        int idx = tidx+from_block_part_index;
                        global_sum_scores[idx] *= max_score_diff[tidx];
                        global_sum_scores[idx] += temp_smem[tidx];
                    }

                }

                wmma::load_matrix_sync(frag_out,&out_temp[(tidy/2)*8 + from_block_part_index][(tidy%2)*32],64+pad,wmma::mem_row_major);
                for(int i=0;i<4;i++)
                {
                    wmma::load_matrix_sync(frag_s[i], &smem_temp_score[(tidy/2)*8][i*16], 64+pad);
                }

                for(int i=0;i<4;i++){
                    wmma::mma_sync(frag_out, frag_s[i], frag_v[i], frag_out);
                }

                wmma::store_matrix_sync(&out_temp[(tidy/2)*8 + from_block_part_index][(tidy%2)*32],frag_out,64+pad,wmma::mem_row_major);

                
            }
        }
        __syncthreads();
        for(int i=0;i<16;i++){
            float sum_score_value = global_sum_scores[tidy*16+i];
            float2 out_temp_value = __half22float2(HALF2(out_temp[tidy*16+i][0])[tidx]);

            out_temp[tidy*16+i][tidx*2] = __float2half(out_temp_value.x/sum_score_value);
            out_temp[tidy*16+i][tidx*2 + 1] = __float2half(out_temp_value.y/sum_score_value);
        }
        __syncthreads();
        for(int i=0;i<4;i++)
            FLOAT4(out[data_offset_q+(i*16 + smem_index_i)*head_size+smem_index_j]) = FLOAT4(out_temp[i*16 + smem_index_i][smem_index_j]);

    }
}
      
void test_gemm_1(half *a, half *b,half *c, float *out,int *seq_len_info,int *from_select_index,int *from_select_index_position,int *to_select_index,int *to_select_index_position,int block_limit, int block_num, int head_num,int block_size,int head_size)
{


    // hipEvent_t start,stop;
    // hipEventCreate( &start );
    // hipEventCreate( &stop ) ;
    // hipEventRecord( start, 0 ) ;
    // 修改成最大线程块数量 80 * 2
    sparse_attention_without_bank<float><<<dim3(block_limit),dim3(32,8)>>>(a,b,c,out,seq_len_info,from_select_index,from_select_index_position,to_select_index,to_select_index_position,2,64,64,11);

    // hipEventRecord(stop,0);
    // float elapsedTime;
    // hipEventSynchronize(stop);
    // hipDeviceSynchronize();
    // hipEventElapsedTime(&elapsedTime, start, stop);
    // printf( "Time to generate:  %f ms\n", elapsedTime );
}
// template <class DataType>
// __global__ void sparse_attention_(half *a,  half *b,  half *c, 
//     half *out,const int *seq_len_info,const int *from_block_index, const int *from_block_index_position, const int *to_select_index,const int *to_select_index_position, const int batch_size,
//     const int block_size,const int head_size,const int select_block_num){

//     const int tidy = threadIdx.y;
//     const int tidx = threadIdx.x;
//     const int bidx = blockIdx.x;
//     const int bidy = blockIdx.y;

//     const int A_BM = 32;
//     const int A_BK = 64;
//     const int B_BK = 32;
//     const int B_BN = 4;
//     const int C_BK = 32;
//     const int C_BN = 4;
//     const int pad = 8;

//     typedef hipcub::WarpReduce<float> WarpReduce;
//     __shared__  typename WarpReduce::TempStorage temp_storage[4];
//     __shared__  float global_sum_scores[64],pre_max_score[64],max_score[64];  
//     __shared__  float temp_smem[16],max_score_diff[16];
//     __shared__  half smem_q[2][16][64+pad],smem_k[64][64+pad],smem_v[64][64+pad],smem_temp_score[16][64+pad],out_temp[64][64+pad];


//     wmma::fragment<wmma::matrix_a, 8, 32, 16, half, wmma::row_major> frag_q[4];
//     wmma::fragment<wmma::matrix_b, 8, 32, 16, half, wmma::col_major> frag_k[4];
//     wmma::fragment<wmma::accumulator, 8, 32, 16, half> frag_s_out;

//     wmma::fragment<wmma::matrix_a, 8, 32, 16, half, wmma::row_major> frag_s[4];
//     wmma::fragment<wmma::matrix_b, 8, 32, 16, half, wmma::row_major> frag_v[4];
//     wmma::fragment<wmma::accumulator, 8, 32, 16, half> frag_out;

//     const int smem_index_i =  tidy*4+tidx/8;
//     const int smem_index_j = (tidx%8)*8;

//     const unsigned long load_q_smem_addr[2] = {__cvta_generic_to_shared(smem_q[0]) + (smem_index_i*(64+pad)+smem_index_j)*2,__cvta_generic_to_shared(smem_q[1]) + (smem_index_i*(64+pad)+smem_index_j)*2};
//     const int load_k_smem_addr = __cvta_generic_to_shared(smem_k[0]) + (smem_index_i*(64+pad)+smem_index_j)*2;
//     const int load_v_smem_addr = __cvta_generic_to_shared(smem_v[0]) + (smem_index_i*(64+pad)+smem_index_j)*2;

    

//     float4 zero4 = {0.0f,0.0f,0.0f,0.0f};

//     // 计算Q的起始位置
//     const int compute_block_start = from_block_index_position[bidx];
//     const int compute_block_end = from_block_index_position[bidx + 1];
//     const int compute_block_num = compute_block_end - compute_block_start;
//     for(int from_block_id_index = compute_block_start;from_block_id_index<compute_block_end;from_block_id_index++){
//         int from_block_id = from_block_index[from_block_id_index];
//         int seq_start_block_index = 0;
//         int seq_block_len = 0;
//         for(int i = 1;i<batch_size+1;i++){
//             if(from_block_id >= seq_len_info[i]*12)
//                 continue;
//             else{
//                 seq_start_block_index = seq_len_info[i-1];
//                 seq_block_len = seq_len_info[i] - seq_len_info[i-1];
//                 break;
//             }
//         }
//         const int head_num = (from_block_id - 12 * seq_start_block_index)/seq_block_len;
//         from_block_id = (from_block_id - 12 * seq_start_block_index)%seq_block_len;

//         const int seq_start_index = 12 * seq_start_block_index * block_size * head_size + head_num * seq_block_len * block_size * head_size;
//         const int data_offset_q = seq_start_index + from_block_id*block_size * head_size;

//         const int to_block_start = to_select_index_position[from_block_id + seq_start_block_index];
//         const int to_block_end = to_select_index_position[from_block_id + seq_start_block_index + 1];
        
//         int load_q_smem_addr_now = load_q_smem_addr[0];

//         int load_q_gmem_addr = data_offset_q+ smem_index_i*head_size+smem_index_j;
//         asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :: "r"(load_q_smem_addr_now), "l"(&a[load_q_gmem_addr]));

//         int to_block_id = to_select_index[0];
//         int data_k_start = seq_start_index + to_block_id * block_size * head_size;
//         int load_k_gmem_addr = data_k_start+ smem_index_i*head_size+smem_index_j;
//         for(int i=0;i<64;i+=16){
//             asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :: "r"(load_k_smem_addr + i*(64+pad)*2), "l"(&b[load_k_gmem_addr+i*head_size]));
//             asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :: "r"(load_v_smem_addr + i*(64+pad)*2), "l"(&c[load_k_gmem_addr+i*head_size]));
//         }

//         for(int i=0;i<4;i++){
//             FLOAT4(out_temp[i*16 + smem_index_i][smem_index_j]) = FLOAT4(zero4);
            
//         }
//         wmma::fill_fragment(frag_out, __float2half(0.0));

//         if(tidy < 2){
//             global_sum_scores[tidy*32+tidx] = 0.0;
//             pre_max_score[tidy*32+tidx] = 0.0f;
//             max_score[tidy*32+tidx] = 0.0f;
//         }

//         if(tidy == 0 && tidx < 16)
//         {
//             temp_smem[tidx] = 0.0;
//             max_score_diff[tidx] = 1;
//         }

//         // 遍历K、V的每一个Block进行计算
//         for(int block_id_index=to_block_start;block_id_index<to_block_end;block_id_index++){

//             __syncthreads();
//             asm ("cp.async.commit_group;\n" ::);
//             asm ("cp.async.wait_group 0;\n" ::);

//             for(int i=0;i<4;i++){
//                 wmma::load_matrix_sync(frag_k[i], &smem_k[(tidy%2)*32][i*16], 64+pad);
//                 wmma::load_matrix_sync(frag_v[i], &smem_v[i*16][(tidy%2)*32], 64+pad);
//             }

//             if(block_id_index != to_block_end - 1){
//                 to_block_id = to_select_index[block_id_index+1];
//                 data_k_start = seq_start_index + to_block_id * block_size * head_size;
//                 load_k_gmem_addr = data_k_start+ smem_index_i*head_size+smem_index_j;
//                 for(int i=0;i<4;i++){
//                 asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :: "r"(load_k_smem_addr + i*(64+pad)*2), "l"(&b[load_k_gmem_addr+i*head_size]));
//                 asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :: "r"(load_v_smem_addr + i*(64+pad)*2), "l"(&c[load_k_gmem_addr+i*head_size]));
//                 }
//             }

//             for(int from_block_part_index = 0;from_block_part_index<block_size;from_block_part_index+=16){
//                 wmma::fill_fragment(frag_s_out, __float2half(0.0));

//                 for(int i=0;i<4;i++)
//                 {
//                     wmma::load_matrix_sync(frag_q[i], &smem_q[(from_block_part_index/16)&1][(tidy/2)*8][i*16], 64+pad);
//                 }
//                 for(int i=0;i<4;i++){
//                     wmma::mma_sync(frag_s_out, frag_q[i], frag_k[i], frag_s_out);
//                 }

//                 if(block_id_index != to_block_end - 1 || from_block_part_index != 48)
//                 {
//                     int load_q_smem_addr_now = load_q_smem_addr[(from_block_part_index/16 + 1)&1];
//                     asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :: "r"(load_q_smem_addr_now), "l"(&a[load_q_gmem_addr + ((from_block_part_index+16)%64)*head_size]));
//                 }
//                 //load next data

//                 wmma::store_matrix_sync(&smem_temp_score[(tidy/2)*8][(tidy%2)*32], frag_s_out, 64+pad, wmma::mem_row_major);
                
//                 __syncthreads();

//                 // 计算最大值 rowmax
//                 {
//                     float value_h2[8];
//                     float score_value[4];
//                     for(int i=0;i<4;i++)
//                     {
//                         value_h2[i*2] = __half2float(smem_temp_score[tidy+i*4][tidx*2]);
//                         value_h2[i*2+1] = __half2float(smem_temp_score[tidy+i*4][tidx*2+1]);
//                     }

//                     for(int i=0;i<4;i++)
//                     {
//                         score_value[i] = value_h2[i*2] > value_h2[i*2+1] ? value_h2[i*2] : value_h2[i*2+1];
//                     }

//                     for(int i=0;i<4;i++)
//                     {
//                         float t = float(score_value[i]);
//                         temp_smem[tidy + i*4] = WarpReduce(temp_storage[tidy]).Reduce(t, hipcub::Max());
//                     }

//                     float max_values_f[4];
                    
//                     for(int i=0;i<4;i++)
//                         max_values_f[4] = max_score[tidy+i*4 + from_block_part_index];

//                     __syncthreads();
//                     if(tidy == 0 && tidx < 16)
//                     {
//                         int idx = tidx+from_block_part_index;
//                         pre_max_score[idx] = max_score[idx];
//                         max_score[idx] = max_score[idx] > temp_smem[tidx]?max_score[idx]:temp_smem[tidx];
//                         max_score_diff[tidx] = exp(pre_max_score[idx]-max_score[idx]);
//                     }

//                     float out_temp_value[8];
//                     for(int i=0;i<4;i++)
//                     {
//                         out_temp_value[i*2] =  __half2float(out_temp[from_block_part_index+i*4+tidy][tidx*2]);
//                         out_temp_value[i*2+1] =  __half2float(out_temp[from_block_part_index+i*4+tidy][tidx*2 + 1]);
//                     }
                    
//                     __syncthreads();
//                     const int t = (tidy/2)*4+(tidx%4)*2;
//                     float value_after_exp[8];
//                     for(int i=0;i<4;i++){
//                         float max_value_h = max_score[tidy+i*4 + from_block_part_index];
//                         float max_score_diff_h2 = max_score_diff[tidy+i*4];
//                         value_after_exp[i*2] = exp(value_h2[i*2]-max_value_h);
//                         value_after_exp[i*2 + 1] = exp(value_h2[i*2 + 1]-max_value_h);
//                         smem_temp_score[tidy+i*4][tidx*2] = __float2half( value_after_exp[i*2]);
//                         smem_temp_score[tidy+i*4][tidx*2 + 1] = __float2half( value_after_exp[i*2 + 1]);

//                         out_temp[from_block_part_index+i*4+tidy][tidx*2] = __float2half(out_temp_value[i*2] * max_score_diff_h2);
//                         out_temp[from_block_part_index+i*4+tidy][tidx*2 + 1] = __float2half(out_temp_value[i*2 + 1] * max_score_diff_h2);

//                         // frag_out[from_block_part_index/16].x[i*2] *= max_score_diff[t];
//                         // frag_out[from_block_part_index/16].x[i*2+1] *= max_score_diff[t+1];
//                     }
//                     // for(int i=0;i<4;i++){
//                     //     float max_score_diff_h2 = max_score_diff[tidy+i*4];
    
//                     //     smem_temp_score[tidy+i*4][tidx*2] = __float2half( value_after_exp[i*2] * max_score_diff_h2);
//                     //     smem_temp_score[tidy+i*4][tidx*2 + 1] = __float2half( value_after_exp[i*2 + 1] * max_score_diff_h2);

                        
//                     //     out_temp[from_block_part_index+i*4+tidy][tidx*2] = out_temp_value[i*2] * max_score_diff_h2;
//                     //     out_temp[from_block_part_index+i*4+tidy][tidx*2 + 1] = out_temp_value[i*2 + 1] * max_score_diff_h2;
//                     // }

//                     for(int i=0;i<4;i++){
//                         float sum_temp = value_after_exp[i*2] + value_after_exp[i*2+1];
//                         temp_smem[tidy + i*4] = WarpReduce(temp_storage[tidy]).Sum(sum_temp);
//                     }

//                     __syncthreads();

//                     if(tidy == 0 && tidx < 16)
//                     {
//                         int idx = tidx+from_block_part_index;
//                         global_sum_scores[idx] *= max_score_diff[tidx];
//                         global_sum_scores[idx] += temp_smem[tidx];
//                     }

//                 }

//                 wmma::load_matrix_sync(frag_out,&out_temp[(tidy/2)*8 + from_block_part_index][(tidy%2)*32],64+pad,wmma::mem_row_major);
//                 for(int i=0;i<4;i++)
//                 {
//                     wmma::load_matrix_sync(frag_s[i], &smem_temp_score[(tidy/2)*8][i*16], 64+pad);
//                 }

//                 for(int i=0;i<4;i++){
//                     wmma::mma_sync(frag_out, frag_s[i], frag_v[i], frag_out);
//                 }

//                 wmma::store_matrix_sync(&out_temp[(tidy/2)*8 + from_block_part_index][(tidy%2)*32],frag_out,64+pad,wmma::mem_row_major);

//                 asm ("cp.async.commit_group;\n" ::);
//                 asm ("cp.async.wait_group 0;\n" ::);
//             }
//         }

//         __syncthreads();
//         for(int i=0;i<16;i++){
//             float sum_score_value = global_sum_scores[tidy*16+i];
//             float2 out_temp_value = __half22float2(HALF2(out_temp[tidy*16+i][0])[tidx]);

//             out_temp[tidy*16+i][tidx*2] = __float2half(out_temp_value.x/sum_score_value);
//             out_temp[tidy*16+i][tidx*2 + 1] = __float2half(out_temp_value.y/sum_score_value);
//         }
//         __syncthreads();
//         for(int i=0;i<4;i++)
//             FLOAT4(out[data_offset_q+(i*16 + smem_index_i)*head_size+smem_index_j]) = FLOAT4(out_temp[i*16 + smem_index_i][smem_index_j]);

//     }
// }

template <class DataType>
__global__ void sparse_attention_banlanced(half *a,  half *b,  half *c, 
    half *out,const int *seq_len_info,const int *from_block_index, const int *from_block_index_position, const int *to_select_index,const int *to_select_index_position, const int batch_size,
    const int block_size,const int head_size){
    
    const int tidy = threadIdx.y;
    const int tidx = threadIdx.x;
    const int bidx = blockIdx.x;
    const int bidy = blockIdx.y;

    const int A_BM = 32;
    const int A_BK = 64;
    const int B_BK = 32;
    const int B_BN = 4;
    const int C_BK = 32;
    const int C_BN = 4;
    const int pad = 8;

    typedef hipcub::WarpReduce<float> WarpReduce;
    __shared__  typename WarpReduce::TempStorage temp_storage[4];
    __shared__  float global_sum_scores[64],pre_max_score[64],max_score[64];  
    __shared__  float temp_smem[32],max_score_diff[16],sum_temp[64];
    __shared__  half smem_q[2][16][64+pad],smem_k[64][64+pad],smem_v[64][64+pad],smem_temp_score[16][64+pad],out_temp[64][64+pad];


    wmma::fragment<wmma::matrix_a, 8, 32, 16, half, wmma::row_major> frag_q[4];
    wmma::fragment<wmma::matrix_b, 8, 32, 16, half, wmma::col_major> frag_k[4];
    wmma::fragment<wmma::accumulator, 8, 32, 16, half> frag_s_out;

    wmma::fragment<wmma::matrix_a, 8, 32, 16, half, wmma::row_major> frag_s[4];
    wmma::fragment<wmma::matrix_b, 8, 32, 16, half, wmma::row_major> frag_v[4];
    wmma::fragment<wmma::accumulator, 8, 32, 16, half> frag_out;

    const int smem_index_i =  tidy*4+tidx/8;
    const int smem_index_j = (tidx%8)*8;

    const int load_k_smem_addr = __cvta_generic_to_shared(smem_k[0]) + (smem_index_i*(64+pad)+smem_index_j)*2;
    const int load_v_smem_addr = __cvta_generic_to_shared(smem_v[0]) + (smem_index_i*(64+pad)+smem_index_j)*2;
    const unsigned long load_q_smem_addr[2] = {__cvta_generic_to_shared(smem_q[0]) + (smem_index_i*(64+pad)+smem_index_j)*2,__cvta_generic_to_shared(smem_q[1]) + (smem_index_i*(64+pad)+smem_index_j)*2};

    float4 zero4 = {0.0f,0.0f,0.0f,0.0f};

    const int compute_block_start = from_block_index_position[bidx];
    const int compute_block_end = from_block_index_position[bidx + 1];
    const int compute_block_num = compute_block_end - compute_block_start;
    for(int from_block_id_index = compute_block_start;from_block_id_index<compute_block_end;from_block_id_index++){
        int from_block_id = from_block_index[from_block_id_index];
        int seq_start_block_index = 0;
        int seq_block_len = 0;
        for(int i=1;i<batch_size+1;i++){
            if(from_block_id >= seq_len_info[i]*12)
                continue;
            else{
                seq_start_block_index = seq_len_info[i-1]; //开始的block id
                seq_block_len = seq_len_info[i] - seq_len_info[i-1]; // seq 拥有的block id
                break;
            }
        }

        //还原原始的headnum和blockid
        const int head_num = (from_block_id - 12*seq_start_block_index)/seq_block_len;
        from_block_id = (from_block_id - 12*seq_start_block_index)%seq_block_len;

        const int seq_start_index = 12*seq_start_block_index*block_size*head_size + head_num*seq_block_len*block_size*head_size;
        const int data_offset_q = seq_start_index + from_block_id*block_size*head_size;

        const int to_block_start = to_select_index_position[from_block_id+seq_start_block_index];
        const int to_block_end = to_select_index_position[from_block_id+seq_start_block_index + 1];

        int load_q_smem_addr_now = load_q_smem_addr[0];

        int load_q_gmem_addr = data_offset_q+ smem_index_i*head_size+smem_index_j;
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :: "r"(load_q_smem_addr_now), "l"(&a[load_q_gmem_addr]));

        
        for(int i=0;i<4;i++){
            FLOAT4(out_temp[i*16 + smem_index_i][smem_index_j]) = FLOAT4(zero4);
        }

        wmma::fill_fragment(frag_out, __float2half(0.0));

        int to_block_id = to_select_index[0];
        int data_k_start = seq_start_index + to_block_id * block_size * head_size;
        int load_k_gmem_addr = data_k_start+ smem_index_i*head_size+smem_index_j;
        for(int i=0;i<64;i+=16){
            asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :: "r"(load_k_smem_addr + i*(64+pad)*2), "l"(&b[load_k_gmem_addr+i*head_size]));
            asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :: "r"(load_v_smem_addr + i*(64+pad)*2), "l"(&c[load_k_gmem_addr+i*head_size]));
        }

        if(tidy == 0 && tidx < 16)
        {
            max_score_diff[tidx] = 1.0f;
        }

        asm ("cp.async.commit_group;\n" ::);
        asm ("cp.async.wait_group 0;\n" ::);


        for(int block_id_index=to_block_start;block_id_index<to_block_end;block_id_index++){
            #pragma unroll
            for(int i=0;i<4;i++){
                wmma::load_matrix_sync(frag_k[i], &smem_k[(tidy%2)*32][i*16], 64+pad);
                wmma::load_matrix_sync(frag_v[i], &smem_v[i*16][(tidy%2)*32], 64+pad);
            }

            __syncthreads();
            if(block_id_index != to_block_end - 1){
                to_block_id = to_select_index[block_id_index+1];
                data_k_start = seq_start_index + to_block_id * block_size * head_size;
                
                int load_k_gmem_addr = data_k_start+ smem_index_i*head_size+smem_index_j;
                for(int i=0;i<64;i+=16){
                    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :: "r"(load_k_smem_addr + i*(64+pad)*2), "l"(&b[load_k_gmem_addr+i*head_size]));
                    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :: "r"(load_v_smem_addr + i*(64+pad)*2), "l"(&c[load_k_gmem_addr+i*head_size]));
                }
            }


            for(int from_block_part_index = 0;from_block_part_index<block_size;from_block_part_index+=16){
                
                // FLOAT4(smem_q[smem_index_i][smem_index_j]) = FLOAT4(a[data_offset_q+ smem_index_i*head_size+smem_index_j + from_block_part_index*head_size]);

                #pragma unroll
                for(int i=0;i<4;i++)
                {
                    wmma::load_matrix_sync(frag_q[i], &smem_q[(from_block_part_index/16)&1][(tidy/2)*8][i*16], 64+pad);
                }

                if(block_id_index != to_block_end - 1 || from_block_part_index != 48)
                {
                    int load_q_smem_addr_now = load_q_smem_addr[(from_block_part_index/16 + 1)&1];
                    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :: "r"(load_q_smem_addr_now), "l"(&a[load_q_gmem_addr + ((from_block_part_index+16)%64)*head_size]));
                }
                wmma::fill_fragment(frag_s_out, __float2half(0.0));

                for(int i=0;i<4;i++){
                    wmma::mma_sync(frag_s_out, frag_q[i], frag_k[i], frag_s_out);
                }

                // if(tidy == from_block_part_index/16)
                wmma::store_matrix_sync(&smem_temp_score[(tidy/2)*8][(tidy%2)*32], frag_s_out, 64+pad, wmma::mem_row_major);

                __syncthreads();

                // 计算最大值 rowmax
                float value_h2[8];
                {
                    float score_value[4];
                    #pragma unroll
                    for(int i=0;i<4;i++)
                    {
                        value_h2[i*2] = __half2float(smem_temp_score[tidy+i*4][tidx*2]);
                        value_h2[i*2+1] = __half2float(smem_temp_score[tidy+i*4][tidx*2+1]);
                    }

                    #pragma unroll
                    for(int i=0;i<4;i++)
                    {
                        score_value[i] = value_h2[i*2] > value_h2[i*2+1] ? value_h2[i*2] : value_h2[i*2+1];
                    }

                    #pragma unroll
                    for(int i=0;i<4;i++)
                    {
                        temp_smem[tidy + i*4] = WarpReduce(temp_storage[tidy]).Reduce(score_value[i], hipcub::Max());
                    }
                }
                __syncthreads();
                if(tidy == 1 && tidx < 16)
                {
                    int idx = tidx+from_block_part_index;
                    pre_max_score[idx] = max_score[idx];
                    max_score[idx] = max(max_score[idx],temp_smem[tidx]);
                    max_score_diff[tidx] = exp(pre_max_score[idx]-max_score[idx]);
                }
                __syncthreads();

                half2 max_s[4];
                half2 value[4];
                for(int i=0;i<4;i++){
                    value[i] =  HALF2(smem_temp_score[tidy*4+i][0])[tidx];
                    max_s[i] = __half2half2(__float2half(max_score[tidy*4 + i + from_block_part_index]));
                }
                half2 diff_x = __half2half2(__float2half(max_score_diff[tidy*4 + tidx/8 ]));
                float4 out_t = FLOAT4(out_temp[from_block_part_index + tidy*4+(tidx*8)/64][(tidx*8)%64]);


                for(int i=0;i<4;i++){
                    half2 t = h2exp(__hsub2(value[i],max_s[i]));
                    HALF2(out_temp[from_block_part_index + tidy*4+(tidx*8)/64][(tidx*8)%64])[i] = __hmul2(HALF2(out_t)[i],diff_x);

                    HALF2(smem_temp_score[tidy*4+i][0])[tidx] = t;
                    float v_ = __half2float(__hadd(HALF(t)[0],HALF(t)[1]));
                    temp_smem[tidy*4 + i] = WarpReduce(temp_storage[tidy]).Sum(v_);
                }

                __syncthreads();

                wmma::load_matrix_sync(frag_out,&out_temp[(tidy/2)*8 + from_block_part_index][(tidy%2)*32],64+pad,wmma::mem_row_major);

                for(int i=0;i<4;i++)
                {
                    wmma::load_matrix_sync(frag_s[i], &smem_temp_score[(tidy/2)*8][i*16], 64+pad);
                }


                for(int i=0;i<4;i++){
                    wmma::mma_sync(frag_out, frag_s[i], frag_v[i], frag_out);
                }

                wmma::store_matrix_sync(&out_temp[(tidy/2)*8 + from_block_part_index][(tidy%2)*32],frag_out,64+pad,wmma::mem_row_major);

                if(tidy == 0 && tidx < 16)
                {
                    int idx = tidx+from_block_part_index;
                    global_sum_scores[idx] *= max_score_diff[tidx];
                    global_sum_scores[idx] += temp_smem[tidx];
                }

                asm ("cp.async.commit_group;\n" ::);
                asm ("cp.async.wait_group 0;\n" ::);
            }
        }

        // if(tidx == 0 && tidy == 0 && from_block_id == 0 && head_num == 0){
        //     for(int i=0;i<64;i++)
        //         printf("%f ",global_sum_scores[i]);
        //     printf("\n");
        // }


        __syncthreads();
        for(int i=0;i<16;i++){
            float sum_score_value = global_sum_scores[tidy*16+i];
            float2 out_temp_value = __half22float2(HALF2(out_temp[tidy*16+i][0])[tidx]);

            out_temp[tidy*16+i][tidx*2] = __float2half(out_temp_value.x/sum_score_value);
            out_temp[tidy*16+i][tidx*2 + 1] = __float2half(out_temp_value.y/sum_score_value);

        }

        __syncthreads();
        for(int i=0;i<4;i++)
            FLOAT4(out[data_offset_q+(i*16 + smem_index_i)*head_size+smem_index_j]) = FLOAT4(out_temp[i*16 + smem_index_i][smem_index_j]);
    }
}  


template <class DataType>
__global__ void sparse_attention_non_balanced(half *a,  half *b,  half *c, 
    half *out,const int *seq_len_info,const int *from_block_index, const int *from_block_index_position, const int *to_select_index,const int *to_select_index_position, const int batch_size,
    const int block_size,const int head_size){

    const int tidy = threadIdx.y;
    const int tidx = threadIdx.x;
    const int head_num = blockIdx.x;
    int from_block_id = blockIdx.y;
    const int g_dimy = gridDim.y;

    const int A_BM = 32;
    const int A_BK = 64;
    const int B_BK = 32;
    const int B_BN = 4;
    const int C_BK = 32;
    const int C_BN = 4;
    const int pad = 8;

    typedef hipcub::WarpReduce<float> WarpReduce;
    __shared__  typename WarpReduce::TempStorage temp_storage[4];
    __shared__  float global_sum_scores[64],pre_max_score[64],max_score[64];  
    __shared__  float temp_smem[32],max_score_diff[16],sum_temp[64];
    __shared__  half smem_q[2][16][64+pad],smem_k[64][64+pad],smem_v[64][64+pad],smem_temp_score[16][64+pad],out_temp[64][64+pad];


    wmma::fragment<wmma::matrix_a, 8, 32, 16, half, wmma::row_major> frag_q[4];
    wmma::fragment<wmma::matrix_b, 8, 32, 16, half, wmma::col_major> frag_k[4];
    wmma::fragment<wmma::accumulator, 8, 32, 16, half> frag_s_out;

    wmma::fragment<wmma::matrix_a, 8, 32, 16, half, wmma::row_major> frag_s[4];
    wmma::fragment<wmma::matrix_b, 8, 32, 16, half, wmma::row_major> frag_v[4];
    wmma::fragment<wmma::accumulator, 8, 32, 16, half> frag_out;

    const int smem_index_i =  tidy*4+tidx/8;
    const int smem_index_j = (tidx%8)*8;

    const unsigned long load_q_smem_addr[2] = {__cvta_generic_to_shared(smem_q[0]) + (smem_index_i*(64+pad)+smem_index_j)*2,__cvta_generic_to_shared(smem_q[1]) + (smem_index_i*(64+pad)+smem_index_j)*2};
    const int load_k_smem_addr = __cvta_generic_to_shared(smem_k[0]) + (smem_index_i*(64+pad)+smem_index_j)*2;
    const int load_v_smem_addr = __cvta_generic_to_shared(smem_v[0]) + (smem_index_i*(64+pad)+smem_index_j)*2;

    

    float4 zero4 = {0.0f,0.0f,0.0f,0.0f};

    // 计算Q的起始位置

        int seq_start_block_index = 0;
        int seq_block_len = 0;
        for(int i = 1;i<batch_size+1;i++){
            if(from_block_id >= seq_len_info[i])
                continue;
            else{
                seq_start_block_index = seq_len_info[i-1];
                seq_block_len = seq_len_info[i] - seq_len_info[i-1];
                break;
            }
        }

        from_block_id -= seq_start_block_index;

        const int seq_start_index = 12*seq_start_block_index*block_size*head_size + head_num*seq_block_len*block_size*head_size;
        const int data_offset_q = seq_start_index + from_block_id*block_size*head_size;

        const int to_block_start = to_select_index_position[from_block_id + seq_start_block_index];
        const int to_block_end = to_select_index_position[from_block_id + seq_start_block_index + 1];
        
        int load_q_smem_addr_now = load_q_smem_addr[0];

        int load_q_gmem_addr = data_offset_q+ smem_index_i*head_size+smem_index_j;
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :: "r"(load_q_smem_addr_now), "l"(&a[load_q_gmem_addr]));

        for(int i=0;i<4;i++){
            FLOAT4(out_temp[i*16 + smem_index_i][smem_index_j]) = FLOAT4(zero4);
        }
        wmma::fill_fragment(frag_out, __float2half(0.0));

        int to_block_id = to_select_index[0];
        int data_k_start = seq_start_index + to_block_id * block_size * head_size;
        int load_k_gmem_addr = data_k_start+ smem_index_i*head_size+smem_index_j;
        for(int i=0;i<64;i+=16){
            asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :: "r"(load_k_smem_addr + i*(64+pad)*2), "l"(&b[load_k_gmem_addr+i*head_size]));
            asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :: "r"(load_v_smem_addr + i*(64+pad)*2), "l"(&c[load_k_gmem_addr+i*head_size]));
        }

        if(tidy < 2){
            global_sum_scores[tidy*32+tidx] = 0.0;
            pre_max_score[tidy*32+tidx] = 0.0f;
            max_score[tidy*32+tidx] = 0.0f;
        }

        if(tidy == 0 && tidx < 16)
        {
            temp_smem[tidx] = 0.0;
            max_score_diff[tidx] = 1.0f;
        }

        asm ("cp.async.commit_group;\n" ::);
        asm ("cp.async.wait_group 0;\n" ::);

        // 遍历K、V的每一个Block进行计算
        for(int block_id_index=to_block_start;block_id_index<to_block_end;block_id_index++){
            __syncthreads();

            #pragma unroll
            for(int i=0;i<4;i++){
                wmma::load_matrix_sync(frag_k[i], &smem_k[(tidy%2)*32][i*16], 64+pad);
                wmma::load_matrix_sync(frag_v[i], &smem_v[i*16][(tidy%2)*32], 64+pad);
            }

            if(block_id_index != to_block_end - 1){
                to_block_id = to_select_index[block_id_index];
                data_k_start = seq_start_index + to_block_id * block_size * head_size;
                
                int load_k_gmem_addr = data_k_start+ smem_index_i*head_size+smem_index_j;
                for(int i=0;i<64;i+=16){
                    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :: "r"(load_k_smem_addr + i*(64+pad)*2), "l"(&b[load_k_gmem_addr+i*head_size]));
                    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :: "r"(load_v_smem_addr + i*(64+pad)*2), "l"(&c[load_k_gmem_addr+i*head_size]));
                }
            }

            for(int from_block_part_index = 0;from_block_part_index<block_size;from_block_part_index+=16){

                #pragma unroll
                for(int i=0;i<4;i++)
                {
                    wmma::load_matrix_sync(frag_q[i], &smem_q[(from_block_part_index/16)&1][(tidy/2)*8][i*16], 64+pad);
                }

                if(block_id_index != to_block_end - 1 || from_block_part_index != 48)
                {
                    int load_q_smem_addr_now = load_q_smem_addr[(from_block_part_index/16 + 1)&1];
                    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :: "r"(load_q_smem_addr_now), "l"(&a[load_q_gmem_addr + ((from_block_part_index+16)%64)*head_size]));
                }
                wmma::fill_fragment(frag_s_out, __float2half(0.0));

                for(int i=0;i<4;i++){
                    wmma::mma_sync(frag_s_out, frag_q[i], frag_k[i], frag_s_out);
                }
                //load next data

                wmma::store_matrix_sync(&smem_temp_score[(tidy/2)*8][(tidy%2)*32], frag_s_out, 64+pad, wmma::mem_row_major);
                
                __syncthreads();

                // 计算最大值 rowmax
                // float value_h2[8];
                // {
                //     float score_value[4];
                //     #pragma unroll
                //     for(int i=0;i<4;i++)
                //     {
                //         value_h2[i*2] = __half2float(smem_temp_score[tidy+i*4][tidx*2]);
                //         value_h2[i*2+1] = __half2float(smem_temp_score[tidy+i*4][tidx*2+1]);
                //     }

                //     #pragma unroll
                //     for(int i=0;i<4;i++)
                //     {
                //         score_value[i] = value_h2[i*2] > value_h2[i*2+1] ? value_h2[i*2] : value_h2[i*2+1];
                //     }

                //     #pragma unroll
                //     for(int i=0;i<4;i++)
                //     {
                //         temp_smem[tidy + i*4] = WarpReduce(temp_storage[tidy]).Reduce(score_value[i], hipcub::Max());
                //     }
                // }

                // __syncthreads();
                // if(tidy == 1 && tidx < 16)
                // {
                //     int idx = tidx+from_block_part_index;
                //     pre_max_score[idx] = max_score[idx];
                //     max_score[idx] = max(max_score[idx],temp_smem[tidx]);
                //     max_score_diff[tidx] = exp(pre_max_score[idx]-max_score[idx]);
                // }
                // __syncthreads();

                // half2 max_s[4];
                // half2 value[4];
                // for(int i=0;i<4;i++){
                //     value[i] =  HALF2(smem_temp_score[tidy*4+i][0])[tidx];
                //     max_s[i] = __half2half2(__float2half(max_score[tidy*4 + i + from_block_part_index]));
                // }
                // half2 diff_x = __half2half2(__float2half(max_score_diff[tidy*4 + tidx/8 ]));
                // float4 out_t = FLOAT4(out_temp[from_block_part_index + tidy*4+(tidx*8)/64][(tidx*8)%64]);


                // for(int i=0;i<4;i++){
                //     half2 t = h2exp(__hsub2(value[i],max_s[i]));
                //     HALF2(out_temp[from_block_part_index + tidy*4+(tidx*8)/64][(tidx*8)%64])[i] = __hmul2(HALF2(out_t)[i],diff_x);

                //     HALF2(smem_temp_score[tidy*4+i][0])[tidx] = t;
                //     float v_ = __half2float(__hadd(HALF(t)[0],HALF(t)[1]));
                //     temp_smem[tidy*4 + i] = WarpReduce(temp_storage[tidy]).Sum(v_);
                // }
                // __syncthreads();
                float max_value[4];
                half2 value_h2[4];
                float max_temp[4];
                half2 diff_x[4];
                half2 out_t[4];
                {
                    float score_value[4];
                    #pragma unroll
                    for(int i=0;i<4;i++)
                    {
                        max_value[i] = max_score[tidy*4+i+from_block_part_index];
                        value_h2[i] = HALF2(smem_temp_score[tidy*4+i][0])[tidx];
                    }
                    // tile_block.sync();
                    // filled.arrive();

                    #pragma unroll
                    for(int i=0;i<4;i++)
                    {
                        score_value[i] = __half2float(__hmax(value_h2[i].x,value_h2[i].y));
                    }

                    #pragma unroll
                    for(int i=0;i<4;i++)
                    {
                        max_temp[i] = WarpReduce(temp_storage[tidy]).Reduce(score_value[i], hipcub::Max());
                    }
    
                    #pragma unroll
                    for(int i=0;i<4;i++){
                        max_temp[i] = __shfl_sync(0xffffffff, max_temp[i], 0); 
                        out_t[i] = HALF2(out_temp[tidy*4+i+from_block_part_index][0])[tidx];
                        max_temp[i] = max(max_temp[i],max_value[i]);
                        diff_x[i] = __half2half2(__float2half(exp(max_temp[i]-max_value[i])));
                        max_score[tidy*4+i+from_block_part_index] = max_temp[i];
                    }

                    for(int i=0;i<4;i++){
                        half2 t = h2exp(__hsub2(value_h2[i],__half2half2(__float2half(max_temp[i]))));
                        HALF2(out_temp[tidy*4+i+from_block_part_index][0])[tidx] = __hmul2(out_t[i],diff_x[i]);
                        HALF2(smem_temp_score[tidy*4+i][0])[tidx] = t;
                        float v_ = __half2float(__hadd(HALF(t)[0],HALF(t)[1]));
                        temp_smem[tidy*4 + i] = WarpReduce(temp_storage[tidy]).Sum(v_);
                    }
                }

                __syncthreads();

                if(tidy == 0 && tidx < 16)
                {
                    int idx = tidx+from_block_part_index;
                    global_sum_scores[idx] *= max_score_diff[tidx];
                    global_sum_scores[idx] += temp_smem[tidx];
                }


                wmma::load_matrix_sync(frag_out,&out_temp[(tidy/2)*8 + from_block_part_index][(tidy%2)*32],64+pad,wmma::mem_row_major);

                for(int i=0;i<4;i++)
                {
                    wmma::load_matrix_sync(frag_s[i], &smem_temp_score[(tidy/2)*8][i*16], 64+pad);
                }


                for(int i=0;i<4;i++){
                    wmma::mma_sync(frag_out, frag_s[i], frag_v[i], frag_out);
                }
                wmma::store_matrix_sync(&out_temp[(tidy/2)*8 + from_block_part_index][(tidy%2)*32],frag_out,64+pad,wmma::mem_row_major);

                if(tidy == 0 && tidx < 16)
                {
                    int idx = tidx+from_block_part_index;
                    global_sum_scores[idx] *= max_score_diff[tidx];
                    global_sum_scores[idx] += temp_smem[tidx];
                }

                asm ("cp.async.commit_group;\n" ::);
                asm ("cp.async.wait_group 0;\n" ::);
            }
        }

        __syncthreads();
        for(int i=0;i<16;i++){
            float sum_score_value = global_sum_scores[tidy*16+i];
            float2 out_temp_value = __half22float2(HALF2(out_temp[tidy*16+i][0])[tidx]);

            out_temp[tidy*16+i][tidx*2] = __float2half(out_temp_value.x/sum_score_value);
            out_temp[tidy*16+i][tidx*2 + 1] = __float2half(out_temp_value.y/sum_score_value);

        }
        __syncthreads();
        for(int i=0;i<4;i++)
            FLOAT4(out[data_offset_q+(i*16 + smem_index_i)*head_size+smem_index_j]) = FLOAT4(out_temp[i*16 + smem_index_i][smem_index_j]);

}


// using barrier = cuda::barrier<cuda::thread_scope_block>;

// // __device__ stage_1(barrier ready[],barrier filled[]){

// // }

// #define _CG_ABI_EXPERIMENTAL

// __device__ void stage1(cooperative_groups::__v1::thread_block_tile<128U, cooperative_groups::__v1::thread_block>& tile_block,
// barrier &ready, barrier &filled, half *a,  half *b, half smem_temp_score[][64+8], const int& compute_block_start, const int& compute_block_end,
// const int *from_block_index, const int& batch_size, const int *seq_len_info, const int& block_size,
// const int& head_size, const int *to_select_index_position, const int *to_select_index, half* test_out){
//     const int tidy = tile_block.thread_rank()/32;
//     const int tidx = threadIdx.x;
//     const int pad = 8;
//     barrier::arrival_token token;

//     __shared__  half smem_q[2][16][64+pad],smem_k[64][64+pad];


//     const int smem_index_i =  tidy*4+tidx/8;
//     const int smem_index_j = (tidx%8)*8;

//     const int load_k_smem_addr = __cvta_generic_to_shared(smem_k[0]) + (smem_index_i*(64+pad)+smem_index_j)*2;

//     wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> frag_q[4];
//     wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::col_major> frag_k[4];
//     wmma::fragment<wmma::accumulator, 16, 16, 16, half> frag_s_out;

//     const unsigned long load_q_smem_addr[2] = {__cvta_generic_to_shared(smem_q[0]) + (smem_index_i*(64+pad)+smem_index_j)*2,__cvta_generic_to_shared(smem_q[1]) + (smem_index_i*(64+pad)+smem_index_j)*2};
    
//     for(int from_block_id_index = compute_block_start;from_block_id_index<compute_block_end;from_block_id_index++){
//         int from_block_id = from_block_index[from_block_id_index];
//         int seq_start_block_index = 0;
//         int seq_block_len = 0;

//         for(int i=1;i<batch_size+1;i++){
//             if(from_block_id >= seq_len_info[i]*12)
//                 continue;
//             else{
//                 seq_start_block_index = seq_len_info[i-1]; //开始的block id
//                 seq_block_len = seq_len_info[i] - seq_len_info[i-1]; // seq 拥有的block id
//                 break;
//             }
//         }

//         const int head_num = (from_block_id - 12*seq_start_block_index)/seq_block_len;
//         from_block_id = (from_block_id - 12*seq_start_block_index)%seq_block_len;

//         const int seq_start_index = 12*seq_start_block_index*block_size*head_size + head_num*seq_block_len*block_size*head_size;
//         const int data_offset_q = seq_start_index + from_block_id*block_size*head_size;

//         const int to_block_start = to_select_index_position[from_block_id+seq_start_block_index];
//         const int to_block_end = to_select_index_position[from_block_id+seq_start_block_index + 1];

//         int load_q_smem_addr_now = load_q_smem_addr[0];
       
//         int load_q_gmem_addr = data_offset_q+ smem_index_i*head_size+smem_index_j;
//         asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :: "r"(load_q_smem_addr_now), "l"(&a[load_q_gmem_addr]));

//         int to_block_id = to_select_index[to_block_start];
//         int data_k_start = seq_start_index + to_block_id * block_size * head_size;
//         int load_k_gmem_addr = data_k_start+ smem_index_i*head_size+smem_index_j;
//         for(int i=0;i<64;i+=16){
//             asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :: "r"(load_k_smem_addr + i*(64+pad)*2), "l"(&b[load_k_gmem_addr+i*head_size]));
//         }

//         asm ("cp.async.commit_group;\n" ::);
//         asm ("cp.async.wait_group 0;\n" ::);

//         // if(tile_block.thread_rank() == 0)
//         // {
//         //     printf("%d %d %d",to_block_id,data_k_start,load_k_gmem_addr);
//         //     for(int i=0;i<64;i++){
//         //         for(int j=0;j<6;j++)
//         //             printf("%f ", __half2float(smem_k[i][j]));
//         //         printf("\n");
//         //     }
//         //     // printf("%d\n",tidy*128*4+from_block_part_index*128+(tidx/8)*128+(tidx%8)*8+64*block_id_index);
//         // }
//         // tile_block.sync();

//         for(int block_id_index=to_block_start;block_id_index<to_block_end;block_id_index++){
//             #pragma unroll
//             for(int i=0;i<4;i++){
//                 wmma::load_matrix_sync(frag_k[i], &smem_k[tidy*16][i*16], 64+pad);
//             }

//             tile_block.sync();

//             if(block_id_index != to_block_end - 1){
//                 to_block_id = to_select_index[block_id_index+1];

//                 // if(tile_block.thread_rank() == 0)
//                 // {
//                 //     printf("%d %d %d %d %d\n",to_block_id,block_id_index,to_block_end - 1,to_select_index[0],to_select_index[1]);
//                 // }

//                 data_k_start = seq_start_index + to_block_id * block_size * head_size;
                
//                 int load_k_gmem_addr = data_k_start+ smem_index_i*head_size+smem_index_j;
//                 for(int i=0;i<64;i+=16){
//                     asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :: "r"(load_k_smem_addr + i*(64+pad)*2), "l"(&b[load_k_gmem_addr+i*head_size]));
//                 }
//             }

//             for(int from_block_part_index = 0;from_block_part_index<block_size;from_block_part_index+=16){
                
//                 #pragma unroll
//                 for(int i=0;i<4;i++)
//                 {
//                     wmma::load_matrix_sync(frag_q[i], &smem_q[(from_block_part_index/16)&1][0][i*16], 64+pad);
//                 }

//                 if(block_id_index != to_block_end - 1 || from_block_part_index != 48)
//                 {
//                     int load_q_smem_addr_now = load_q_smem_addr[(from_block_part_index/16 + 1)&1];
//                     asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :: "r"(load_q_smem_addr_now), "l"(&a[load_q_gmem_addr + ((from_block_part_index+16)%64)*head_size]));
//                 }

//                 wmma::fill_fragment(frag_s_out, __float2half(0.0));

//                 for(int i=0;i<4;i++){
//                     wmma::mma_sync(frag_s_out, frag_q[i], frag_k[i], frag_s_out);
//                 }

//                 // get single

//                 ready.arrive_and_wait();

//                 wmma::store_matrix_sync(&smem_temp_score[0][tidy*16], frag_s_out, 64+pad, wmma::mem_row_major);

//                 // 同步 
//                 // send single
//                 token = filled.arrive();

//                 // FLOAT4(test_out[tidy*128*4+from_block_part_index*128+(tidx/8)*128+(tidx%8)*8+64*block_id_index]) = FLOAT4(smem_temp_score[tidy*4+tidx/8][(tidx%8)*8]);
                

//                 asm ("cp.async.commit_group;\n" ::);
//                 asm ("cp.async.wait_group 0;\n" ::);  
//             }
//         }
//     }
// }
// __device__ void stage_2(cooperative_groups::__v1::thread_block_tile<128U, cooperative_groups::__v1::thread_block>& tile_block,
// barrier& ready, barrier& filled, const int& compute_block_start, const int& compute_block_end, const int *to_select_index_position,
// const int *from_block_index, const int& batch_size, const int *seq_len_info, const int& block_size, half smem_temp_score[][64+8], 
// float *temp_smem, float* pre_max_score, float* max_score, float* max_score_diff){

//     const int tidy = tile_block.thread_rank()/32;
//     const int tidx = threadIdx.x;
//     barrier::arrival_token token;

//     typedef hipcub::WarpReduce<float> WarpReduce;
//     __shared__  typename WarpReduce::TempStorage temp_storage[2];



//     for(int from_block_id_index = compute_block_start;from_block_id_index<compute_block_end;from_block_id_index++){
        
//         int from_block_id = from_block_index[from_block_id_index];
//         int seq_start_block_index = 0;
//         int seq_block_len = 0;
//         for(int i=1;i<batch_size+1;i++){
//             if(from_block_id >= seq_len_info[i]*12)
//                 continue;
//             else{
//                 seq_start_block_index = seq_len_info[i-1]; //开始的block id
//                 seq_block_len = seq_len_info[i] - seq_len_info[i-1]; // seq 拥有的block id
//                 break;
//             }
//         }

//         if(tidy < 2){
//             pre_max_score[tidy*32+tidx] = 0.0f;
//             max_score[tidy*32+tidx] = 0.0f;
//         }

//         //还原原始的headnum和blockid
//         const int head_num = (from_block_id - 12*seq_start_block_index)/seq_block_len;
//         from_block_id = (from_block_id - 12*seq_start_block_index)%seq_block_len;

//         const int to_block_start = to_select_index_position[from_block_id+seq_start_block_index];
//         const int to_block_end = to_select_index_position[from_block_id+seq_start_block_index + 1];

//         for(int block_id_index=to_block_start;block_id_index<to_block_end;block_id_index++){
            
//             for(int from_block_part_index = 0;from_block_part_index<block_size;from_block_part_index+=16){
                
//                 // 计算最大值 rowmax

//                 ready.arrive_and_wait();

//                 // if(tile_block.thread_rank()<32){
//                 //     float value_h2[4];
//                 //     float score_value[2];

//                 //     for(int i=0;i<8;i++){
//                 //         value_h2[0] = __half2float(smem_temp_score[i*2][tidx]);
//                 //         value_h2[1] = __half2float(smem_temp_score[i*2][tidx+32]);
//                 //         value_h2[2] = __half2float(smem_temp_score[i*2+1][tidx]);
//                 //         value_h2[3] = __half2float(smem_temp_score[i*2+1][tidx+32]);
//                 //         score_value[0] = value_h2[0] > value_h2[1] ? value_h2[0] : value_h2[1];
//                 //         score_value[1] = value_h2[2] > value_h2[3] ? value_h2[2] : value_h2[3];
//                 //         temp_smem[i*2] = WarpReduce(temp_storage[0]).Reduce(score_value[0], hipcub::Max());
//                 //         temp_smem[i*2+1] = WarpReduce(temp_storage[1]).Reduce(score_value[1], hipcub::Max());
//                 //     }

//                 //     if(tidx < 16){
//                 //         int idx = tidx+from_block_part_index;
//                 //         pre_max_score[idx] = max_score[idx];
//                 //         max_score[idx] = max(max_score[idx],temp_smem[tidx]);
//                 //         max_score_diff[tidx] = exp(pre_max_score[idx]-max_score[idx]);
//                 //     }

//                 // }

//                 tile_block.sync();
//                 token = filled.arrive();
//             }
            
//         }
//     }
// }

// __device__ void stage_3(cooperative_groups::__v1::thread_block_tile<128U, cooperative_groups::__v1::thread_block>& tile_block,
// barrier& ready, barrier& filled, const int& compute_block_start, const int& compute_block_end, half* c, half* out,const int *to_select_index_position,
// const int *from_block_index, const int& batch_size, const int& head_size, const int *seq_len_info, const int& block_size, half smem_temp_score[][64+8], 
// float *temp_smem, float* pre_max_score, float* max_score, float* max_score_diff, const int *to_select_index, float* global_sum_scores){

//     const int tidy = tile_block.thread_rank()/32;
//     const int tidx = threadIdx.x;
//     const int pad = 8;
//     barrier::arrival_token token;

//     __shared__  half out_temp[64][64+pad],smem_v[64][64+pad];

//     typedef hipcub::WarpReduce<float> WarpReduce;
//     __shared__  typename WarpReduce::TempStorage temp_storage[4];

//     wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> frag_s[4];
//     wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> frag_v[4];
//     wmma::fragment<wmma::accumulator, 16, 16, 16, half> frag_out;

//     const int smem_index_i =  tidy*4+tidx/8;
//     const int smem_index_j = (tidx%8)*8;

//     filled.arrive();

//     const int load_v_smem_addr = __cvta_generic_to_shared(smem_v[0]) + (smem_index_i*(64+pad)+smem_index_j)*2;


//     float4 zero4 = {0.0f,0.0f,0.0f,0.0f};

//     for(int from_block_id_index = compute_block_start;from_block_id_index<compute_block_end;from_block_id_index++){
        
//         int from_block_id = from_block_index[from_block_id_index];
//         int seq_start_block_index = 0;
//         int seq_block_len = 0;
//         for(int i=1;i<batch_size+1;i++){
//             if(from_block_id >= seq_len_info[i]*12)
//                 continue;
//             else{
//                 seq_start_block_index = seq_len_info[i-1]; //开始的block id
//                 seq_block_len = seq_len_info[i] - seq_len_info[i-1]; // seq 拥有的block id
//                 break;
//             }
//         }

//         //还原原始的headnum和blockid
//         const int head_num = (from_block_id - 12*seq_start_block_index)/seq_block_len;
//         from_block_id = (from_block_id - 12*seq_start_block_index)%seq_block_len;

//         const int seq_start_index = 12*seq_start_block_index*block_size*head_size + head_num*seq_block_len*block_size*head_size;
//         const int data_offset_q = seq_start_index + from_block_id*block_size*head_size;

//         const int to_block_start = to_select_index_position[from_block_id+seq_start_block_index];
//         const int to_block_end = to_select_index_position[from_block_id+seq_start_block_index + 1];

//         tile_block.sync();

//         for(int i=0;i<4;i++){
//             FLOAT4(out_temp[i*16 + smem_index_i][smem_index_j]) = FLOAT4(zero4);
//         }

//         if(tidy < 2){
//             global_sum_scores[tidy*32+tidx] = 0.0;
//         }

//         int to_block_id = to_select_index[to_block_start];
//         int data_k_start = seq_start_index + to_block_id * block_size * head_size;
//         int load_k_gmem_addr = data_k_start+ smem_index_i*head_size+smem_index_j;
//         for(int i=0;i<64;i+=16){
//             asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :: "r"(load_v_smem_addr + i*(64+pad)*2), "l"(&c[load_k_gmem_addr+i*head_size]));
//         }

//         asm ("cp.async.commit_group;\n" ::);
//         asm ("cp.async.wait_group 0;\n" ::);

//         for(int block_id_index=to_block_start;block_id_index<to_block_end;block_id_index++){

//             #pragma unroll
//             for(int i=0;i<4;i++){
//                 wmma::load_matrix_sync(frag_v[i], &smem_v[i*16][tidy*16], 64+pad);
//             }

//             if(block_id_index != to_block_end - 1){
//                 to_block_id = to_select_index[block_id_index+1];
//                 data_k_start = seq_start_index + to_block_id * block_size * head_size;
                
//                 int load_k_gmem_addr = data_k_start+ smem_index_i*head_size+smem_index_j;
//                 for(int i=0;i<64;i+=16){
//                     asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :: "r"(load_v_smem_addr + i*(64+pad)*2), "l"(&c[load_k_gmem_addr+i*head_size]));
//                 }
//             }

//             for(int from_block_part_index = 0;from_block_part_index<block_size;from_block_part_index+=16){
                
//                 ready.arrive_and_wait();

//                 half2 max_s[4];
//                 half2 value[4];
//                 for(int i=0;i<4;i++){
//                     value[i] =  HALF2(smem_temp_score[tidy*4+i][0])[tidx];
//                     max_s[i] = __half2half2(__float2half(max_score[tidy*4 + i + from_block_part_index]));
//                 }
//                 half2 diff_x = __half2half2(__float2half(max_score_diff[tidy*4 + tidx/8 ]));
//                 float4 out_t = FLOAT4(out_temp[from_block_part_index + tidy*4+(tidx*8)/64][(tidx*8)%64]);
                
//                 for(int i=0;i<4;i++){
//                     half2 t = h2exp(__hsub2(value[i],max_s[i]));
//                     HALF2(out_temp[from_block_part_index + tidy*4+(tidx*8)/64][(tidx*8)%64])[i] = __hmul2(HALF2(out_t)[i],diff_x);

//                     HALF2(smem_temp_score[tidy*4+i][0])[tidx] = t;
//                     float v_ = __half2float(__hadd(HALF(t)[0],HALF(t)[1]));
//                     temp_smem[tidy*4 + i] = WarpReduce(temp_storage[tidy]).Sum(v_);
//                 }

//                 // if(tile_block.thread_rank() == 0 && from_block_part_index == 0){
//                 //     for(int i=0;i<64;i++){
//                 //         printf("%f ",__half2float(smem_temp_score[0][i]));
//                 //     }

//                 //     printf("\n");

//                 //     printf("%f %f %f %f\n",max_score[0],max_score_diff[0],global_sum_scores[0],__half2float(out_temp[0][0]));
                    
//                 // }

//                 // tile_block.sync();
                
//                 // for(int i=0;i<8;i++){
//                 //     half value = smem_temp_score[(tile_block.thread_rank()*8)/64][(tile_block.thread_rank()*8)%64+i];
//                 //     smem_temp_score[(tile_block.thread_rank()*8)/64][(tile_block.thread_rank()*8)%64+i] =  hexp(__hsub(value, max_score[(tile_block.thread_rank()*8)/64 + from_block_part_index]));
//                 //     value = out_temp[(tile_block.thread_rank()*8)/64 + from_block_part_index][(tile_block.thread_rank()*8)%64+i];
//                 //     out_temp[(tile_block.thread_rank()*8)/64 + from_block_part_index][(tile_block.thread_rank()*8)%64+i] = __hmul(value,max_score_diff[(tile_block.thread_rank()*8)/64]);
//                 // }

//                 // tile_block.sync();

//                 // if(tile_block.thread_rank()<16){
//                 //     float sum_temp = 0.0f;
//                 //     for(int i=0;i<64;i++){
//                 //         sum_temp += __half2float(smem_temp_score[tidx][i]);
//                 //     }
//                 //     temp_smem[tidx] = sum_temp;
//                 //     // printf("%f\n",sum_temp);
//                 // }

//                // 同步
//                 tile_block.sync();

//                 if(tidy == 0 && tidx < 16)
//                 {
//                     int idx = tidx+from_block_part_index;
//                     global_sum_scores[idx] *= max_score_diff[tidx];
//                     global_sum_scores[idx] += temp_smem[tidx];
//                 }

//                 // if(tile_block.thread_rank() == 0 && from_block_part_index == 0){
//                 //     for(int i=0;i<64;i++){
//                 //         printf("%f ",__half2float(smem_temp_score[0][i]));
//                 //     }

//                 //     printf("\n");

//                 //     printf("%f %f %f\n",temp_smem[0],global_sum_scores[0],__half2float(out_temp[0][0]));
                    
//                 // }
//                 // tile_block.sync();

//                 for(int i=0;i<4;i++)
//                 {
//                     wmma::load_matrix_sync(frag_s[i], &smem_temp_score[0][i*16], 64+pad);
//                 }

//                 tile_block.sync();
//                 token = filled.arrive();

//                 wmma::load_matrix_sync(frag_out,&out_temp[from_block_part_index][tidy*16],64+pad,wmma::mem_row_major);

//                 for(int i=0;i<4;i++){
//                     wmma::mma_sync(frag_out, frag_s[i], frag_v[i], frag_out);
//                 }

//                 wmma::store_matrix_sync(&out_temp[from_block_part_index][tidy*16],frag_out,64+pad,wmma::mem_row_major);

//             }

//             asm ("cp.async.commit_group;\n" ::);
//             asm ("cp.async.wait_group 0;\n" ::);
//         }

//         tile_block.sync();

//         for(int i=0;i<16;i++){
//             float sum_score_value = global_sum_scores[tidy*16+i];
//             float2 out_temp_value = __half22float2(HALF2(out_temp[tidy*16+i][0])[tidx]);

//             out_temp[tidy*16+i][tidx*2] = __float2half(out_temp_value.x/sum_score_value);
//             out_temp[tidy*16+i][tidx*2 + 1] = __float2half(out_temp_value.y/sum_score_value);

//         }

//         tile_block.sync();

//         for(int i=0;i<4;i++)
//             FLOAT4(out[data_offset_q+(i*16 + smem_index_i)*head_size+smem_index_j]) = FLOAT4(out_temp[i*16 + smem_index_i][smem_index_j]);


//     }

// }

// __global__ void sparse_attention_test(half *a,  half *b,  half *c, 
//     half *out, const int *seq_len_info,const int *from_block_index, 
//     const int *from_block_index_position, const int *to_select_index,
//     const int *to_select_index_position, const int batch_size,
//     const int block_size, const int head_size){

//     const int bidx = blockIdx.x;
//     const int bidy = blockIdx.y;

//     __shared__ barrier bar[3];

//     thread_block thb = this_thread_block();
//     auto tile_block = cg::tiled_partition<128>(thb);


//     if(thb.thread_rank() == 0)
//     {
//         init(bar+0, 256);
//         init(bar+1, 256);
//         init(bar+2, 256);
//     }


//     __shared__ __align__(32) float global_sum_scores[64],pre_max_score[64],max_score[64];  
//     __shared__ __align__(32) half smem_temp_score[16][64+8];
//     __shared__  float temp_smem[32],max_score_diff[16];



//     const int compute_block_start = from_block_index_position[bidx];
//     const int compute_block_end = from_block_index_position[bidx + 1];
//     const int compute_block_num = compute_block_end - compute_block_start;

//     thb.sync();

//     if(tile_block.meta_group_rank() == 0)
//         stage1(tile_block,bar[1],bar[0],a,b,smem_temp_score,compute_block_start,
//         compute_block_end,from_block_index,batch_size,seq_len_info,block_size,
//         head_size,to_select_index_position,to_select_index,out);
    
//     if(tile_block.meta_group_rank() == 1)
//         stage_2(tile_block,bar[0],bar[2],compute_block_start,compute_block_end,to_select_index_position,
//         from_block_index,batch_size,seq_len_info,block_size,smem_temp_score,temp_smem,
//         pre_max_score,max_score,max_score_diff);


//     if(tile_block.meta_group_rank() == 2)
//         stage_3(tile_block,bar[2],bar[1],compute_block_start,compute_block_end,c,out,
//         to_select_index_position,from_block_index,batch_size,head_size,seq_len_info,block_size,
//         smem_temp_score,temp_smem,pre_max_score,max_score,max_score_diff,to_select_index,
//         global_sum_scores);

    
//     // thb.sync();
//     // if(thb.thread_rank() == 0){
//     //     for(int i=0;i<64;i++)
//     //     {
//     //         for(int j=0;j<64;j++){
//     //             printf("%f ",__half2float(out[i*64+j]));
//     //         }
//     //         printf("\n");
//     //     }
//     //     printf("\n");
//     // }
//     // thb.sync();

// }


__inline__ __device__ void warpReduceSum(half values[][72],const int tidx, const int tidy, float *store_v){
    float4 value[2] = {FLOAT4(values[tidx][tidy*16]),FLOAT4(values[tidx][tidy*16 + 8])};

    float v[2];
    float result[16];
    float temp[2];
    for(int i=0;i<8;i++){
        v[0] = __half2float(HALF(value)[i*2]);
        v[1] = __half2float(HALF(value)[i*2 + 1]);
        for(int j=16; j>=1; j/=2){
            temp[0] += __shfl_xor_sync(0xffffffff, v[0], j, 32);
            temp[1] += __shfl_xor_sync(0xffffffff, v[1], j, 32);
            // v[0] += temp[0];
            // v[1] += temp[1];

        }
        result[i*2] = __shfl_sync(0xffffffff, v[0], 0);
        result[i*2 + 1] = __shfl_sync(0xffffffff, v[1], 0);
    }
    store_v[0] = result[tidx/4];
    store_v[1] = result[tidx/4 + 8];

}

__inline__ __device__ void warpReduceMax(half values[][72],const int tidx, const int tidy, float *store_v){
    float4 value[2] = {FLOAT4(values[tidx][tidy*16]),FLOAT4(values[tidx][tidy*16 + 8])};
    float v[2];
    float result[16];
    for(int i=0;i<8;i++){
        v[0] = __half2float(HALF(value)[i*2]);
        v[1] = __half2float(HALF(value)[i*2 + 1]);
        for(int j=16; j>=1; j/=2){
            v[0] = max(v[0], __shfl_xor_sync(0xffffffff, v[0], j, 32));
            v[1] = max(v[1], __shfl_xor_sync(0xffffffff, v[1], j, 32));
        }
        result[i*2] = __shfl_sync(0xffffffff, v[0], 0);
        result[i*2 + 1] = __shfl_sync(0xffffffff, v[1], 0);
    }

    store_v[0] = result[tidx/4];
    store_v[1] = result[tidx/4 + 8];
}

__global__ void sparse_attention_lastest(half *a,  half *b,  half *c, 
    half *out,const int *seq_len_info,const int *from_block_index, 
    const int *from_block_index_position, const int *to_select_index,
    const int *to_select_index_position, const int batch_size,
    const int block_size,const int head_size){

    const int tidy = threadIdx.y;
    const int tidx = threadIdx.x;
    const int bidx = blockIdx.x;
    const int bidy = blockIdx.y;
    const int pad = 8;
    
    // typedef hipcub::WarpReduce<float> WarpReduce;
    // __shared__ __align__(32)  typename WarpReduce::TempStorage temp_storage[4];
    __shared__ __align__(32)  float global_sum_scores[64],pre_max_score[64],max_score[64];  
    __shared__ __align__(32)  float temp_smem[64],max_score_diff[16],sum_temp[64];
    __shared__ __align__(32)  half smem_q[64][64+pad],smem_k[2][32][64+pad],smem_v[2][32][64+pad],smem_temp_score[16][64+pad],out_temp[64][64+pad];

    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> frag_q[4];
    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::col_major> frag_k[2][4];
    wmma::fragment<wmma::accumulator, 16, 16, 16, half> frag_s_out[2];

    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> frag_s[2];
    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> frag_v[4][2];
    wmma::fragment<wmma::accumulator, 16, 16, 16, half> frag_out[4];

    const int smem_index_i =  tidy*4+tidx/8;
    const int smem_index_j = (tidx%8)*8;

    const int load_k_smem_addr[2] = {__cvta_generic_to_shared(smem_k[0]) + (smem_index_i*(64+pad)+smem_index_j)*2,__cvta_generic_to_shared(smem_k[1]) + (smem_index_i*(64+pad)+smem_index_j)*2};
    const int load_v_smem_addr[2] = {__cvta_generic_to_shared(smem_v[0]) + (smem_index_i*(64+pad)+smem_index_j)*2,__cvta_generic_to_shared(smem_v[1]) + (smem_index_i*(64+pad)+smem_index_j)*2};
    const int load_q_smem_addr = __cvta_generic_to_shared(smem_q[0]) + (smem_index_i*(64+pad)+smem_index_j)*2;
    const int compute_block_start = from_block_index_position[bidx];
    const int compute_block_end = from_block_index_position[bidx + 1];
    const int compute_block_num = compute_block_end - compute_block_start;

    for(int from_block_id_index = compute_block_start;from_block_id_index<compute_block_end;from_block_id_index++){
    
        int from_block_id = from_block_index[from_block_id_index];
        int seq_start_block_index = 0;
        int seq_block_len = 0;
        for(int i=1;i<batch_size+1;i++){
            if(from_block_id >= seq_len_info[i]*12)
                continue;
            else{
                seq_start_block_index = seq_len_info[i-1]; //开始的block id
                seq_block_len = seq_len_info[i] - seq_len_info[i-1]; // seq 拥有的block id
                break;
            }
        }

        //还原原始的headnum和blockid
        const int head_num = (from_block_id - 12*seq_start_block_index)/seq_block_len;
        from_block_id = (from_block_id - 12*seq_start_block_index)%seq_block_len;

        const int seq_start_index = 12*seq_start_block_index*block_size*head_size + head_num*seq_block_len*block_size*head_size;
        const int data_offset_q = seq_start_index + from_block_id*block_size*head_size;

        const int to_block_start = to_select_index_position[from_block_id+seq_start_block_index];
        const int to_block_end = to_select_index_position[from_block_id+seq_start_block_index + 1];
        const int to_block_num = to_block_end - to_block_start;

        int to_block_id = to_select_index[0];
        int data_k_start = seq_start_index + to_block_id * block_size * head_size;
        int load_k_gmem_addr = data_k_start+ smem_index_i*head_size+smem_index_j;
        int load_q_gmem_addr = data_offset_q+ smem_index_i*head_size+smem_index_j;

        for(int i=0;i<64;i+=16){
            asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :: "r"(load_q_smem_addr + i*(64+pad)*2), "l"(&a[load_q_gmem_addr+i*head_size]));
        }

        for(int i=0;i<32;i+=16){
            asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :: "r"(load_k_smem_addr[0] + i*(64+pad)*2), "l"(&b[load_k_gmem_addr+i*head_size]));
            asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :: "r"(load_v_smem_addr[0] + i*(64+pad)*2), "l"(&c[load_k_gmem_addr+i*head_size]));
        }
        
        for(int i=0;i<4;i++)
            wmma::fill_fragment(frag_out[i], __float2half(0.0));

        asm ("cp.async.commit_group;\n" ::);
        asm ("cp.async.wait_group 0;\n" ::);

        for(int i=0;i<4;i++){
            wmma::load_matrix_sync(frag_q[i], &smem_q[tidy*16][i*16], 64+pad);
        }

        float pre_max_score[2] = {FLT_MIN,FLT_MIN};
        float max_score[2] = {FLT_MIN,FLT_MIN};
        half2 max_diff[2];
        float global_sum[2] = {0.0f, 0.0f};


        for(int block_id_index=0;block_id_index<to_block_num*2;block_id_index++){
            #pragma unroll
            for(int i=0;i<2;i++)
            {
                #pragma
                for(int j=0;j<4;j++){
                    wmma::load_matrix_sync(frag_k[i][j], &smem_k[block_id_index & 1][i*16][j*16], 64+pad);
                    wmma::load_matrix_sync(frag_v[j][i], &smem_v[block_id_index & 1][i*16][j*16], 64+pad);
                }

                 wmma::fill_fragment(frag_s_out[i], __float2half(0.0));
            }

            if(block_id_index != to_block_num*2 - 1){
                to_block_id = to_select_index[(block_id_index+1)/2];
                data_k_start = seq_start_index + to_block_id * block_size * head_size ;
                
                int load_k_gmem_addr = data_k_start+ smem_index_i*head_size+smem_index_j;
                if((block_id_index+1) & 1){
                    load_k_gmem_addr += 32*head_size;
                }

                for(int i=0;i<32;i+=16){
                    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :: "r"(load_k_smem_addr[(block_id_index+1) & 1] + i*(64+pad)*2), "l"(&b[load_k_gmem_addr+i*head_size]));
                    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :: "r"(load_v_smem_addr[(block_id_index+1) & 1] + i*(64+pad)*2), "l"(&c[load_k_gmem_addr+i*head_size]));
                }
            }

            #pragma unroll
            for(int i=0;i<2;i++)
            {
                #pragma
                for(int j=0;j<4;j++){
                    wmma::mma_sync(frag_s_out[i], frag_q[j], frag_k[i][j], frag_s_out[i]);
                }
                 wmma::store_matrix_sync(&smem_q[i*16][tidy*16], frag_s_out[i], 64+pad, wmma::mem_col_major);
            }

            float value[4];

            float max_t[2];
            unsigned maxk_temp = 0x0000000f;

            // for(int i=0;i<4;i++){
            //     value[0] = __half2float(smem_q[tidy*16+i*4+0][tidx]);
            //     value[1] = __half2float(smem_q[tidy*16+i*4+1][tidx]);
            //     value[2] = __half2float(smem_q[tidy*16+i*4+2][tidx]);
            //     value[3] = __half2float(smem_q[tidy*16+i*4+3][tidx]);
            //     temp_smem[tidy*16+i*4+0] = WarpReduce(temp_storage[tidy]).Reduce(value[0], hipcub::Max());
            //     temp_smem[tidy*16+i*4+1] = WarpReduce(temp_storage[tidy]).Reduce(value[1], hipcub::Max());
            //     temp_smem[tidy*16+i*4+2] = WarpReduce(temp_storage[tidy]).Reduce(value[2], hipcub::Max());
            //     temp_smem[tidy*16+i*4+3] = WarpReduce(temp_storage[tidy]).Reduce(value[3], hipcub::Max());
            // }
            warpReduceMax(smem_q,tidx,tidy,max_t);

            // max_t[0] = temp_smem[tidy*16+tidx/4];
            // max_t[1] = temp_smem[tidy*16+tidx/4 + 8];

            FLOAT2(pre_max_score[0])[0] = FLOAT2(max_score[0])[0];
            max_score[0] = max(max_score[0],max_t[0]);
            max_score[1] = max(max_score[1],max_t[1]);



            max_diff[0] = __half2half2(__float2half(exp(pre_max_score[0]-max_score[0])));
            max_diff[1] = __half2half2(__float2half(exp(pre_max_score[1]-max_score[1])));

            for(int i=0;i<frag_s_out[0].num_elements;i+=4){
                HALF2(frag_s_out[0].x[i])[0]  = h2exp(__hsub2(HALF2(frag_s_out[0].x[i])[0],__half2half2(__float2half(max_score[0]))));
                HALF2(frag_s_out[0].x[i+2])[0]  = h2exp(__hsub2(HALF2(frag_s_out[0].x[i+2])[0],__half2half2(__float2half(max_score[1]))));
                HALF2(frag_s_out[1].x[i])[0]  = h2exp(__hsub2(HALF2(frag_s_out[1].x[i])[0],__half2half2(__float2half(max_score[0]))));
                HALF2(frag_s_out[1].x[i+2])[0]  = h2exp(__hsub2(HALF2(frag_s_out[1].x[i+2])[0],__half2half2(__float2half(max_score[1]))));   
            }

            for(int i=0;i<2;i++){
                 wmma::store_matrix_sync(&smem_q[i*16][tidy*16], frag_s_out[i], 64+pad, wmma::mem_col_major);
            }


            // float tt;

            // for(int i=0;i<4;i++){
            //     value[0] = __half2float(smem_q[tidy*16+i*4+0][tidx]);
            //     value[1] = __half2float(smem_q[tidy*16+i*4+1][tidx]);
            //     value[2] = __half2float(smem_q[tidy*16+i*4+2][tidx]);
            //     value[3] = __half2float(smem_q[tidy*16+i*4+3][tidx]);
            //     temp_smem[tidy*16+i*4+0] = WarpReduce(temp_storage[tidy]).Reduce(value[0], hipcub::Sum());
            //     temp_smem[tidy*16+i*4+1] = WarpReduce(temp_storage[tidy]).Reduce(value[1], hipcub::Sum());
            //     temp_smem[tidy*16+i*4+2] = WarpReduce(temp_storage[tidy]).Reduce(value[2], hipcub::Sum());
            //     temp_smem[tidy*16+i*4+3] = WarpReduce(temp_storage[tidy]).Reduce(value[3], hipcub::Sum());
            // }
            float global_sum_t[2];

            warpReduceSum(smem_q,tidx,tidy,global_sum_t);

            // 没问题
            global_sum[0] *= __half2float(max_diff[0].x);
            global_sum[1] *= __half2float(max_diff[1].x);
            global_sum[0] += global_sum_t[0];
            global_sum[1] += global_sum_t[1];


            // 没问题
            for(int i=0;i<frag_s[0].num_elements/2;i+=4){
                HALF2(frag_s[0].x[i])[0] = HALF2(frag_s_out[0].x[i])[0];
                HALF2(frag_s[0].x[i+2])[0] = HALF2(frag_s_out[0].x[i+2])[0];
                HALF2(frag_s[0].x[i+8])[0] = HALF2(frag_s_out[0].x[i])[0];
                HALF2(frag_s[0].x[i+10])[0] = HALF2(frag_s_out[0].x[i+2])[0];
                HALF2(frag_s[1].x[i])[0] = HALF2(frag_s_out[1].x[i])[0];
                HALF2(frag_s[1].x[i+2])[0] = HALF2(frag_s_out[1].x[i+2])[0];
                HALF2(frag_s[1].x[i+8])[0] = HALF2(frag_s_out[1].x[i])[0];
                HALF2(frag_s[1].x[i+10])[0] = HALF2(frag_s_out[1].x[i+2])[0];
            }

            for(int i=0;i<4;i++){
                for(int j=0;j<frag_out[0].num_elements;j+=4){
                    HALF2(frag_out[i].x[j])[0] = __hmul2(HALF2(frag_out[i].x[j])[0],max_diff[0]);
                    HALF2(frag_out[i].x[j+2])[0] = __hmul2(HALF2(frag_out[i].x[j+2])[0],max_diff[1]);
                }
            }

            for(int i=0;i<4;i++){
                for(int j=0;j<2;j++)
                    wmma::mma_sync(frag_out[i], frag_s[j], frag_v[i][j], frag_out[i]);
            }

            asm ("cp.async.commit_group;\n" ::);
            asm ("cp.async.wait_group 0;\n" ::);

        }

        for(int i=0;i<4;i++){
            for(int j=0;j<frag_out[0].num_elements;j+=4){
                HALF2(frag_out[i].x[j])[0] = __h2div(HALF2(frag_out[i].x[j])[0],__half2half2(__float2half(global_sum[0])));
                HALF2(frag_out[i].x[j+2])[0] = __h2div(HALF2(frag_out[i].x[j+2])[0],__half2half2(__float2half(global_sum[1])));
            }
        }

        for(int i=0;i<4;i++)
            wmma::store_matrix_sync(&smem_q[tidy*16][i*16],frag_out[i],64+pad,wmma::mem_row_major);

        for(int i=0;i<4;i++)
            FLOAT4(out[data_offset_q+(i*16 + smem_index_i)*head_size+smem_index_j]) = FLOAT4(smem_q[i*16 + smem_index_i][smem_index_j]);

    }
}

void test_gemm_1(half *a, half *b,half *c, half *out,int batch_size,int *seq_len_info,int *from_select_index,int *from_select_index_position,int *to_select_index,int *to_select_index_position,int block_limit, int block_num, int head_num,int block_size,int head_size,std::map<std::string,float>& info,bool balanced)
{


    // hipEvent_t start,stop;
    // hipEventCreate( &start );
    // hipEventCreate( &stop ) ;
    // hipEventRecord( start, 0 ) ;
    // 修改成最大线程块数量 80 * 2

    auto start_time = std::chrono::system_clock::now();

    if(balanced)
        // sparse_attention_banlanced<half><<<dim3(block_limit),dim3(32,4)>>>(a,b,c,out,seq_len_info,from_select_index,from_select_index_position,to_select_index,to_select_index_position,batch_size,block_size,head_size);
        sparse_attention_lastest<<<dim3(block_limit),dim3(32,4)>>>(a,b,c,out,seq_len_info,from_select_index,from_select_index_position,to_select_index,to_select_index_position,batch_size,block_size,head_size);
    else{
        sparse_attention_non_balanced<half><<<dim3(head_num,block_num),dim3(32,4)>>>(a,b,c,out,seq_len_info,from_select_index,from_select_index_position,to_select_index,to_select_index_position,batch_size,block_size,head_size);
    }

    // hipError_t cudaerr = hipDeviceSynchronize();
    // if (cudaerr != hipSuccess)
    //     printf("kernel launch failed with error \"%s\".\n",
    //            hipGetErrorString(cudaerr));
    // printf("ending\n");

    auto end_time = std::chrono::system_clock::now();
    if(info.find("attention") != info.end())
    {    auto dura = (std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time)).count();
        info["attention"] += dura;
    }
    // hipEventRecord(stop,0);
    // float elapsedTime;
    // hipEventSynchronize(stop);
    // hipDeviceSynchronize();
    // hipEventElapsedTime(&elapsedTime, start, stop);
    // printf( "Time to generate:  %f ms\n", elapsedTime );
}

void test_gemm_(float *a, float *b,float *c, float *out,int *to_select_index,int *to_select_index_position, int block_num, int head_num,int block_size,int head_size)
{
    // std::cout<<*a<<std::endl;
    // sparse_attention<float><<<dim3(block_num,head_num),dim3(11,32)>>>(a,b,c,out,to_select_index,64,64);

    hipEvent_t start,stop;
    hipEventCreate( &start );
    hipEventCreate( &stop ) ;
    // test_gemm<float><<<1,dim3(11,32)>>>(a,b,c,m,n,k,64);

    hipEventRecord( start, 0 ) ;
    sparse_attention<float><<<dim3(head_num,block_num),dim3(32,8)>>>(a,b,c,out,to_select_index,to_select_index_position,64,64,11);

    // test_gpu<<<1,1>>>();
    // test_cpu();
    hipEventRecord(stop,0);
    float elapsedTime;
    hipEventSynchronize(stop);
    hipDeviceSynchronize();
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf( "Time to generate:  %f ms\n", elapsedTime );
    // printf("%f\n",*a);
}


}
}
}