#include "hip/hip_runtime.h"
#include "kernel.h"
#include "hip/hip_runtime.h"
#include "hip/hip_fp16.h"
#include <hip/hip_fp16.h>
#include ""
#include <stdio.h>
#include <iostream>
#include <numeric>
#include <cooperative_groups/memcpy_async.h>
#include <thrust/extrema.h>
#include <mma.h>
// #include <hipcub/hipcub.hpp>
#define FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])
#define FLOAT(pointer) (reinterpret_cast<float*>(&(pointer)))
using namespace nvcuda;
namespace sparse_transformers {
namespace layers {
namespace kernels {

// a is not transpose
template <class DataType>
__global__ void sparse_attention_with_tensor_core(DataType *a,  DataType *b,  DataType *c, DataType *out, const int *to_select_index,const int *to_select_index_position,  const int block_size,const int head_size,const int select_block_num){

    const int tidy = threadIdx.y;
    const int tidx = threadIdx.x;
    const int bidx = blockIdx.x;
    const int bidy = blockIdx.y;
    const int b_dimx = 8;
    const int g_dimy = gridDim.y;

    // 计算Q的起始位置
    const int from_block_id = bidy;
    const int data_offset_q = (bidx*g_dimy + from_block_id) * block_size * head_size;

    const int A_BM = 32;
    const int A_BK = 64;
    const int B_BK = 32;
    const int B_BN = 4;
    const int C_BK = 32;
    const int C_BN = 4;


    __shared__ half smem_q[32 * 64],smem_k[32 * 64],smem_v[32 * 64],smem_temp_half[64][32];
    __shared__ float temp_score[32][32],out_temp[32][64],max_score_diff[32];
    __shared__ half temp_score_half[32][32];

    __shared__ float global_sum_scores[32],pre_max_score[32],max_score[32],temp_smem[32];

    typedef hipcub::WarpReduce<float> WarpReduce;
    __shared__ typename WarpReduce::TempStorage temp_storage[8];

    float zero4[4] = {0.0f,0.0f,0.0f,0.0f};
    wmma::fragment<wmma::matrix_a, 8, 32, 16, half, wmma::row_major> frag_q[2];
    wmma::fragment<wmma::matrix_b, 8, 32, 16, half, wmma::col_major> frag_k[2];
    wmma::fragment<wmma::accumulator, 8, 32, 16, half> frag_s_out;

    wmma::fragment<wmma::matrix_a, 8, 32, 16, half, wmma::row_major> frag_s[2];
    wmma::fragment<wmma::matrix_b, 8, 32, 16, half, wmma::row_major> frag_v[2];
    wmma::fragment<wmma::accumulator, 8, 32, 16, float> frag_out;

    auto block_k = cooperative_groups::this_thread_block();
    auto block_v = cooperative_groups::this_thread_block();

    for(int a_bm = 0; a_bm< block_size/A_BM; a_bm++){

        const int to_block_start = to_select_index_position[from_block_id];
        const int to_block_end = to_select_index_position[from_block_id+1];

        

        int to_block_id = to_select_index[to_block_start];

        int data_b_start = (bidx*g_dimy+to_block_id) * block_size * head_size;
        const int smem_index =  32*8*tidy + tidx*8; // warp_size * 8

        // 加载Q的部分数据,32*64
        // Q  tidy
        //      --------------------
        //          0     |     1
        //      --------------------
        //          2     |     3
        //      --------------------
        //          4     |     5
        //      --------------------
        //          6     |     7
        //      --------------------
        const int smem_index_q = 32*8*tidy + (tidx/16)*16*8 + (tidx%16)*8;
        const int global_q_index_i = (a_bm*A_BM + (tidx%16)/2 + (tidy/2)*8);
        const int global_q_index_j = ((tidx%2)*8 + (tidx/16)*16 + (tidy%2)*32);
        FLOAT4(smem_q[smem_index_q]) = FLOAT4(a[data_offset_q+global_q_index_i*head_size+global_q_index_j]); 

        FLOAT4(out_temp[global_q_index_i - a_bm*A_BM][global_q_index_j]) = FLOAT4(zero4[0]);
        wmma::fill_fragment(frag_s_out, 0.0);
        wmma::fill_fragment(frag_out, 0.0);


        // KT 32*16  tidy  Q*K 8*16*32
        //      -----------------------------------------
        //          0     |     2   |   4     |     6   
        //      -----------------------------------------
        //          1     |     3   |   5     |     7   
        //      -----------------------------------------

        const int global_k_index_i = tidx/2 + (tidy%2)*16;
        const int global_k_index_j = (tidx%2)*8 + (tidy/2)*16;
        FLOAT4(smem_k[smem_index]) = FLOAT4(b[data_b_start + global_k_index_i*head_size+global_k_index_j]); 


        // V 16*32  tidy  S*K 8*16*32
        //      --------------------
        //          0     |     4
        //      --------------------
        //          1     |     5
        //      --------------------
        //          2     |     6
        //      --------------------
        //          3     |     7
        //      --------------------
        const int global_v_index_i = (tidx/4 + ((tidy/2)%2)*16 + (tidy%2)*8);
        const int global_v_index_j = ((tidx*8)%32 + ((tidy/2)/2)*32);
        FLOAT4(smem_v[smem_index]) = FLOAT4(c[data_b_start + global_v_index_i*head_size+global_v_index_j]); 

        wmma::load_matrix_sync(frag_q[0], &smem_q[tidy*2*8*16], 16);
        wmma::load_matrix_sync(frag_q[1], &smem_q[(tidy*2+1)*8*16], 16);

        // 初始化sharedmem
        if(tidy == 0){
            max_score[tidx] = 0.0f;
            // sum_score_max[tidx] = 0.0f;
            pre_max_score[tidx] = 0.0f;
            global_sum_scores[tidx] = 0.0f;
        }


        // 遍历K、V的每一个Block进行计算
        for(int block_id=to_block_start;block_id<to_block_end;block_id++)
        {
            // KV按照 32*64 的大小进行加载计算
            for(int b_bn=0;b_bn<block_size/32;b_bn++){
                
                // 计算Q*K
                // cooperative_groups::wait(block_k);
                wmma::fill_fragment(frag_s_out, 0.0);
                if(tidy % 2 == 0){
                    wmma::load_matrix_sync(frag_k[0], &smem_k[0], 16);
                    wmma::load_matrix_sync(frag_k[1], &smem_k[16*32], 16);
                }
                else{
                    wmma::load_matrix_sync(frag_k[0], &smem_k[16*32*2], 16);
                    wmma::load_matrix_sync(frag_k[1], &smem_k[16*32*3], 16);
                }
                wmma::mma_sync(frag_s_out, frag_q[0], frag_k[0], frag_s_out);
                wmma::mma_sync(frag_s_out, frag_q[1], frag_k[1], frag_s_out);
                wmma::store_matrix_sync(&smem_temp_half[(tidy/2)*8 + (tidy % 2)*32][0], frag_s_out, 32, wmma::mem_row_major);
                
                __syncthreads();

                for(int i=0;i<4;i++){
                    temp_score[i*8+tidy][tidx] = __half2float(smem_temp_half[i*8+tidy][tidx] + smem_temp_half[i*8+tidy+32][tidx]);
                }

                //加载下一次使用的数据
                const int next_block_id = b_bn == 1 ? block_id+1:block_id;
                const int next_bn = (b_bn + 1) & 1;
                to_block_id = to_select_index[next_block_id];
                data_b_start = (bidx*g_dimy+to_block_id) * block_size * head_size;
                if(block_id != to_block_end - 1 || b_bn != 1)
                {
                    FLOAT4(smem_k[smem_index]) = FLOAT4(b[data_b_start + next_bn*32*head_size + global_k_index_i*head_size+global_k_index_j]);
                }
            
                //计算最大值 rowmax
                {
                    float value1 = temp_score[tidy][tidx];
                    float value2 = temp_score[tidy+16][tidx];
                    float value3 = temp_score[tidy+8][tidx];
                    float value4 = temp_score[tidy+24][tidx];
                    
                    temp_smem[tidy] = WarpReduce(temp_storage[tidy]).Reduce(value1, hipcub::Max());
                    temp_smem[tidy+16] = WarpReduce(temp_storage[tidy]).Reduce(value2, hipcub::Max());
                    temp_smem[tidy+8] = WarpReduce(temp_storage[tidy]).Reduce(value3, hipcub::Max());
                    temp_smem[tidy+24] = WarpReduce(temp_storage[tidy]).Reduce(value4, hipcub::Max());

                    __syncthreads();
                    if(tidy == 0)
                    {
                        pre_max_score[tidx] = max_score[tidx];
                        max_score[tidx] = max_score[tidx]>temp_smem[tidx]?max_score[tidx]:temp_smem[tidx];
                        max_score_diff[tidx] = exp(pre_max_score[tidx] - max_score[tidx]);
                    }
                }

                //计算差值
                {
                    __syncthreads();
                    for(int i=0;i<4;i++)
                    {
                        float temp =  exp(temp_score[(i+tidy*4)][tidx] - max_score[tidy*4+i]);
                        temp_score[(i+tidy*4)][tidx] = temp;
                        temp_score_half[(i+tidy*4)][tidx] = __float2half(temp);
                    }

                    const int t = (tidy/2)*8+(tidx%4)*2;
                    for(int i=0;i<4;i++){
                        frag_out.x[i*2] *= max_score_diff[t];
                        frag_out.x[i*2+1] *= max_score_diff[t+1];
                    }
                    __syncthreads();  
                }

                {
                    float value1 = temp_score[tidy][tidx];
                    float value2 = temp_score[tidy+16][tidx];
                    float value3 = temp_score[tidy+8][tidx];
                    float value4 = temp_score[tidy+24][tidx];
                    
                    temp_smem[tidy] = WarpReduce(temp_storage[tidy]).Sum(value1);
                    temp_smem[tidy+16] = WarpReduce(temp_storage[tidy]).Sum(value2);
                    temp_smem[tidy+8] = WarpReduce(temp_storage[tidy]).Sum(value3);
                    temp_smem[tidy+24] = WarpReduce(temp_storage[tidy]).Sum(value4);
                    
                    __syncthreads();

                    if(tidy == 0)
                    {
                        global_sum_scores[tidx] *= max_score_diff[tidx];
                        global_sum_scores[tidx] += temp_smem[tidx];
                    }
                }

                // //计算S*V
                wmma::load_matrix_sync(frag_s[0], &temp_score_half[(tidy/2)*8][0], 32);
                wmma::load_matrix_sync(frag_s[1], &temp_score_half[(tidy/2)*8][16], 32);
                if(tidy % 2 == 0){
                    wmma::load_matrix_sync(frag_v[0], &smem_v[0], 32);
                    wmma::load_matrix_sync(frag_v[1], &smem_v[32*16], 32);
                }
                else{
                    wmma::load_matrix_sync(frag_v[0], &smem_v[32*16*2], 32);
                    wmma::load_matrix_sync(frag_v[1], &smem_v[32*16*3], 32);
                }

                wmma::mma_sync(frag_out, frag_s[0], frag_v[0], frag_out);
                wmma::mma_sync(frag_out, frag_s[1], frag_v[1], frag_out);
                
                //加载下一次使用的数据
                if(block_id != to_block_end - 1 || b_bn != 1)
                {
                    FLOAT4(smem_v[smem_index]) = FLOAT4(c[data_b_start + next_bn*32*head_size + global_v_index_i*head_size+global_v_index_j]); 
                }

            }
        }
        // __syncthreads();

        wmma::store_matrix_sync(&out_temp[(tidy/2)*8][(tidy % 2)*32], frag_out, 64, wmma::mem_row_major);

        __syncthreads();

        const int index_x = (tidy%4)*8;
        const int index_y = tidx + (tidy/4)*32;
        // 结果写入global mem
        #pragma unroll
        for(int i=0;i<8;i+=1){
            out[data_offset_q+(a_bm*A_BM+index_x+i)*head_size+index_y] = out_temp[(index_x+i)][index_y] / global_sum_scores[index_x+i];
        }
        __syncthreads();
    }
}

template <class DataType>
__global__ void sparse_attention(DataType *a,  DataType *b,  DataType *c, 
    DataType *out, const int *to_select_index,const int *to_select_index_position, 
    const int block_size,const int head_size,const int select_block_num){


    const int tidy = threadIdx.y;
    const int tidx = threadIdx.x;
    const int bidx = blockIdx.x;
    const int bidy = blockIdx.y;
    const int b_dimx = 8;
    const int g_dimy = gridDim.y;

    // 计算Q的起始位置
    const int from_block_id = bidy;
    const int data_offset_a = (bidx*g_dimy + from_block_id) * block_size * head_size;

    const int A_BM = 32;
    const int A_BK = 64;
    const int B_BK = 32;
    const int B_BN = 4;
    const int C_BK = 32;
    const int C_BN = 4;


    __shared__ float smem_q[64][32],smem_k[32][64],temp_score[32][32],smem_v[32][64];

    __shared__ float out_temp[32][64],global_sum_scores[32],temp_smem[16][32],pre_max_score[32],max_score[32];

    float zero4[4] = {0.0f,0.0f,0.0f,0.0f};

    auto block_k = cooperative_groups::this_thread_block();
    auto block_v = cooperative_groups::this_thread_block();

    for(int a_bm = 0; a_bm< block_size/A_BM; a_bm++){

        const int to_block_start = to_select_index_position[from_block_id];
        const int to_block_end = to_select_index_position[from_block_id+1];

        int to_block_id = to_select_index[to_block_start];

        int data_b_start = (bidx*g_dimy+to_block_id) * block_size * head_size;
        
        cooperative_groups::memcpy_async(block_k, smem_k[0], b+data_b_start, sizeof(float)*32*64);
        cooperative_groups::memcpy_async(block_v, smem_v[0], c+data_b_start, sizeof(float)*32*64);


        const int smem_index =  32*8*tidy + tidx*8; // warp_size * 8
        const int global_a_index_i = (smem_index / 32 );
        const int global_a_index_j = (smem_index % 32 + a_bm*A_BM);

        // 加载Q的部分数据,32*64
        #pragma unroll
        for(int i=0;i<8;i+=4){
            FLOAT4(smem_q[smem_index/32][smem_index % 32+i]) = FLOAT4(a[data_offset_a + global_a_index_i*head_size+global_a_index_j+i]); 
            FLOAT4(out_temp[smem_index/64][smem_index % 64+i]) = FLOAT4(zero4[0]);
        }

        // 初始化sharedmem
        if(tidy == 0){
            max_score[tidx] = 0.0f;
            // sum_score_max[tidx] = 0.0f;
            pre_max_score[tidx] = 0.0f;
            global_sum_scores[tidx] = 0.0f;
        }

        __syncthreads();

        // 遍历K、V的每一个Block进行计算
        for(int block_id=to_block_start;block_id<to_block_end;block_id++)
        {

            // if(bidx == 0 && bidy == 4 && tidx == 0 && tidy == 0)
            //     printf("%d %d %d %d\n",to_block_start,to_block_end,block_id,to_select_index[block_id]);
            // 计算KV块的起始位置
            
            // KV按照 32*64 的大小进行加载计算
            for(int b_bn=0;b_bn<block_size/32;b_bn++){
                
                // 计算Q*K
                cooperative_groups::wait(block_k);

                //32*64 64*32

                for(int i=0;i<4;i++){
                    float temp = 0.0f;
                    for(int j=0;j<64;j++){
                        temp += smem_q[j][tidx] * smem_k[tidy*4+i][j];
                    }
                    temp_score[tidy*4+i][tidx] = temp;
                }

                //加载下一次使用的数据
                const int next_block_id = b_bn == 1 ? block_id+1:block_id;
                const int next_bn = (b_bn + 1) & 1;
                to_block_id = to_select_index[next_block_id];
                const int data_b_start = (bidx*g_dimy+to_block_id) * block_size * head_size;
                __syncthreads();
                if(block_id != to_block_end - 1 || b_bn != 1)
                {
                    cooperative_groups::memcpy_async(block_k, smem_k[0], b+data_b_start+next_bn*32*head_size, sizeof(float)*32*64);
                }
            
                //计算最大值 rowmax
                {
                    int num = 16;
                    while(num >= 1)
                    {
                        if(num == 16)
                        {
                            float value1 = temp_score[tidy][tidx];
                            float value2 = temp_score[tidy+16][tidx];
                            float value3 = temp_score[tidy+8][tidx];
                            float value4 = temp_score[tidy+24][tidx];

                            temp_smem[tidy][tidx] = value1>value2?value1:value2;
                            temp_smem[tidy+8][tidx] = value3>value4?value3:value4;
                        }
                        else if(tidy < num){
                            float value1 = temp_smem[tidy][tidx];
                            float value2 = temp_smem[tidy+num][tidx];
                            temp_smem[tidy][tidx] = value1>value2?value1:value2;
                        }
                        num = num >> 1;
                        __syncthreads();
                    }
                    if(tidy == 0)
                    {
                        pre_max_score[tidx] = max_score[tidx];
                        max_score[tidx] = max_score[tidx]>temp_smem[0][tidx]?max_score[tidx]:temp_smem[0][tidx];
                    }
                }

                //计算差值
                {
                    __syncthreads();
                    for(int i=0;i<4;i++)
                    {
                        float temp =  exp(temp_score[(i+tidy*4)][tidx] - max_score[tidx]);
                        temp_score[(i+tidy*4)][tidx] = temp;
                    }

                    #pragma unroll
                    for(int i=0;i<4;i++){
                        float diff = (pre_max_score[tidy*4+i] - max_score[tidy*4+i]);
                        if(diff != 0){
                            diff = exp(diff);
                            out_temp[tidy*4+i][tidx] *= diff;
                            out_temp[tidy*4+i][tidx+32] *= diff;
                        }
                    }
                    __syncthreads();
                    if(tidy == 0){
                        float diff = exp(pre_max_score[tidx] - max_score[tidx]);
                        global_sum_scores[tidx] *= diff;
                    }
                    

                }
                
                {
                    int num = 16;
                    while(num >= 1)
                    {
                        if(num == 16)
                        {
                            float value1 = temp_score[tidy][tidx];
                            float value2 = temp_score[tidy+16][tidx];
                            float value3 = temp_score[tidy+8][tidx];
                            float value4 = temp_score[tidy+24][tidx];
                            temp_smem[tidy][tidx] = value1 + value2;
                            temp_smem[tidy+8][tidx] = value3 + value4;
                        }
                        else if(tidy < num){
                            float value1 = temp_smem[tidy][tidx];
                            float value2 = temp_smem[tidy+num][tidx];
                            temp_smem[tidy][tidx] = value1 + value2;
                        }
                        num = num >> 1;
                        __syncthreads();
                    }
                    if(tidy == 0)
                        global_sum_scores[tidx] += temp_smem[0][tidx];
                }

                //计算S*V
                cooperative_groups::wait(block_v);

                #pragma unroll
                for(int i = 0;i<4; i++){
                    for(int j=0;j<32;j++){
                        out_temp[tidy*4+i][tidx] += temp_score[j][tidy*4+i]*smem_v[j][tidx];
                        out_temp[tidy*4+i][tidx+32] += temp_score[j][tidy*4+i]*smem_v[j][tidx+32];
                    }
                }
                __syncthreads();

                //加载下一次使用的数据
                if(block_id != to_block_end - 1 || b_bn != 1)
                {
                    cooperative_groups::memcpy_async(block_v, smem_v[0], c+data_b_start+next_bn*32*head_size, sizeof(float)*32*64);
                }
            }
        }

        const int index_x = (tidy%4)*8;
        const int index_y = tidx + (tidy/4)*32;
        // 结果写入global mem
        #pragma unroll
        for(int i=0;i<8;i+=1){
            out[data_offset_a+(a_bm*A_BM+index_x+i)*head_size+index_y] = out_temp[(index_x+i)][index_y] / global_sum_scores[index_x+i];
        }

        __syncthreads();
        // printf("111\n");
    }
}
    //(12,64)(32*8)  a is transpose 
template <class DataType>
__global__ void sparse_attention_(DataType *a,  DataType *b,  DataType *c, 
    DataType *out,const int *seq_len_info,const int *from_block_index, const int *from_block_index_position, const int *to_select_index,const int *to_select_index_position, const int batch_size,
    const int block_size,const int head_size,const int select_block_num){


    const int tidy = threadIdx.y;
    const int tidx = threadIdx.x;
    const int bidx = blockIdx.x;
    const int b_dimx = 8;
    const int g_dimy = gridDim.y;

    const int A_BM = 32;
    const int A_BK = 64;
    const int B_BK = 32;
    const int B_BN = 4;
    const int C_BK = 32;
    const int C_BN = 4;


    __shared__ half smem_q[32 * 64],smem_k[32 * 64],smem_v[32 * 64],smem_temp_half[64][32];
    __shared__ float temp_score[32][32],out_temp[32][64],max_score_diff[32];
    __shared__ half temp_score_half[32][32];

    __shared__ float global_sum_scores[32],pre_max_score[32],max_score[32],temp_smem[32];

    typedef hipcub::WarpReduce<float> WarpReduce;
    __shared__ typename WarpReduce::TempStorage temp_storage[8];
    wmma::fragment<wmma::matrix_a, 8, 32, 16, half, wmma::row_major> frag_q[2];
    wmma::fragment<wmma::matrix_b, 8, 32, 16, half, wmma::col_major> frag_k[2];
    wmma::fragment<wmma::accumulator, 8, 32, 16, half> frag_s_out;

    wmma::fragment<wmma::matrix_a, 8, 32, 16, half, wmma::row_major> frag_s[2];
    wmma::fragment<wmma::matrix_b, 8, 32, 16, half, wmma::row_major> frag_v[2];
    wmma::fragment<wmma::accumulator, 8, 32, 16, float> frag_out;


    float zero4[4] = {0.0f,0.0f,0.0f,0.0f};

    auto block_k = cooperative_groups::this_thread_block();
    auto block_v = cooperative_groups::this_thread_block();


    // 计算Q的起始位置
    const int compute_block_start = from_block_index_position[bidx];
    const int compute_block_end = from_block_index_position[bidx + 1];
    const int compute_block_num = compute_block_end - compute_block_start;
    for(int from_block_id_index = compute_block_start;from_block_id_index<compute_block_end;from_block_id_index++){
        int from_block_id = from_block_index[from_block_id_index];
        int seq_start_block_index = 0;
        int seq_block_len = 0;
        for(int i = 1;i<batch_size+1;i++){
            if(from_block_id >= seq_len_info[i]*12)
                continue;
            else{
                seq_start_block_index = seq_len_info[i-1];
                seq_block_len = seq_len_info[i] - seq_len_info[i-1];
                break;
            }
        }
        const int head_num = (from_block_id - 12 * seq_start_block_index)/seq_block_len;
        from_block_id = (from_block_id - 12 * seq_start_block_index)%seq_block_len;

        const int seq_start_index = 12 * seq_start_block_index * block_size * head_size + head_num * seq_block_len * block_size * head_size;
        const int data_offset_q = seq_start_index + from_block_id*block_size * head_size;


        // if(tidx == 0 && tidy == 0 ){
        //     printf("%d  %d  %d  %d  %d  %d\n",bidx,head_num,from_block_id,seq_start_block_index,seq_start_index,data_offset_a);
        // }
        
        for(int a_bm = 0; a_bm< block_size/A_BM; a_bm++){
            const int to_block_start = to_select_index_position[from_block_id + seq_start_block_index];
            const int to_block_end = to_select_index_position[from_block_id + seq_start_block_index + 1];

            int to_block_id = to_select_index[to_block_start];
            int data_b_start = seq_start_index + to_block_id * block_size * head_size; 

            const int smem_index =  32*8*tidy + tidx*8; // warp_size * 8

            // 加载Q的部分数据,32*64
            // Q  tidy
            //      --------------------
            //          0     |     1
            //      --------------------
            //          2     |     3
            //      --------------------
            //          4     |     5
            //      --------------------
            //          6     |     7
            //      --------------------
            const int smem_index_q = 32*8*tidy + (tidx/16)*16*8 + (tidx%16)*8;
            const int global_q_index_i = (a_bm*A_BM + (tidx%16)/2 + (tidy/2)*8);
            const int global_q_index_j = ((tidx%2)*8 + (tidx/16)*16 + (tidy%2)*32);
            FLOAT4(smem_q[smem_index_q]) = FLOAT4(a[data_offset_q+global_q_index_i*head_size+global_q_index_j]); 
    
            FLOAT4(out_temp[global_q_index_i - a_bm*A_BM][global_q_index_j]) = FLOAT4(zero4[0]);
            wmma::fill_fragment(frag_s_out, 0.0);
            wmma::fill_fragment(frag_out, 0.0);
    
    
            // KT 32*16  tidy  Q*K 8*16*32
            //      -----------------------------------------
            //          0     |     2   |   4     |     6   
            //      -----------------------------------------
            //          1     |     3   |   5     |     7   
            //      -----------------------------------------
    
            const int global_k_index_i = tidx/2 + (tidy%2)*16;
            const int global_k_index_j = (tidx%2)*8 + (tidy/2)*16;
            FLOAT4(smem_k[smem_index]) = FLOAT4(b[data_b_start + global_k_index_i*head_size+global_k_index_j]); 
    
    
            // V 16*32  tidy  S*K 8*16*32
            //      --------------------
            //          0     |     4
            //      --------------------
            //          1     |     5
            //      --------------------
            //          2     |     6
            //      --------------------
            //          3     |     7
            //      --------------------
            const int global_v_index_i = (tidx/4 + ((tidy/2)%2)*16 + (tidy%2)*8);
            const int global_v_index_j = ((tidx*8)%32 + ((tidy/2)/2)*32);
            FLOAT4(smem_v[smem_index]) = FLOAT4(c[data_b_start + global_v_index_i*head_size+global_v_index_j]); 
    
            wmma::load_matrix_sync(frag_q[0], &smem_q[tidy*2*8*16], 16);
            wmma::load_matrix_sync(frag_q[1], &smem_q[(tidy*2+1)*8*16], 16);
    
            // 初始化sharedmem
            if(tidy == 0){
                max_score[tidx] = 0.0f;
                // sum_score_max[tidx] = 0.0f;
                pre_max_score[tidx] = 0.0f;
                global_sum_scores[tidx] = 0.0f;
            }
            __syncthreads();

            // 遍历K、V的每一个Block进行计算
            for(int block_id=to_block_start;block_id<to_block_end;block_id++)
            {
                // KV按照 32*64 的大小进行加载计算
                for(int b_bn=0;b_bn<block_size/32;b_bn++){
                    // 计算Q*K
                    // cooperative_groups::wait(block_k);
                    wmma::fill_fragment(frag_s_out, 0.0);
                    if(tidy % 2 == 0){
                        wmma::load_matrix_sync(frag_k[0], &smem_k[0], 16);
                        wmma::load_matrix_sync(frag_k[1], &smem_k[16*32], 16);
                    }
                    else{
                        wmma::load_matrix_sync(frag_k[0], &smem_k[16*32*2], 16);
                        wmma::load_matrix_sync(frag_k[1], &smem_k[16*32*3], 16);
                    }
                    wmma::mma_sync(frag_s_out, frag_q[0], frag_k[0], frag_s_out);
                    wmma::mma_sync(frag_s_out, frag_q[1], frag_k[1], frag_s_out);
                    wmma::store_matrix_sync(&smem_temp_half[(tidy/2)*8 + (tidy % 2)*32][0], frag_s_out, 32, wmma::mem_row_major);

                    __syncthreads();

                    for(int i=0;i<4;i++){
                        temp_score[i*8+tidy][tidx] = __half2float(smem_temp_half[i*8+tidy][tidx] + smem_temp_half[i*8+tidy+32][tidx]);
                    }

                    //加载下一次使用的数据
                    const int next_block_id = b_bn == 1 ? block_id+1:block_id;
                    const int next_bn = (b_bn + 1) & 1;
                    to_block_id = to_select_index[next_block_id];
                    int data_b_start = seq_start_index + to_block_id * block_size * head_size;

                    if(block_id != to_block_end - 1 || b_bn != 1)
                    {
                        FLOAT4(smem_k[smem_index]) = FLOAT4(b[data_b_start + next_bn*32*head_size + global_k_index_i*head_size+global_k_index_j]);

                    }

                    //计算最大值 rowmax
                    {
                        float value1 = temp_score[tidy][tidx];
                        float value2 = temp_score[tidy+16][tidx];
                        float value3 = temp_score[tidy+8][tidx];
                        float value4 = temp_score[tidy+24][tidx];

                        temp_smem[tidy] = WarpReduce(temp_storage[tidy]).Reduce(value1, hipcub::Max());
                        temp_smem[tidy+16] = WarpReduce(temp_storage[tidy]).Reduce(value2, hipcub::Max());
                        temp_smem[tidy+8] = WarpReduce(temp_storage[tidy]).Reduce(value3, hipcub::Max());
                        temp_smem[tidy+24] = WarpReduce(temp_storage[tidy]).Reduce(value4, hipcub::Max());

                        __syncthreads();
                        if(tidy == 0)
                        {
                            pre_max_score[tidx] = max_score[tidx];
                            max_score[tidx] = max_score[tidx]>temp_smem[tidx]?max_score[tidx]:temp_smem[tidx];
                            max_score_diff[tidx] = exp(pre_max_score[tidx] - max_score[tidx]);
                        }
                    }
                    
                    //计算差值
                    {
                        __syncthreads();
                        for(int i=0;i<4;i++)
                        {
                            float temp =  exp(temp_score[(i+tidy*4)][tidx] - max_score[tidy*4+i]);
                            temp_score[(i+tidy*4)][tidx] = temp;
                            temp_score_half[(i+tidy*4)][tidx] = __float2half(temp);
                        }

                        const int t = (tidy/2)*8+(tidx%4)*2;
                        for(int i=0;i<4;i++){
                            frag_out.x[i*2] *= max_score_diff[t];
                            frag_out.x[i*2+1] *= max_score_diff[t+1];
                        }
                        __syncthreads();  
                    }
                    {
                        float value1 = temp_score[tidy][tidx];
                        float value2 = temp_score[tidy+16][tidx];
                        float value3 = temp_score[tidy+8][tidx];
                        float value4 = temp_score[tidy+24][tidx];
                        
                        temp_smem[tidy] = WarpReduce(temp_storage[tidy]).Sum(value1);
                        temp_smem[tidy+16] = WarpReduce(temp_storage[tidy]).Sum(value2);
                        temp_smem[tidy+8] = WarpReduce(temp_storage[tidy]).Sum(value3);
                        temp_smem[tidy+24] = WarpReduce(temp_storage[tidy]).Sum(value4);
                        
                        __syncthreads();

                        if(tidy == 0)
                        {
                            global_sum_scores[tidx] *= max_score_diff[tidx];
                            global_sum_scores[tidx] += temp_smem[tidx];
                        }
                    }


                    // //计算S*V
                    wmma::load_matrix_sync(frag_s[0], &temp_score_half[(tidy/2)*8][0], 32);
                    wmma::load_matrix_sync(frag_s[1], &temp_score_half[(tidy/2)*8][16], 32);
                    if(tidy % 2 == 0){
                        wmma::load_matrix_sync(frag_v[0], &smem_v[0], 32);
                        wmma::load_matrix_sync(frag_v[1], &smem_v[32*16], 32);
                    }
                    else{
                        wmma::load_matrix_sync(frag_v[0], &smem_v[32*16*2], 32);
                        wmma::load_matrix_sync(frag_v[1], &smem_v[32*16*3], 32);
                    }

                    wmma::mma_sync(frag_out, frag_s[0], frag_v[0], frag_out);
                    wmma::mma_sync(frag_out, frag_s[1], frag_v[1], frag_out);
                    
                    __syncthreads();

                    //加载下一次使用的数据
                    if(block_id != to_block_end - 1 || b_bn != 1)
                    {
                        FLOAT4(smem_v[smem_index]) = FLOAT4(c[data_b_start + next_bn*32*head_size + global_v_index_i*head_size+global_v_index_j]); 
                    }

                }
            }

            wmma::store_matrix_sync(&out_temp[(tidy/2)*8][(tidy % 2)*32], frag_out, 64, wmma::mem_row_major);

            __syncthreads();
    
            const int index_x = (tidy%4)*8;
            const int index_y = tidx + (tidy/4)*32;
            // 结果写入global mem
            #pragma unroll
            for(int i=0;i<8;i+=1){
                out[data_offset_q+(a_bm*A_BM+index_x+i)*head_size+index_y] = out_temp[(index_x+i)][index_y] / global_sum_scores[index_x+i];
            }
            __syncthreads();
        }
    }
}              
      
void test_gemm_1(half *a, half *b,half *c, half *out,int *seq_len_info,int *from_select_index,int *from_select_index_position,int *to_select_index,int *to_select_index_position, int block_num, int head_num,int block_size,int head_size)
{
    // std::cout<<*a<<std::endl;
    // sparse_attention<float><<<dim3(block_num,head_num),dim3(11,32)>>>(a,b,c,out,to_select_index,64,64);

    hipEvent_t start,stop;
    hipEventCreate( &start );
    hipEventCreate( &stop ) ;
    // test_gemm<float><<<1,dim3(11,32)>>>(a,b,c,m,n,k,64);

    hipEventRecord( start, 0 ) ;
    // 修改成最大线程块数量 80 * 2
    sparse_attention_<half><<<dim3(160),dim3(32,8)>>>(a,b,c,out,seq_len_info,from_select_index,from_select_index_position,to_select_index,to_select_index_position,2,64,64,11);

    // test_gpu<<<1,1>>>();
    // test_cpu();
    hipEventRecord(stop,0);
    float elapsedTime;
    hipEventSynchronize(stop);
    hipDeviceSynchronize();
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf( "Time to generate:  %f ms\n", elapsedTime );
    // printf("%f\n",*a);
}

void test_gemm_(float *a, float *b,float *c, float *out,int *to_select_index,int *to_select_index_position, int block_num, int head_num,int block_size,int head_size)
{
    // std::cout<<*a<<std::endl;
    // sparse_attention<float><<<dim3(block_num,head_num),dim3(11,32)>>>(a,b,c,out,to_select_index,64,64);

    hipEvent_t start,stop;
    hipEventCreate( &start );
    hipEventCreate( &stop ) ;
    // test_gemm<float><<<1,dim3(11,32)>>>(a,b,c,m,n,k,64);

    hipEventRecord( start, 0 ) ;
    sparse_attention<float><<<dim3(head_num,block_num),dim3(32,8)>>>(a,b,c,out,to_select_index,to_select_index_position,64,64,11);

    // test_gpu<<<1,1>>>();
    // test_cpu();
    hipEventRecord(stop,0);
    float elapsedTime;
    hipEventSynchronize(stop);
    hipDeviceSynchronize();
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf( "Time to generate:  %f ms\n", elapsedTime );
    // printf("%f\n",*a);
}

void test_gemm_(half *a, half *b,half *c, half *out,int *to_select_index,int *to_select_index_position, int block_num, int head_num,int block_size,int head_size)
{
    // std::cout<<*a<<std::endl;
    // sparse_attention<float><<<dim3(block_num,head_num),dim3(11,32)>>>(a,b,c,out,to_select_index,64,64);

    hipEvent_t start,stop;
    hipEventCreate( &start );
    hipEventCreate( &stop ) ;
    // test_gemm<float><<<1,dim3(11,32)>>>(a,b,c,m,n,k,64);

    hipEventRecord( start, 0 ) ;
    sparse_attention_with_tensor_core<half><<<dim3(head_num,block_num),dim3(32,8)>>>(a,b,c,out,to_select_index,to_select_index_position,64,64,11);

    // test_gpu<<<1,1>>>();
    // test_cpu();
    hipEventRecord(stop,0);
    float elapsedTime;
    hipEventSynchronize(stop);
    hipDeviceSynchronize();
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf( "Time to generate:  %f ms\n", elapsedTime );
    // printf("%f\n",*a);
}


}
}
}