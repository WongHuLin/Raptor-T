#include "hip/hip_runtime.h"
#include "kernel.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <numeric>
#include <cooperative_groups/memcpy_async.h>
#include <thrust/extrema.h>
#include <hip/hip_fp16.h>
// #include <hipcub/hipcub.hpp>
#define FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])
#define FLOAT(pointer) (reinterpret_cast<float*>(&(pointer)))
#define HALF(pointer) (reinterpret_cast<half*>(&(pointer)))
#define HALF2(pointer) (reinterpret_cast<half2*>(&(pointer)))

namespace sparse_transformers {
namespace layers {
namespace kernels {
    //(12,64)(32*8)
//input_data: seq_len * all_head_size * 3   bias: all_head_size  q,k,v: seq * all_head_size
template <class DataType>
__global__ void add_bias_and_transpose(DataType *input_data, DataType *bias, 
    half *q, half *k, half *v, int q_offset, int k_offset, 
    int v_offset,int *seq_len_info,int batch_size, int head_num, int block_size, 
    int head_size, int block_num){
    // For K and V: batch_size * seq_len * all_head_size -> batch_size * head_num * block_num * block_size * head_size
    // For Q: batch_size * seq_len * all_head_size -> batch_size * head_num * block_num * head_size* block_size
    const int tidy = threadIdx.y;
    const int tidx = threadIdx.x;
    const int bidx = blockIdx.x;
    const int bidy = blockIdx.y;
    const int all_head_size = head_num * head_size;
    __shared__ float q_bias[64],k_bias[64],v_bias[64];

    int read_seq_data_start = 0;
    int write_seq_data_start = 0;

    for(int i=1;i<batch_size+1;i++)
        if(bidy >= seq_len_info[i])
            continue;
        else{
            int seq_start_index = seq_len_info[i-1];
            int seq_len_index = bidy - seq_start_index;  //seq内的index
            int len = (seq_len_info[i] - seq_len_info[i-1]); //seq 的长度
            // 数据开始读取为：seq的开始位置 + 当前token block的index开始位置 + 
            read_seq_data_start =  seq_start_index*block_size*head_num*head_size*3 + seq_len_index*block_size*head_num*head_size*3 + bidx*head_size;
            write_seq_data_start = seq_start_index*block_size*head_num*head_size + bidx*len*block_size*head_size + seq_len_index*block_size*head_size;
            // if(tidx == 0 && tidy == 0){
            //     printf("%d %d %d %d %d %d\n",bidx,bidy,seq_start_index*block_size*head_num*head_size,bidx*len*block_size*head_size, seq_len_index*block_size*head_size,write_seq_data_start);
            // }
            break;
        }

    // load bias
    auto block = cooperative_groups::this_thread_block();
    cooperative_groups::memcpy_async(block, q_bias, bias+bidx*head_size+q_offset, sizeof(float)*64);
    cooperative_groups::memcpy_async(block, k_bias, bias+bidx*head_size+all_head_size, sizeof(float)*64);
    cooperative_groups::memcpy_async(block, v_bias, bias+bidx*head_size+all_head_size*2, sizeof(float)*64);
    
    //smem_q 为32*33 是为了转置时，避免bank conflict
    __shared__ float smem_k[16*64],smem_v[16*64],smem_q[16*64];
    int smem_index = (tidy*32+tidx)*4;
    for(int block_row=0;block_row<block_size;block_row+=32){
        for(int block_col=0;block_col<head_size;block_col+=32){

            int smem_row_index = smem_index / 32;
            int smem_col_index = smem_index % 32;
            int read_index = read_seq_data_start + head_num*head_size*3*(block_row+smem_row_index) + block_col + smem_col_index;
            // load q k v

            FLOAT4(smem_k[smem_index]) = FLOAT4(input_data[read_index+k_offset]);
            FLOAT4(smem_v[smem_index]) = FLOAT4(input_data[read_index+v_offset]);
            FLOAT4(smem_q[smem_index]) = FLOAT4(input_data[read_index+q_offset]);

            // smem_q[smem_row_index][smem_col_index] = input_data[read_index+q_offset];
            // smem_q[smem_row_index][smem_col_index+1] = input_data[read_index+q_offset+1];
            // smem_q[smem_row_index][smem_col_index+2] = input_data[read_index+q_offset+2];
            // smem_q[smem_row_index][smem_col_index+3] = input_data[read_index+q_offset+3];

            cooperative_groups::wait(block);
            for(int i=0;i<4;i++){
                smem_k[(tidy*4+i)*32+tidx] += k_bias[tidx+block_col];
                smem_v[(tidy*4+i)*32+tidx] += v_bias[tidx+block_col];
                smem_q[(tidy*4+i)*32+tidx] += q_bias[tidx+block_col];

                // smem_q[(tidy*4+i)][tidx] += q_bias[tidx+block_col];
            }

            int write_index = write_seq_data_start + (block_row+smem_row_index)*head_size + block_col + smem_col_index;
            // if(tidx == 0 && tidy == 0 && block_row == 0 && block_col == 0){
            //     printf("%d %d %f\n",bidx,write_index,smem_k[0]);
            // }
            // __syncthreads();
            // FLOAT4(k[write_index]) = FLOAT4(smem_k[smem_index]);
            // FLOAT4(v[write_index]) = FLOAT4(smem_v[smem_index]);
            // FLOAT4(q[write_index]) = FLOAT4(smem_q[smem_index]);
            for(int i=0;i<4;i++){
                k[write_index+i] = __float2half(smem_k[smem_index]);
                v[write_index+i] = __float2half(smem_v[smem_index]);
                q[write_index+i] = __float2half(smem_q[smem_index]);
            }


            __syncthreads();
            // if(bidx == 0 && bidy == 0  && block_col == 0){
            //     printf("%d %d %d %d %d %d %f\n",tidx,tidy,block_row,block_col,write_seq_data_start,write_seq_data_start + (tidy*4+1+block_col)*head_size + tidx + block_row,smem_q[tidx][(tidy*4+1)]);
            // }
            // for(int i=0;i<4;i++){
            //     q[write_seq_data_start + (tidy*4+i+block_col)*head_size + tidx + block_row] = smem_q[tidx][(tidy*4+i)];
            //     // if(write_seq_data_start + (tidy*4+i+block_col)*head_size + tidx + block_row == 3079)
            //     //     printf("%d %d %d %d %d 3079 %f \n",bidx,bidy,tidx,tidy,write_seq_data_start,smem_q[tidx][(tidy*4+i)]);
            // }
            // __syncthreads();
        }
    }
}

__global__ void add_bias_and_transpose(half *input_data, half *bias, 
    half *q, half *k, half *v, int q_offset, int k_offset, 
    int v_offset,int *seq_len_info,int batch_size, int head_num, int block_size, 
    int head_size, int block_num){
    // For K and V: batch_size * seq_len * all_head_size -> batch_size * head_num * block_num * block_size * head_size
    // For Q: batch_size * seq_len * all_head_size -> batch_size * head_num * block_num * head_size* block_size
    const int tidy = threadIdx.y;
    const int tidx = threadIdx.x;
    const int bidx = blockIdx.x;
    const int bidy = blockIdx.y;
    const int all_head_size = head_num * head_size;
    __shared__ half q_bias[64],k_bias[64],v_bias[64];

    int read_seq_data_start = 0;
    int write_seq_data_start = 0;

    for(int i=1;i<batch_size+1;i++)
        if(bidy >= seq_len_info[i])
            continue;
        else{
            int seq_start_index = seq_len_info[i-1];
            int seq_len_index = bidy - seq_start_index;  //seq内的index
            int len = (seq_len_info[i] - seq_len_info[i-1]); //seq 的长度
            // 数据开始读取为：seq的开始位置 + 当前token block的index开始位置 + 
            read_seq_data_start =  seq_start_index*block_size*head_num*head_size*3 + seq_len_index*block_size*head_num*head_size*3 + bidx*head_size;
            write_seq_data_start = seq_start_index*block_size*head_num*head_size + bidx*len*block_size*head_size + seq_len_index*block_size*head_size;
            // if(tidx == 0 && tidy == 0){
            //     printf("%d %d %d %d %d %d\n",bidx,bidy,seq_start_index*block_size*head_num*head_size,bidx*len*block_size*head_size, seq_len_index*block_size*head_size,write_seq_data_start);
            // }
            break;
        }

    // load bias
    auto block = cooperative_groups::this_thread_block();
    cooperative_groups::memcpy_async(block, q_bias, bias+bidx*head_size+q_offset, sizeof(half)*64);
    cooperative_groups::memcpy_async(block, k_bias, bias+bidx*head_size+all_head_size, sizeof(half)*64);
    cooperative_groups::memcpy_async(block, v_bias, bias+bidx*head_size+all_head_size*2, sizeof(half)*64);

    half2 q_bias_re,k_bias_re,v_bias_re;

    __shared__ half smem_k[16*64],smem_v[16*64],smem_q[16*64];
    int smem_index = (tidy*32+tidx)*8;
    for(int block_row=0;block_row<block_size;block_row+=16){
        int smem_row_index = smem_index / 64;
        int smem_col_index = smem_index % 64;
        int read_index = read_seq_data_start + head_num*head_size*3*(block_row+smem_row_index) + smem_col_index;

        FLOAT4(smem_k[smem_index]) = FLOAT4(input_data[read_index+k_offset]);
        FLOAT4(smem_v[smem_index]) = FLOAT4(input_data[read_index+v_offset]);
        FLOAT4(smem_q[smem_index]) = FLOAT4(input_data[read_index+q_offset]);

        cooperative_groups::wait(block);

        if(block_row == 0)
        {
            q_bias_re = HALF2(q_bias)[tidx];
            k_bias_re = HALF2(k_bias)[tidx];
            v_bias_re = HALF2(v_bias)[tidx];
        }

        for(int i=0;i<4;i++){
            half2 q_temp =  HALF2(smem_q[(tidy*4+i)*64])[tidx];
            half2 k_temp =  HALF2(smem_k[(tidy*4+i)*64])[tidx];
            half2 v_temp =  HALF2(smem_v[(tidy*4+i)*64])[tidx];
            HALF2(smem_q[(tidy*4+i)*64])[tidx] = __hadd2(q_bias_re,q_temp);
            HALF2(smem_k[(tidy*4+i)*64])[tidx] = __hadd2(k_bias_re,k_temp);
            HALF2(smem_v[(tidy*4+i)*64])[tidx] = __hadd2(v_bias_re,v_temp);
        }
        __syncthreads();
        int write_index = write_seq_data_start + (block_row+smem_row_index)*head_size  + smem_col_index;

        FLOAT4(q[write_index]) =  FLOAT4(smem_q[smem_index]);
        FLOAT4(k[write_index]) =  FLOAT4(smem_k[smem_index]);
        FLOAT4(v[write_index]) =  FLOAT4(smem_v[smem_index]);
        __syncthreads();
    }
}


void test_add_bias_and_transpose(float *bias,float *input_data,half *q, half *k,half *v, int q_offset, int k_offset, int v_offset,int *seq_len_info,int batch_size, int head_num, int block_size,int block_num, int head_size){
//     hipEvent_t start,stop;
//     hipEventCreate( &start );
//     hipEventCreate( &stop ) ;
//     hipEventRecord( start, 0 ) ;
    add_bias_and_transpose<float><<<dim3(head_num,block_num),dim3(32,8)>>>(input_data,bias,q,k,v,q_offset,k_offset,v_offset,seq_len_info,batch_size,head_num,block_size,head_size,block_num);
//     hipEventRecord(stop,0);
//     float elapsedTime;
//     hipEventSynchronize(stop);
//     hipDeviceSynchronize();
//     hipEventElapsedTime(&elapsedTime, start, stop);
//     printf( "Time to generate:  %f ms\n", elapsedTime );
}

void test_add_bias_and_transpose(half *bias,half *input_data,half *q, half *k,half *v, int q_offset, int k_offset, int v_offset,int *seq_len_info,int batch_size, int head_num, int block_size,int block_num, int head_size){
        // hipEvent_t start,stop;
        // hipEventCreate( &start );
        // hipEventCreate( &stop ) ;
        // hipEventRecord( start, 0 ) ;
        add_bias_and_transpose<<<dim3(head_num,block_num),dim3(32,4)>>>(input_data,bias,q,k,v,q_offset,k_offset,v_offset,seq_len_info,batch_size,head_num,block_size,head_size,block_num);
        // hipEventRecord(stop,0);
        // float elapsedTime;
        // hipEventSynchronize(stop);
        // hipDeviceSynchronize();
        // hipEventElapsedTime(&elapsedTime, start, stop);
        // printf( "test_add_bias_and_transpose   Time to generate:  %f ms\n", elapsedTime );
    }
}
}
}