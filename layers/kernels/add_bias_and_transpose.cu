#include "hip/hip_runtime.h"
#include "kernel.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <numeric>
#include <cooperative_groups/memcpy_async.h>
#include <thrust/extrema.h>
// #include <hipcub/hipcub.hpp>
#define FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])
#define FLOAT(pointer) (reinterpret_cast<float*>(&(pointer)))

namespace sparse_transformers {
namespace layers {
namespace kernels {
    //(12,64)(32*8)
//input_data: seq_len * all_head_size * 3   bias: all_head_size  q,k,v: seq * all_head_size
template <class DataType>
__global__ void add_bias_and_transpose(DataType *input_data, DataType *bias, 
    DataType *q, DataType *k, DataType *v, int q_offset, int k_offset, 
    int v_offset,int batch_size, int seq_len, int head_num, int block_size, 
    int head_size, int block_num){
    // For K and V: batch_size * seq_len * all_head_size -> batch_size * head_num * block_num * blcok_size * head_size
    // For Q: batch_size * seq_len * all_head_size -> batch_size * head_num * block_num * head_size* blcok_size
    const int tidy = threadIdx.y;
    const int tidx = threadIdx.x;
    const int bidx = blockIdx.x;
    const int bidy = blockIdx.y;
    const int all_head_size = head_num * head_size;
    const int start_read_data_index = bidy * block_size * head_size * 3 * head_num + bidx * head_size;
    const int start_write_data_index = bidx * block_num * block_size * head_size + bidy * block_size * head_size;
    __shared__ float q_bias[64],k_bias[64],v_bias[64];

    // load bias
    auto block = cooperative_groups::this_thread_block();
    cooperative_groups::memcpy_async(block, q_bias, bias+bidx*head_size+q_offset, sizeof(float)*64);
    cooperative_groups::memcpy_async(block, k_bias, bias+bidx*head_size+all_head_size, sizeof(float)*64);
    cooperative_groups::memcpy_async(block, v_bias, bias+bidx*head_size+all_head_size*2, sizeof(float)*64);
    
    //smem_q 为32*33 是为了转置时，避免bank conflict
    __shared__ float smem_k[16*64],smem_v[16*64],smem_q[32][33];
    int smem_index = (tidy*32+tidx)*4;
    for(int block_row=0;block_row<block_size;block_row+=32){
        for(int block_col=0;block_col<head_size;block_col+=32){

            int smem_row_index = smem_index / 32;
            int smem_col_index = smem_index % 32;
            int read_index = start_read_data_index + head_num*head_size*3*(block_row+smem_row_index) + block_col + smem_col_index;
            // load q k v

            FLOAT4(smem_k[smem_index]) = FLOAT4(input_data[read_index+k_offset]);
            FLOAT4(smem_v[smem_index]) = FLOAT4(input_data[read_index+v_offset]);
            smem_q[smem_row_index][smem_col_index] = input_data[read_index+q_offset];
            smem_q[smem_row_index][smem_col_index+1] = input_data[read_index+q_offset+1];
            smem_q[smem_row_index][smem_col_index+2] = input_data[read_index+q_offset+2];
            smem_q[smem_row_index][smem_col_index+3] = input_data[read_index+q_offset+3];

            cooperative_groups::wait(block);
            for(int i=0;i<4;i++){
                smem_k[(tidy*4+i)*32+tidx] += k_bias[tidx+block_col];
                smem_v[(tidy*4+i)*32+tidx] += v_bias[tidx+block_col];
                smem_q[(tidy*4+i)][tidx] += q_bias[tidx+block_col];
            }

            int write_index = start_write_data_index + (block_row+smem_row_index)*head_size + block_col + smem_col_index;
            FLOAT4(k[write_index]) = FLOAT4(smem_k[smem_index]);
            FLOAT4(v[write_index]) = FLOAT4(smem_v[smem_index]);

            __syncthreads();
            for(int i=0;i<4;i++){
                q[start_write_data_index + (tidy*4+i+block_col)*64 + tidx + block_row] = smem_q[tidx][(tidy*4+i)];
            }
            __syncthreads();
        }
    }
}

void test_add_bias_and_transpose(float *bias,float *input_data,float *q, float *k,float *v, int q_offset, int k_offset, int v_offset,int batch_size, int seq_len, int head_num, int block_size,int block_num, int head_size){
    hipEvent_t start,stop;
    hipEventCreate( &start );
    hipEventCreate( &stop ) ;
    hipEventRecord( start, 0 ) ;
    add_bias_and_transpose<float><<<dim3(head_num,block_num),dim3(32,8)>>>(input_data,bias,q,k,v,q_offset,k_offset,v_offset,batch_size,seq_len,head_num,block_size,head_size,block_num);
    hipEventRecord(stop,0);
    float elapsedTime;
    hipEventSynchronize(stop);
    hipDeviceSynchronize();
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf( "Time to generate:  %f ms\n", elapsedTime );

}
}
}
}