#include "hip/hip_runtime.h"
#include "attention.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <cooperative_groups/memcpy_async.h>
#include <thrust/extrema.h>

#define FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])


// lock-based
__device__ volatile int g_mutex;
// GPU lock-based synchronization function
__device__ void __gpu_sync(int goalVal )
{
    // thread ID in a block
    int tid_in_block = threadIdx.x * blockDim.y + threadIdx.y;
    // only thread 0 is used for synchronization
    if (tid_in_block == 0)
    {
    	atomicAdd((int*) &g_mutex, 1);
    	// only when all blocks add 1 go g_mutex
    	// will g_mutex equal to goalVal
    	while (g_mutex != goalVal)
    	{
    		// Do nothing here
    	}
    }
    __syncthreads();
}
  

template <class DataType>
__global__ void sparse_attention(DataType *a,  DataType *b,  DataType *c, DataType *out,const int *select_index,const int block_size,const int head_size,const int select_block_num){


    const int tidy = threadIdx.y;
    const int tidx = threadIdx.x;
    const int bidx = blockIdx.x;
    const int bidy = blockIdx.y;
    const int b_dimx = 8;
    const int g_dimy = gridDim.y;

    // 计算Q的起始位置
    const int data_offset_a = (bidx*g_dimy + bidy) * block_size * head_size;

    const int A_BM = 32;
    const int A_BK = 64;
    const int B_BK = 32;
    const int B_BN = 4;
    const int C_BK = 32;
    const int C_BN = 4;


    __shared__ float smem_q[64][32],smem_k[32][64],temp_score[32][32],smem_v[32][64];

    __shared__ float out_temp[32][64],global_sum_scores[32],temp_smem[16][32],pre_max_score[32],max_score[32];

    float zero4[4] = {0.0f,0.0f,0.0f,0.0f};

    auto block_k = cooperative_groups::this_thread_block();
    auto block_v = cooperative_groups::this_thread_block();


    const int block_dim_x = blockDim.x;

    for(int a_bm = 0; a_bm< block_size/A_BM; a_bm++){
        
        int data_b_start = (select_index[(bidx*g_dimy+bidy)*11+0]*g_dimy +bidy) * block_size * head_size;
        
        cooperative_groups::memcpy_async(block_k, smem_k[0], b+data_b_start, sizeof(float)*32*64);
        cooperative_groups::memcpy_async(block_v, smem_v[0], c+data_b_start, sizeof(float)*32*64);


        const int smem_index =  32*8*tidy + tidx*8; // warp_size * 8
        const int global_a_index_i = (smem_index / 32 );
        const int global_a_index_j = (smem_index % 32 + a_bm*A_BM);

        // 加载Q的部分数据,32*64
        #pragma unroll
        for(int i=0;i<8;i+=4){
            FLOAT4(smem_q[smem_index/32][smem_index % 32+i]) = FLOAT4(a[data_offset_a + global_a_index_i*head_size+global_a_index_j+i]); 
            FLOAT4(out_temp[smem_index/64][smem_index % 64+i]) = FLOAT4(zero4[0]);
        }

        // for(int i=0;i<64;i++)
        //     cooperative_groups::memcpy_async(block_k, smem_k[i], a+data_offset_a+i*64+a_bm*32, sizeof(float)*32);

        // for(int i=0;i<4;i++)
        // {
        //     out_temp[tidy*4+i][tidx] = 0.0f;
        //     out_temp[tidy*4+i][tidx+32] = 0.0f;
        // }
        
        // 初始化sharedmem
        if(tidy == 0){
            max_score[tidx] = 0.0f;
            // sum_score_max[tidx] = 0.0f;
            pre_max_score[tidx] = 0.0f;
            global_sum_scores[tidx] = 0.0f;
        }

        

        __syncthreads();

        // float temp_q[64];
        // for(int i=0;i<64;i++)
        // {
        //     temp_q[i] = smem_q[i][tidx];
        // }
        // __syncthreads();

        // 遍历K、V的每一个Block进行计算
        for(int block_id=0;block_id<select_block_num;block_id++)
        {
            // 计算KV块的起始位置
            const int data_offset_b = (select_index[(bidx*g_dimy+bidy)*11+block_id]*g_dimy +bidy) * block_size * head_size;
            
            // KV按照 32*64 的大小进行加载计算
            for(int b_bn=0;b_bn<block_size/32;b_bn++){
                
                cooperative_groups::wait(block_k);

                // 计算Q*K
                // for(int i=0;i<4;i++){
                //     float4 t = {0.0f,0.0f,0.0f,0.0f};
                //     float4 k_re[2];
                //     k_re[0] = FLOAT4(smem_k[tidy*4+i][0]);
                //     for(int j=0;j<16;j++){
                //         if(j != 15)
                //             k_re[1&(j+1)] = FLOAT4(smem_k[tidy*4+i][(j+1)*4]);
                //         t.x += k_re[j&1].x*temp_q[j*4];
                //         t.y += k_re[j&1].y*temp_q[j*4+1];
                //         t.z += k_re[j&1].z*temp_q[j*4+2];
                //         t.w += k_re[j&1].w*temp_q[j*4+3];
                //     }
                //     temp_score[tidy*4+i][tidx] = (t.x + t.y + t.z + t.w);
                // }


                for(int i=0;i<4;i++){
                    float temp = 0.0f;
                    for(int j=0;j<64;j++){
                        temp += smem_q[j][tidx] * smem_k[tidy*4+i][j];
                    }
                    temp_score[tidy*4+i][tidx] = temp;
                }


                const int next_block_id = b_bn == 1 ? block_id+1:block_id;
                const int next_bn = (b_bn + 1) & 1;
                const int data_b_start = (select_index[(bidx*g_dimy+bidy)*11+next_block_id]*g_dimy +bidy) * block_size * head_size;
                __syncthreads();
                if(block_id != select_block_num - 1 || b_bn != 1)
                {
                    cooperative_groups::memcpy_async(block_k, smem_k[0], b+data_b_start+next_bn*32*head_size, sizeof(float)*32*64);
                }
            
                // if(a_bm == 0 && block_id == 0 && b_bn == 1 && tidx == 0 && tidy == 0 && bidx == 0 && bidy == 0)
                // {
                //     for(int i=0;i<32;i++)
                //     {
                //         for(int j=0;j<32;j++)
                //             printf("%.f ",temp_score[i][j]);
                //         printf("\n");
                //     }
                // }
                // __syncthreads();
                //计算最大值 rowmax
                {
                    int num = 16;
                    while(num >= 1)
                    {
                        if(num == 16)
                        {
                            float value1 = temp_score[tidy][tidx];
                            float value2 = temp_score[tidy+16][tidx];
                            float value3 = temp_score[tidy+8][tidx];
                            float value4 = temp_score[tidy+24][tidx];

                            temp_smem[tidy][tidx] = value1>value2?value1:value2;
                            temp_smem[tidy+8][tidx] = value3>value4?value3:value4;
                        }
                        else if(tidy < num){
                            float value1 = temp_smem[tidy][tidx];
                            float value2 = temp_smem[tidy+num][tidx];
                            temp_smem[tidy][tidx] = value1>value2?value1:value2;
                        }
                        num = num >> 1;
                        __syncthreads();
                    }
                    if(tidy == 0)
                    {
                        pre_max_score[tidx] = max_score[tidx];
                        max_score[tidx] = max_score[tidx]>temp_smem[0][tidx]?max_score[tidx]:temp_smem[0][tidx];
                    }
                }

                // __syncthreads();
                // if( a_bm == 0 && block_id == 0 && b_bn == 1 && tidx == 0 && tidy == 0 && bidx == 0 && bidy == 0)
                // {
                //     for(int i=0;i<32;i++)
                //         printf("%f ",max_score[i]);
                //     printf("\n");
                // }
                // __syncthreads();
                //计算差值

                {
                    __syncthreads();
                    for(int i=0;i<4;i++)
                    {
                        float temp =  exp(temp_score[(i+tidy*4)][tidx] - max_score[tidx]);
                        temp_score[(i+tidy*4)][tidx] = temp;
                    }

                    #pragma unroll
                    for(int i=0;i<4;i++){
                        float diff = (pre_max_score[tidy*4+i] - max_score[tidy*4+i]);
                        if(diff != 0){
                            diff = exp(diff);
                            out_temp[tidy*4+i][tidx] *= diff;
                            out_temp[tidy*4+i][tidx+32] *= diff;
                        }
                    }
                    __syncthreads();
                    if(tidy == 0){
                        float diff = exp(pre_max_score[tidx] - max_score[tidx]);
                        global_sum_scores[tidx] *= diff;
                    }
                    

                }
                {
                    int num = 16;
                    while(num >= 1)
                    {
                        if(num == 16)
                        {
                            int value1 = temp_score[tidy][tidx];
                            int value2 = temp_score[tidy+16][tidx];
                            int value3 = temp_score[tidy+8][tidx];
                            int value4 = temp_score[tidy+24][tidx];
                            temp_smem[tidy][tidx] = value1 + value2;
                            temp_smem[tidy+8][tidx] = value3 + value4;
                        }
                        else if(tidy < num){
                            int value1 = temp_smem[tidy][tidx];
                            int value2 = temp_smem[tidy+num][tidx];
                            temp_smem[tidy][tidx] = value1 + value2;
                        }
                        num = num >> 1;
                        __syncthreads();
                    }
                    if(tidy == 0)
                        global_sum_scores[tidx] += temp_smem[0][tidx];
                }

                //计算S*V
                cooperative_groups::wait(block_v);

                #pragma unroll
                for(int i = 0;i<4; i++){
                    for(int j=0;j<32;j++){
                        out_temp[tidy*4+i][tidx] += temp_score[j][tidy*4+i]*smem_v[j][tidx];
                        out_temp[tidy*4+i][tidx+32] += temp_score[j][tidy*4+i]*smem_v[j][tidx+32];
                    }
                }
                __syncthreads();
                if(block_id != select_block_num - 1 || b_bn != 1)
                {
                    cooperative_groups::memcpy_async(block_v, smem_v[0], c+data_b_start+next_bn*32*head_size, sizeof(float)*32*64);
                }
            }
        }

        const int index_x = (tidy%4)*8;
        const int index_y = tidx + (tidy/4)*32;
        // 结果写入global mem
        #pragma unroll
        for(int i=0;i<8;i+=1){
            out[data_offset_a+(a_bm*A_BM+index_x+i)*head_size+index_y] = out_temp[(index_x+i)][index_y] / global_sum_scores[index_x+i];
        }

        __syncthreads();
        // printf("111\n");
    }
}


void test_cpu(){
    float a = 1008521344.0;
    float b = 3995.0;
    float c = 19228.0;
    float d = 0;
    d = a + b * c;
    printf("%.6f %.6f %.6f %.6f\n",a,b,c,d);
}

void test_gemm_(float *a, float *b,float *c, float *out,int *select_index, int block_num, int head_num,int block_size,int head_size)
{
    // std::cout<<*a<<std::endl;
    // sparse_attention<float><<<dim3(block_num,head_num),dim3(11,32)>>>(a,b,c,out,select_index,64,64);

    hipEvent_t start,stop;
    hipEventCreate( &start );
    hipEventCreate( &stop ) ;
    // test_gemm<float><<<1,dim3(11,32)>>>(a,b,c,m,n,k,64);

    hipEventRecord( start, 0 ) ;
    sparse_attention<float><<<dim3(block_num,head_num),dim3(32,8)>>>(a,b,c,out,select_index,64,64,11);

    // test_gpu<<<1,1>>>();
    // test_cpu();
    hipEventRecord(stop,0);
    float elapsedTime;
    hipEventSynchronize(stop);
    hipDeviceSynchronize();
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf( "Time to generate:  %f ms\n", elapsedTime );
    // printf("%f\n",*a);

}
